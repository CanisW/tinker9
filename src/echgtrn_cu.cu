#include "hip/hip_runtime.h"
#include "add.h"
#include "echgtrn.h"
#include "glob.spatial.h"
#include "image.h"
#include "launch.h"
#include "md.h"
#include "mod.chgpot.h"
#include "seq_bsplgen.h"
#include "seq_pair_chgtrn.h"
#include "seq_switch.h"
#include "seq_triangle.h"
#include "switch.h"
#include "tool/gpu_card.h"
#include <cassert>


namespace tinker {
template <class Ver>
__global__
void echgtrn_cu1(
   int n, TINKER_IMAGE_PARAMS, count_buffer restrict nc,
   energy_buffer restrict ec, virial_buffer restrict vc, grad_prec* restrict gx,
   grad_prec* restrict gy, grad_prec* restrict gz, real cut, real off,
   const unsigned* restrict minfo, int nexclude,
   const int (*restrict exclude)[2], const real (*restrict exclude_scale)[3],
   const real* restrict x, const real* restrict y, const real* restrict z,
   const Spatial::SortedAtom* restrict sorted, int nakpl,
   const int* restrict iakpl, int niak, const int* restrict iak,
   const int* restrict lst, real* restrict chgct, real* restrict dmpct, real f)
{
   constexpr bool do_a = Ver::a;
   constexpr bool do_e = Ver::e;
   constexpr bool do_v = Ver::v;
   constexpr bool do_g = Ver::g;


   const int ithread = threadIdx.x + blockIdx.x * blockDim.x;
   const int iwarp = ithread / WARP_SIZE;
   const int nwarp = blockDim.x * gridDim.x / WARP_SIZE;
   const int ilane = threadIdx.x & (WARP_SIZE - 1);


   int nctl;
   if CONSTEXPR (do_a) {
      nctl = 0;
   }
   using ebuf_prec = energy_buffer_traits::type;
   ebuf_prec ectl;
   if CONSTEXPR (do_e) {
      ectl = 0;
   }
   using vbuf_prec = virial_buffer_traits::type;
   vbuf_prec vctlxx, vctlyx, vctlzx, vctlyy, vctlzy, vctlzz;
   if CONSTEXPR (do_v) {
      vctlxx = 0;
      vctlyx = 0;
      vctlzx = 0;
      vctlyy = 0;
      vctlzy = 0;
      vctlzz = 0;
   }


   real shxi;
   real shyi;
   real shzi;
   real xk;
   real yk;
   real zk;
   real shgxi;
   real shgyi;
   real shgzi;
   real gxk;
   real gyk;
   real gzk;
   real shchgi;
   real shalphai;
   real chgk;
   real alphak;


   //* /
   // exclude
   for (int ii = ithread; ii < nexclude; ii += blockDim.x * gridDim.x) {
      if CONSTEXPR (do_g) {
         shgxi = 0;
         shgyi = 0;
         shgzi = 0;
         gxk = 0;
         gyk = 0;
         gzk = 0;
      }


      int shi = exclude[ii][0];
      int k = exclude[ii][1];
      real scalea = exclude_scale[ii][0];


      real xi = x[shi];
      real yi = y[shi];
      real zi = z[shi];
      xk = x[k];
      yk = y[k];
      zk = z[k];
      real chgi = chgct[shi];
      real alphai = dmpct[shi];
      chgk = chgct[k];
      alphak = dmpct[k];


      constexpr bool incl = true;
      real xr = xk - xi;
      real yr = yk - yi;
      real zr = zk - zi;
      real r2 = image2(xr, yr, zr);
      if (r2 <= off * off and incl) {
         real r = REAL_SQRT(r2);
         e_prec e, de;
         pair_chgtrn<do_g>(r, cut, off, scalea, f, alphai, chgi, alphak, chgk,
                           e, de);
         if CONSTEXPR (do_a)
            if (e != 0)
               nctl += 1;
         if CONSTEXPR (do_e)
            ectl += cvt_to<ebuf_prec>(e);
         if CONSTEXPR (do_g) {
            de *= REAL_RECIP(r);
            real dedx = de * xr;
            real dedy = de * yr;
            real dedz = de * zr;

            shgxi -= dedx;
            shgyi -= dedy;
            shgzi -= dedz;
            gxk += dedx;
            gyk += dedy;
            gzk += dedz;

            if CONSTEXPR (do_v) {
               vctlxx += cvt_to<vbuf_prec>(xr * dedx);
               vctlyx += cvt_to<vbuf_prec>(yr * dedx);
               vctlzx += cvt_to<vbuf_prec>(zr * dedx);
               vctlyy += cvt_to<vbuf_prec>(yr * dedy);
               vctlzy += cvt_to<vbuf_prec>(zr * dedy);
               vctlzz += cvt_to<vbuf_prec>(zr * dedz);
            }
         }
      } // end if (include)


      if CONSTEXPR (do_g) {
         atomic_add(shgxi, gx, shi);
         atomic_add(shgyi, gy, shi);
         atomic_add(shgzi, gz, shi);
         atomic_add(gxk, gx, k);
         atomic_add(gyk, gy, k);
         atomic_add(gzk, gz, k);
      }
   }
   // */


   //* /
   // block pairs that have scale factors
   for (int iw = iwarp; iw < nakpl; iw += nwarp) {
      if CONSTEXPR (do_g) {
         shgxi = 0;
         shgyi = 0;
         shgzi = 0;
         gxk = 0;
         gyk = 0;
         gzk = 0;
      }


      int tri, tx, ty;
      tri = iakpl[iw];
      tri_to_xy(tri, tx, ty);


      int shiid = ty * WARP_SIZE + ilane;
      int shatomi = min(shiid, n - 1);
      int shi = sorted[shatomi].unsorted;
      int kid = tx * WARP_SIZE + ilane;
      int atomk = min(kid, n - 1);
      int k = sorted[atomk].unsorted;
      shxi = sorted[shatomi].x;
      shyi = sorted[shatomi].y;
      shzi = sorted[shatomi].z;
      xk = sorted[atomk].x;
      yk = sorted[atomk].y;
      zk = sorted[atomk].z;


      shchgi = chgct[shi];
      shalphai = dmpct[shi];
      chgk = chgct[k];
      alphak = dmpct[k];


      unsigned int minfo0 = minfo[iw * WARP_SIZE + ilane];


      for (int j = 0; j < WARP_SIZE; ++j) {
         int srclane = (ilane + j) & (WARP_SIZE - 1);
         int srcmask = 1 << srclane;

         int iid = shiid;
         real xi = shxi;
         real yi = shyi;
         real zi = shzi;
         real chgi = shchgi;
         real alphai = shalphai;


         bool incl = iid < kid and kid < n;
         incl = incl and (minfo0 & srcmask) == 0;
         real scalea = 1;
         real xr = xk - xi;
         real yr = yk - yi;
         real zr = zk - zi;
         real r2 = image2(xr, yr, zr);
         if (r2 <= off * off and incl) {
            real r = REAL_SQRT(r2);
            e_prec e, de;
            pair_chgtrn<do_g>(r, cut, off, scalea, f, alphai, chgi, alphak,
                              chgk, e, de);
            if CONSTEXPR (do_a)
               if (e != 0)
                  nctl += 1;
            if CONSTEXPR (do_e)
               ectl += cvt_to<ebuf_prec>(e);
            if CONSTEXPR (do_g) {
               de *= REAL_RECIP(r);
               real dedx = de * xr;
               real dedy = de * yr;
               real dedz = de * zr;

               shgxi -= dedx;
               shgyi -= dedy;
               shgzi -= dedz;
               gxk += dedx;
               gyk += dedy;
               gzk += dedz;

               if CONSTEXPR (do_v) {
                  vctlxx += cvt_to<vbuf_prec>(xr * dedx);
                  vctlyx += cvt_to<vbuf_prec>(yr * dedx);
                  vctlzx += cvt_to<vbuf_prec>(zr * dedx);
                  vctlyy += cvt_to<vbuf_prec>(yr * dedy);
                  vctlzy += cvt_to<vbuf_prec>(zr * dedy);
                  vctlzz += cvt_to<vbuf_prec>(zr * dedz);
               }
            }
         } // end if (include)


         shiid = __shfl_sync(ALL_LANES, shiid, ilane + 1);
         shxi = __shfl_sync(ALL_LANES, shxi, ilane + 1);
         shyi = __shfl_sync(ALL_LANES, shyi, ilane + 1);
         shzi = __shfl_sync(ALL_LANES, shzi, ilane + 1);
         shchgi = __shfl_sync(ALL_LANES, shchgi, ilane + 1);
         shalphai = __shfl_sync(ALL_LANES, shalphai, ilane + 1);
         if CONSTEXPR (do_g) {
            shgxi = __shfl_sync(ALL_LANES, shgxi, ilane + 1);
            shgyi = __shfl_sync(ALL_LANES, shgyi, ilane + 1);
            shgzi = __shfl_sync(ALL_LANES, shgzi, ilane + 1);
         }
      }


      if CONSTEXPR (do_g) {
         atomic_add(shgxi, gx, shi);
         atomic_add(shgyi, gy, shi);
         atomic_add(shgzi, gz, shi);
         atomic_add(gxk, gx, k);
         atomic_add(gyk, gy, k);
         atomic_add(gzk, gz, k);
      }
   }
   // */


   //* /
   // block-atoms
   for (int iw = iwarp; iw < niak; iw += nwarp) {
      if CONSTEXPR (do_g) {
         shgxi = 0;
         shgyi = 0;
         shgzi = 0;
         gxk = 0;
         gyk = 0;
         gzk = 0;
      }


      int ty = iak[iw];
      int shatomi = ty * WARP_SIZE + ilane;
      int shi = sorted[shatomi].unsorted;
      int atomk = lst[iw * WARP_SIZE + ilane];
      int k = sorted[atomk].unsorted;
      shxi = sorted[shatomi].x;
      shyi = sorted[shatomi].y;
      shzi = sorted[shatomi].z;
      xk = sorted[atomk].x;
      yk = sorted[atomk].y;
      zk = sorted[atomk].z;


      shchgi = chgct[shi];
      shalphai = dmpct[shi];
      chgk = chgct[k];
      alphak = dmpct[k];


      for (int j = 0; j < WARP_SIZE; ++j) {

         real xi = shxi;
         real yi = shyi;
         real zi = shzi;
         real chgi = shchgi;
         real alphai = shalphai;


         bool incl = atomk > 0;
         real scalea = 1;
         real xr = xk - xi;
         real yr = yk - yi;
         real zr = zk - zi;
         real r2 = image2(xr, yr, zr);
         if (r2 <= off * off and incl) {
            real r = REAL_SQRT(r2);
            e_prec e, de;
            pair_chgtrn<do_g>(r, cut, off, scalea, f, alphai, chgi, alphak,
                              chgk, e, de);
            if CONSTEXPR (do_a)
               if (e != 0)
                  nctl += 1;
            if CONSTEXPR (do_e)
               ectl += cvt_to<ebuf_prec>(e);
            if CONSTEXPR (do_g) {
               de *= REAL_RECIP(r);
               real dedx = de * xr;
               real dedy = de * yr;
               real dedz = de * zr;

               shgxi -= dedx;
               shgyi -= dedy;
               shgzi -= dedz;
               gxk += dedx;
               gyk += dedy;
               gzk += dedz;

               if CONSTEXPR (do_v) {
                  vctlxx += cvt_to<vbuf_prec>(xr * dedx);
                  vctlyx += cvt_to<vbuf_prec>(yr * dedx);
                  vctlzx += cvt_to<vbuf_prec>(zr * dedx);
                  vctlyy += cvt_to<vbuf_prec>(yr * dedy);
                  vctlzy += cvt_to<vbuf_prec>(zr * dedy);
                  vctlzz += cvt_to<vbuf_prec>(zr * dedz);
               }
            }
         } // end if (include)


         shxi = __shfl_sync(ALL_LANES, shxi, ilane + 1);
         shyi = __shfl_sync(ALL_LANES, shyi, ilane + 1);
         shzi = __shfl_sync(ALL_LANES, shzi, ilane + 1);
         shchgi = __shfl_sync(ALL_LANES, shchgi, ilane + 1);
         shalphai = __shfl_sync(ALL_LANES, shalphai, ilane + 1);
         if CONSTEXPR (do_g) {
            shgxi = __shfl_sync(ALL_LANES, shgxi, ilane + 1);
            shgyi = __shfl_sync(ALL_LANES, shgyi, ilane + 1);
            shgzi = __shfl_sync(ALL_LANES, shgzi, ilane + 1);
         }
      }


      if CONSTEXPR (do_g) {
         atomic_add(shgxi, gx, shi);
         atomic_add(shgyi, gy, shi);
         atomic_add(shgzi, gz, shi);
         atomic_add(gxk, gx, k);
         atomic_add(gyk, gy, k);
         atomic_add(gzk, gz, k);
      }
   }
   // */


   if CONSTEXPR (do_a) {
      atomic_add(nctl, nc, ithread);
   }
   if CONSTEXPR (do_e) {
      atomic_add(ectl, ec, ithread);
   }
   if CONSTEXPR (do_v) {
      atomic_add(vctlxx, vctlyx, vctlzx, vctlyy, vctlzy, vctlzz, vc, ithread);
   }
} // generated by ComplexKernelBuilder (ck.py) 1.5.3


template <class Ver>
void echgtrn_cu2()
{
   const auto& st = *mspatial_v2_unit;
   real cut = switch_cut(switch_chgtrn);
   real off = switch_off(switch_chgtrn);
   real f = electric / dielec;


   assert(ctrntyp == chgtrn_t::SEPARATE);
   int ngrid = get_grid_size(BLOCK_DIM);
   echgtrn_cu1<Ver><<<ngrid, BLOCK_DIM, 0, g::s0>>>(
      st.n, TINKER_IMAGE_ARGS, nct, ect, vir_ect, dectx, decty, dectz, cut, off,
      st.si1.bit0, nmdwexclude, mdwexclude, mdwexclude_scale, st.x, st.y, st.z,
      st.sorted, st.nakpl, st.iakpl, st.niak, st.iak, st.lst, chgct, dmpct, f);
}


void echgtrn_cu(int vers)
{
   if (vers == calc::v0)
      echgtrn_cu2<calc::V0>();
   else if (vers == calc::v1)
      echgtrn_cu2<calc::V1>();
   else if (vers == calc::v3)
      echgtrn_cu2<calc::V3>();
   else if (vers == calc::v4)
      echgtrn_cu2<calc::V4>();
   else if (vers == calc::v5)
      echgtrn_cu2<calc::V5>();
   else if (vers == calc::v6)
      echgtrn_cu2<calc::V6>();
}
}
