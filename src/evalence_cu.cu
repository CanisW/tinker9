#include "hip/hip_runtime.h"
#include "add.h"
#include "energy.h"
#include "glob.group.h"
#include "glob.molecule.h"
#include "image.h"
#include "launch.h"
#include "md.h"
#include "potent.h"
#include "seq_geom.h"
#include "seq_pitors.h"
#include "seq_torsion.h"
#include "seq_tortor.h"
#include "tool/gpu_card.h"
#include "tool/host_zero.h"


namespace tinker {
template <class Ver, bool rc_a>
__global__
void evalence_cu1(
   // etors
   energy_buffer restrict et, virial_buffer restrict vir_et,
   grad_prec* restrict detx, grad_prec* restrict dety, grad_prec* restrict detz,

   real torsunit, int ntors, const int (*restrict itors)[4],
   const real (*restrict tors1)[4], const real (*restrict tors2)[4],
   const real (*restrict tors3)[4], const real (*restrict tors4)[4],
   const real (*restrict tors5)[4], const real (*restrict tors6)[4],

   // epitors
   energy_buffer restrict ept, virial_buffer restrict vir_ept,
   grad_prec* restrict deptx, grad_prec* restrict depty,
   grad_prec* restrict deptz,

   real ptorunit, int npitors, const int (*restrict ipit)[6],
   const real* restrict kpit,

   // etortor
   energy_buffer restrict ett, virial_buffer restrict vir_ett,
   grad_prec* restrict dettx, grad_prec* restrict detty,
   grad_prec* restrict dettz,

   real ttorunit, int ntortor, const int (*restrict itt)[3],
   const int (*restrict ibitor)[5], const int* restrict chkttor_ia_,

   const int* restrict tnx, const int* restrict tny,
   const real (*restrict ttx)[ktrtor::maxtgrd],
   const real (*restrict tty)[ktrtor::maxtgrd],
   const real (*restrict tbf)[ktrtor::maxtgrd2],
   const real (*restrict tbx)[ktrtor::maxtgrd2],
   const real (*restrict tby)[ktrtor::maxtgrd2],
   const real (*restrict tbxy)[ktrtor::maxtgrd2],

   // egeom
   energy_buffer restrict eg, virial_buffer restrict vir_eg,
   grad_prec* restrict degx, grad_prec* restrict degy, grad_prec* restrict degz,

   int ngfix, const int (*restrict igfix)[2], const real (*restrict gfix)[3],

   // total
   energy_buffer restrict ebuf, virial_buffer restrict vbuf,

   // other
   const real* restrict x, const real* restrict y, const real* restrict z,
   const mass_prec* restrict mass, const int* restrict molec,
   const int (*restrict igrp)[2], const int* restrict kgrp,
   const mass_prec* restrict grpmass, TINKER_IMAGE_PARAMS)
{
   constexpr bool do_e = Ver::e;
   constexpr bool do_v = Ver::v;
   const int ithread = threadIdx.x + blockIdx.x * blockDim.x;
   const int stride = blockDim.x * gridDim.x;


   using ebuf_prec = energy_buffer_traits::type;
   ebuf_prec e0t;  // etors
   ebuf_prec e0pt; // epitors
   ebuf_prec e0tt; // etortor
   ebuf_prec e0g;  // egeom
   if CONSTEXPR (do_e) {
      e0t = 0;
      e0pt = 0;
      e0tt = 0;
      e0g = 0;
   }
   using vbuf_prec = virial_buffer_traits::type;
   vbuf_prec v0txx, v0tyx, v0tzx, v0tyy, v0tzy, v0tzz;       // etors
   vbuf_prec v0ptxx, v0ptyx, v0ptzx, v0ptyy, v0ptzy, v0ptzz; // epitors
   vbuf_prec v0ttxx, v0ttyx, v0ttzx, v0ttyy, v0ttzy, v0ttzz; // etors
   vbuf_prec v0gxx, v0gyx, v0gzx, v0gyy, v0gzy, v0gzz;       // egeom
   if CONSTEXPR (do_v) {
      v0txx = 0, v0tyx = 0, v0tzx = 0, v0tyy = 0, v0tzy = 0, v0tzz = 0;
      v0ptxx = 0, v0ptyx = 0, v0ptzx = 0, v0ptyy = 0, v0ptzy = 0, v0ptzz = 0;
      v0ttxx = 0, v0ttyx = 0, v0ttzx = 0, v0ttyy = 0, v0ttzy = 0, v0ttzz = 0;
      v0gxx = 0, v0gyx = 0, v0gzx = 0, v0gyy = 0, v0gzy = 0, v0gzz = 0;
   }


   // etors
   for (int i = ithread; i < ntors; i += stride) {
      real e, vxx, vyx, vzx, vyy, vzy, vzz;
      dk_tors<Ver>(e, vxx, vyx, vzx, vyy, vzy, vzz,

                   detx, dety, detz,

                   torsunit, i, itors,

                   tors1, tors2, tors3, tors4, tors5, tors6, x, y, z);
      if CONSTEXPR (do_e) {
         e0t += cvt_to<ebuf_prec>(e);
      }
      if CONSTEXPR (do_v) {
         v0txx += cvt_to<vbuf_prec>(vxx);
         v0tyx += cvt_to<vbuf_prec>(vyx);
         v0tzx += cvt_to<vbuf_prec>(vzx);
         v0tyy += cvt_to<vbuf_prec>(vyy);
         v0tzy += cvt_to<vbuf_prec>(vzy);
         v0tzz += cvt_to<vbuf_prec>(vzz);
      }
   }
   if CONSTEXPR (do_e and rc_a) {
      if (ntors > 0)
         atomic_add(e0t, et, ithread);
   }
   if CONSTEXPR (do_v and rc_a) {
      if (ntors > 0)
         atomic_add(v0txx, v0tyx, v0tzx, v0tyy, v0tzy, v0tzz, vir_et, ithread);
   }


   // epitors
   for (int i = ithread; i < npitors; i += stride) {
      real e, vxx, vyx, vzx, vyy, vzy, vzz;
      dk_pitors<Ver>(e, vxx, vyx, vzx, vyy, vzy, vzz,

                     deptx, depty, deptz,

                     ptorunit, i, ipit, kpit, x, y, z);
      if CONSTEXPR (do_e) {
         e0pt += cvt_to<ebuf_prec>(e);
      }
      if CONSTEXPR (do_v) {
         v0ptxx += cvt_to<vbuf_prec>(vxx);
         v0ptyx += cvt_to<vbuf_prec>(vyx);
         v0ptzx += cvt_to<vbuf_prec>(vzx);
         v0ptyy += cvt_to<vbuf_prec>(vyy);
         v0ptzy += cvt_to<vbuf_prec>(vzy);
         v0ptzz += cvt_to<vbuf_prec>(vzz);
      }
   }
   if CONSTEXPR (do_e and rc_a) {
      if (npitors > 0)
         atomic_add(e0pt, ept, ithread);
   }
   if CONSTEXPR (do_v and rc_a) {
      if (npitors > 0)
         atomic_add(v0ptxx, v0ptyx, v0ptzx, v0ptyy, v0ptzy, v0ptzz, vir_ept,
                    ithread);
   }


   // etortor
   for (int i = ithread; i < ntortor; i += stride) {
      real e, vxx, vyx, vzx, vyy, vzy, vzz;
      dk_tortor<Ver>(e, vxx, vyx, vzx, vyy, vzy, vzz,

                     dettx, detty, dettz,

                     ttorunit, i, itt, ibitor, chkttor_ia_,

                     tnx, tny, ttx, tty, tbf, tbx, tby, tbxy,

                     x, y, z);
      if CONSTEXPR (do_e) {
         e0tt += cvt_to<ebuf_prec>(e);
      }
      if CONSTEXPR (do_v) {
         v0ttxx += cvt_to<vbuf_prec>(vxx);
         v0ttyx += cvt_to<vbuf_prec>(vyx);
         v0ttzx += cvt_to<vbuf_prec>(vzx);
         v0ttyy += cvt_to<vbuf_prec>(vyy);
         v0ttzy += cvt_to<vbuf_prec>(vzy);
         v0ttzz += cvt_to<vbuf_prec>(vzz);
      }
   }
   if CONSTEXPR (do_e and rc_a) {
      if (ntortor > 0)
         atomic_add(e0tt, ett, ithread);
   }
   if CONSTEXPR (do_v and rc_a) {
      if (ntortor > 0)
         atomic_add(v0ttxx, v0ttyx, v0ttzx, v0ttyy, v0ttzy, v0ttzz, vir_ett,
                    ithread);
   }


   // egeom
   for (int i = ithread; i < ngfix; i += stride) {
      real e, vxx, vyx, vzx, vyy, vzy, vzz;
      dk_geom<Ver>(e, vxx, vyx, vzx, vyy, vzy, vzz,

                   degx, degy, degz,

                   i, igfix, gfix,

                   x, y, z, mass, molec, igrp, kgrp, grpmass,
                   TINKER_IMAGE_ARGS);
      if CONSTEXPR (do_e) {
         e0g += cvt_to<ebuf_prec>(e);
      }
      if CONSTEXPR (do_v) {
         v0gxx += cvt_to<vbuf_prec>(vxx);
         v0gyx += cvt_to<vbuf_prec>(vyx);
         v0gzx += cvt_to<vbuf_prec>(vzx);
         v0gyy += cvt_to<vbuf_prec>(vyy);
         v0gzy += cvt_to<vbuf_prec>(vzy);
         v0gzz += cvt_to<vbuf_prec>(vzz);
      }
   }
   if CONSTEXPR (do_e and rc_a) {
      if (ngfix > 0)
         atomic_add(e0g, eg, ithread);
   }
   if CONSTEXPR (do_v and rc_a) {
      if (ngfix > 0)
         atomic_add(v0gxx, v0gyx, v0gzx, v0gyy, v0gzy, v0gzz, vir_eg, ithread);
   }


   // total energy and virial
   if CONSTEXPR (do_e and not rc_a) {
      ebuf_prec etl = 0;
      etl += e0t;  // etors
      etl += e0pt; // epitors
      etl += e0g;  // egeom
      atomic_add(etl, ebuf, ithread);
   }
   if CONSTEXPR (do_v and not rc_a) {
      vbuf_prec vtlxx = 0, vtlyx = 0, vtlzx = 0;
      vbuf_prec vtlyy = 0, vtlzy = 0, vtlzz = 0;
      // etors
      vtlxx += v0txx, vtlyx += v0tyx, vtlzx += v0tzx;
      vtlyy += v0tyy, vtlzy += v0tzy, vtlzz += v0tzz;
      // epitors
      vtlxx += v0ptxx, vtlyx += v0ptyx, vtlzx += v0ptzx;
      vtlyy += v0ptyy, vtlzy += v0ptzy, vtlzz += v0ptzz;
      // egeom
      vtlxx += v0gxx, vtlyx += v0gyx, vtlzx += v0gzx;
      vtlyy += v0gyy, vtlzy += v0gzy, vtlzz += v0gzz;
      atomic_add(vtlxx, vtlyx, vtlzx, vtlyy, vtlzy, vtlzz, vbuf, ithread);
   }
}


void evalence_cu2(int vers, bool flag_tors, bool flag_pitors, bool flag_tortor,
                  bool flag_geom)
{
#define EVALENCE_ARGS                                                          \
   /* etors */ et, vir_et, detx, dety, detz, torsunit, flag_tors ? ntors : 0,  \
      itors, tors1, tors2, tors3, tors4, tors5, tors6, /* epitors */ ept,      \
      vir_ept, deptx, depty, deptz, ptorunit, flag_pitors ? npitors : 0, ipit, \
      kpit, /* etortor */ ett, vir_ett, dettx, detty, dettz, ttorunit,         \
      flag_tortor ? ntortor : 0, itt, ibitor, chkttor_ia_, tnx, tny, ttx, tty, \
      tbf, tbx, tby, tbxy, /* egeom */ eg, vir_eg, degx, degy, degz,           \
      flag_geom ? ngfix : 0, igfix, gfix, /* total */ eng_buf, vir_buf,        \
      /* other */ x, y, z, mass, molecule.molecule, grp.igrp, grp.kgrp,        \
      grp.grpmass, TINKER_IMAGE_ARGS


   int ngrid = get_grid_size(BLOCK_DIM);
   if (rc_flag & calc::analyz) {
      if (vers == calc::v0)
         evalence_cu1<calc::V0, true>
            <<<ngrid, BLOCK_DIM, 0, nonblk>>>(EVALENCE_ARGS);
      else if (vers == calc::v1)
         evalence_cu1<calc::V1, true>
            <<<ngrid, BLOCK_DIM, 0, nonblk>>>(EVALENCE_ARGS);
      else if (vers == calc::v3)
         evalence_cu1<calc::V3, true>
            <<<ngrid, BLOCK_DIM, 0, nonblk>>>(EVALENCE_ARGS);
      else if (vers == calc::v4)
         evalence_cu1<calc::V4, true>
            <<<ngrid, BLOCK_DIM, 0, nonblk>>>(EVALENCE_ARGS);
      else if (vers == calc::v5)
         evalence_cu1<calc::V5, true>
            <<<ngrid, BLOCK_DIM, 0, nonblk>>>(EVALENCE_ARGS);
      else if (vers == calc::v6)
         evalence_cu1<calc::V6, true>
            <<<ngrid, BLOCK_DIM, 0, nonblk>>>(EVALENCE_ARGS);
   } else {
      if (vers == calc::v0)
         evalence_cu1<calc::V0, false>
            <<<ngrid, BLOCK_DIM, 0, nonblk>>>(EVALENCE_ARGS);
      else if (vers == calc::v1)
         evalence_cu1<calc::V1, false>
            <<<ngrid, BLOCK_DIM, 0, nonblk>>>(EVALENCE_ARGS);
      else if (vers == calc::v3)
         assert(false);
      else if (vers == calc::v4)
         evalence_cu1<calc::V4, false>
            <<<ngrid, BLOCK_DIM, 0, nonblk>>>(EVALENCE_ARGS);
      else if (vers == calc::v5)
         evalence_cu1<calc::V5, false>
            <<<ngrid, BLOCK_DIM, 0, nonblk>>>(EVALENCE_ARGS);
      else if (vers == calc::v6)
         evalence_cu1<calc::V6, false>
            <<<ngrid, BLOCK_DIM, 0, nonblk>>>(EVALENCE_ARGS);
   }
}


void evalence_cu(int vers)
{
   bool rc_a = rc_flag & calc::analyz;
   bool do_e = vers & calc::energy;
   bool do_v = vers & calc::virial;
   bool do_g = vers & calc::grad;


   bool flag_tors = use_potent(torsion_term);
   bool flag_pitors = use_potent(pitors_term);
   bool flag_tortor = use_potent(tortor_term);
   bool flag_geom = use_potent(geom_term);


   size_t bsize = buffer_size();
   if (rc_a and flag_tors) {
      host_zero(energy_et, virial_et);
      if (do_e)
         darray::zero(PROCEED_NEW_Q, bsize, et);
      if (do_v)
         darray::zero(PROCEED_NEW_Q, bsize, vir_et);
      if (do_g)
         darray::zero(PROCEED_NEW_Q, n, detx, dety, detz);
   }
   if (rc_a and flag_pitors) {
      host_zero(energy_ept, virial_ept);
      if (do_e)
         darray::zero(PROCEED_NEW_Q, bsize, ept);
      if (do_v)
         darray::zero(PROCEED_NEW_Q, bsize, vir_ept);
      if (do_g)
         darray::zero(PROCEED_NEW_Q, n, deptx, depty, deptz);
   }
   if (rc_a and flag_tortor) {
      host_zero(energy_ett, virial_ett);
      if (do_e)
         darray::zero(PROCEED_NEW_Q, bsize, ett);
      if (do_v)
         darray::zero(PROCEED_NEW_Q, bsize, vir_ett);
      if (do_g)
         darray::zero(PROCEED_NEW_Q, n, dettx, detty, dettz);
   }
   if (rc_a and flag_geom) {
      host_zero(energy_eg, virial_eg);
      if (do_e)
         darray::zero(PROCEED_NEW_Q, bsize, eg);
      if (do_v)
         darray::zero(PROCEED_NEW_Q, bsize, vir_eg);
      if (do_g)
         darray::zero(PROCEED_NEW_Q, n, degx, degy, degz);
   }


   if (flag_tors or flag_pitors or flag_tortor or flag_geom) {
      evalence_cu2(vers, flag_tors, flag_pitors, flag_tortor, flag_geom);
   }


   if (rc_a and flag_tors) {
      if (do_e) {
         energy_et = energy_reduce(et);
         energy_valence += energy_et;
      }
      if (do_v) {
         virial_reduce(virial_et, vir_et);
         for (int iv = 0; iv < 9; ++iv)
            virial_valence[iv] += virial_et[iv];
      }
      if (do_g)
         sum_gradient(gx, gy, gz, detx, dety, detz);
   }
   if (rc_a and flag_pitors) {
      if (do_e) {
         energy_ept = energy_reduce(ept);
         energy_valence += energy_ept;
      }
      if (do_v) {
         virial_reduce(virial_ept, vir_ept);
         for (int iv = 0; iv < 9; ++iv)
            virial_valence[iv] += virial_ept[iv];
      }
      if (do_g)
         sum_gradient(gx, gy, gz, deptx, depty, deptz);
   }
   if (rc_a and flag_tortor) {
      if (do_e) {
         energy_ett = energy_reduce(ett);
         energy_valence += energy_ett;
      }
      if (do_v) {
         virial_reduce(virial_ett, vir_ett);
         for (int iv = 0; iv < 9; ++iv)
            virial_valence[iv] += virial_ett[iv];
      }
      if (do_g)
         sum_gradient(gx, gy, gz, dettx, detty, dettz);
   }
   if (rc_a and flag_geom) {
      if (do_e) {
         energy_eg = energy_reduce(eg);
         energy_valence += energy_eg;
      }
      if (do_v) {
         virial_reduce(virial_eg, vir_eg);
         for (int iv = 0; iv < 9; ++iv)
            virial_valence[iv] += virial_eg[iv];
      }
      if (do_g)
         sum_gradient(gx, gy, gz, degx, degy, degz);
   }
}
}
