#include "hip/hip_runtime.h"
#include "add.h"
#include "empole_chgpen.h"
#include "epolar.h"
#include "epolar_chgpen.h"
#include "glob.spatial.h"
#include "image.h"
#include "induce_donly.h"
#include "launch.h"
#include "mdpq.h"
#include "seq_damp_chgpen.h"
#include "switch.h"


namespace tinker {
__global__
void sparse_precond_cu3(const real (*restrict rsd)[3], real (*restrict zrsd)[3],
                        const real* restrict polarity, int n, real udiag)
{
   for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < n;
        i += blockDim.x * gridDim.x) {
      real poli = udiag * polarity[i];
      #pragma unroll
      for (int j = 0; j < 3; ++j) {
         zrsd[i][j] = poli * rsd[i][j];
         real test = zrsd[i][j];
         //printf("udiag zrsd %14.6e\n", test);
      }
   }
}


__launch_bounds__(BLOCK_DIM) __global__
void sparse_precond_cu4(const real (*restrict rsd)[3], real (*restrict zrsd)[3],
                        real *restrict palpha,
                        const real* restrict polarity, TINKER_IMAGE_PARAMS,
                        real cutbuf2, int n,
                        const Spatial::SortedAtom* restrict sorted, int niak,
                        const int* restrict iak, const int* restrict lst)
{
   const int iwarp = (threadIdx.x + blockIdx.x * blockDim.x) / WARP_SIZE;
   const int nwarp = blockDim.x * gridDim.x / WARP_SIZE;
   const int ilane = threadIdx.x & (WARP_SIZE - 1);


   struct Data
   {
      real3 fkd;
      real3 rk, ukd;
      real  alpha, polk;
   };
   __shared__ Data data[BLOCK_DIM];


   for (int iw = iwarp; iw < niak; iw += nwarp) {
      real3 fid = make_real3(0, 0, 0);
      int atomi = min(iak[iw] * WARP_SIZE + ilane, n - 1);
      real3 ri = make_real3(sorted[atomi].x, sorted[atomi].y, sorted[atomi].z);
      int i = sorted[atomi].unsorted;
      real3 uid = make_real3(rsd[i][0], rsd[i][1], rsd[i][2]);
      real3 zd = make_real3(zrsd[i][0], zrsd[i][1], zrsd[i][2]);
      real alphai = palpha[i];
      real poli = polarity[i];


      data[threadIdx.x].fkd = make_real3(0, 0, 0);
      int shatomk = lst[iw * WARP_SIZE + ilane];
      data[threadIdx.x].rk =
         make_real3(sorted[shatomk].x, sorted[shatomk].y, sorted[shatomk].z);
      int shk = sorted[shatomk].unsorted;
      data[threadIdx.x].ukd = make_real3(rsd[shk][0], rsd[shk][1], rsd[shk][2]);
      data[threadIdx.x].alpha = palpha[shk];
      data[threadIdx.x].polk = polarity[shk];


      for (int j = 0; j < WARP_SIZE; ++j) {
         int srclane = (ilane + j) & (WARP_SIZE - 1);
         int klane = srclane + threadIdx.x - ilane;
         int atomk = __shfl_sync(ALL_LANES, shatomk, srclane);
         real alphak = data[klane].alpha;
         real3 dr = data[klane].rk - ri;


         real r2 = image2(dr.x, dr.y, dr.z);
         if (atomi < atomk && r2 <= cutbuf2) {
            real r = REAL_SQRT(r2);
            real dmpik[3];
            damp_mut(dmpik,r,alphai,alphak);

            real polik = poli * data[klane].polk;
            real rr3 = dmpik[1] * polik * REAL_RECIP(r * r2);
            real rr5 = 3 * dmpik[2] * polik * REAL_RECIP(r * r2 * r2);


            real c;
            c = rr5 * dot3(dr, data[klane].ukd);
            real3 inci = c * dr - rr3 * data[klane].ukd;
            fid += c * dr - rr3 * data[klane].ukd;

            c = rr5 * dot3(dr, uid);
            data[klane].fkd += c * dr - rr3 * uid;

            //printf("1 %16.8e %16.8e %16.8e\n", fid.x, fid.y, fid.z);
            // printf("2 %16.8e %16.8e %16.8e\n", zd.x, zd.y, zd.z);

         } // end if (include)
      }

      // printf("%16.8e %16.8e %16.8e\n", aa, ab, ac);

      atomic_add(fid.x, &zrsd[i][0]);
      atomic_add(fid.y, &zrsd[i][1]);
      atomic_add(fid.z, &zrsd[i][2]);
      atomic_add(data[threadIdx.x].fkd.x, &zrsd[shk][0]);
      atomic_add(data[threadIdx.x].fkd.y, &zrsd[shk][1]);
      atomic_add(data[threadIdx.x].fkd.z, &zrsd[shk][2]);
   } // end for (iw)
}


__global__
void sparse_precond_cu5(const real (*restrict rsd)[3], real (*restrict zrsd)[3],
                        real *restrict palpha,
                        const real* restrict polarity, TINKER_IMAGE_PARAMS,
                        real cutbuf2, const real* restrict x,
                        const real* restrict y, const real* restrict z,
                        int nwexclude, const int (*restrict wexclude)[2],
                        const real* restrict wexclude_scale)
{
   for (int ii = threadIdx.x + blockIdx.x * blockDim.x; ii < nwexclude;
        ii += blockDim.x * gridDim.x) {
      int i = wexclude[ii][0];
      int k = wexclude[ii][1];
      real wscale = wexclude_scale[ii];


      real xi = x[i];
      real yi = y[i];
      real zi = z[i];
      real alphai = palpha[i];
      real poli = polarity[i];

      real alphak = palpha[k];
      real xr = x[k] - xi;
      real yr = y[k] - yi;
      real zr = z[k] - zi;
      real r2 = image2(xr, yr, zr);
      if (r2 <= cutbuf2) {
         real r = REAL_SQRT(r2);
         real dmpik[3];
         damp_mut(dmpik,r,alphai,alphak);
         real scale3 = wscale * dmpik[1];
         real scale5 = wscale * dmpik[2];

         real polik = poli * polarity[k];
         real rr3 = scale3 * polik * REAL_RECIP(r * r2);
         real rr5 = 3 * scale5 * polik * REAL_RECIP(r * r2 * r2);


         real c;
         real3 dr = make_real3(xr, yr, zr);
         real3 uid = make_real3(rsd[i][0], rsd[i][1], rsd[i][2]);
         real3 ukd = make_real3(rsd[k][0], rsd[k][1], rsd[k][2]);


         c = rr5 * dot3(dr, ukd);
         real3 fid = c * dr - rr3 * ukd;
         c = rr5 * dot3(dr, uid);
         real3 fkd = c * dr - rr3 * uid;

         atomic_add(fid.x, &zrsd[i][0]);
         atomic_add(fid.y, &zrsd[i][1]);
         atomic_add(fid.z, &zrsd[i][2]);
         atomic_add(fkd.x, &zrsd[k][0]);
         atomic_add(fkd.y, &zrsd[k][1]);
         atomic_add(fkd.z, &zrsd[k][2]);

      }
   }
}


void sparse_precond_apply_cu2(const real (*rsd)[3], real (*zrsd)[3])
{
   const auto& st = *uspatial_unit;
   const real off = switch_off(switch_usolve);
   const real cutbuf2 = (off + st.buffer) * (off + st.buffer);


   launch_k1s(nonblk, n, sparse_precond_cu3, //
              rsd, zrsd, polarity, n, udiag);
   if (st.niak > 0)
      launch_k1s(nonblk, WARP_SIZE * st.niak, sparse_precond_cu4, //
                 rsd, zrsd, palpha, polarity, TINKER_IMAGE_ARGS,
                 cutbuf2, //
                 n, st.sorted, st.niak, st.iak, st.lst);

   if (nwexclude > 0)
      launch_k1s(nonblk, nwexclude, sparse_precond_cu5, //
                 rsd, zrsd, palpha, polarity, TINKER_IMAGE_ARGS,
                 cutbuf2, //
                 x, y, z, nwexclude, wexclude, wexclude_scale);
}
}
