#include "hip/hip_runtime.h"
#include "add.h"
#include "evdw.h"
#include "image.h"
#include "launch.h"
#include "md.h"
#include "named_struct.h"
#include "seq_pair_lj.h"
#include "seq_switch.h"
#include "spatial.h"
#include "switch.h"


TINKER_NAMESPACE_BEGIN
#define LJ_ARGS                                                                \
   size_t bufsize, count_buffer restrict nev, energy_buffer restrict ev,       \
      virial_buffer restrict vir_ev, grad_prec *restrict gx,                   \
      grad_prec *restrict gy, grad_prec *restrict gz, TINKER_IMAGE_PARAMS,     \
      int njvdw, const int *restrict jvdw, const real *restrict radmin,        \
      const real *restrict epsilon, real cut, real off


template <class Ver>
__global__
void elj_cu1(LJ_ARGS, int n, const Spatial::SortedAtom* restrict sorted,
             int niak, const int* restrict iak, const int* restrict lst)
{
   constexpr bool do_e = Ver::e;
   constexpr bool do_a = Ver::a;
   constexpr bool do_g = Ver::g;
   constexpr bool do_v = Ver::v;


   const int ithread = threadIdx.x + blockIdx.x * blockDim.x;
   const int iwarp = ithread / WARP_SIZE;
   const int nwarp = blockDim.x * gridDim.x / WARP_SIZE;
   const int ilane = threadIdx.x & (WARP_SIZE - 1);
   const int offset = ithread & (bufsize - 1);


   // thread local variables
   using ebuf_prec = energy_buffer_traits::type;
   using vbuf_prec = virial_buffer_traits::type;
   MAYBE_UNUSED int ctl;
   MAYBE_UNUSED ebuf_prec etl;
   MAYBE_UNUSED grad_prec gxi, gyi, gzi, gxk, gyk, gzk;
   MAYBE_UNUSED vbuf_prec vtlxx, vtlyx, vtlzx, vtlyy, vtlzy, vtlzz;


   const real cut2 = cut * cut;
   const real off2 = off * off;
   for (int iw = iwarp; iw < niak; iw += nwarp) {
      if CONSTEXPR (do_a)
         ctl = 0;
      if CONSTEXPR (do_e)
         etl = 0;
      if CONSTEXPR (do_g) {
         gxi = 0;
         gyi = 0;
         gzi = 0;
         gxk = 0;
         gyk = 0;
         gzk = 0;
      }
      if CONSTEXPR (do_v) {
         vtlxx = 0;
         vtlyx = 0;
         vtlzx = 0;
         vtlyy = 0;
         vtlzy = 0;
         vtlzz = 0;
      }


      int atomi = min(iak[iw] * WARP_SIZE + ilane, n - 1);
      real xi = sorted[atomi].x;
      real yi = sorted[atomi].y;
      real zi = sorted[atomi].z;
      int i = sorted[atomi].unsorted;
      int it = jvdw[i];


      int shatomk = lst[iw * WARP_SIZE + ilane];
      real shx = sorted[shatomk].x;
      real shy = sorted[shatomk].y;
      real shz = sorted[shatomk].z;
      int shk = sorted[shatomk].unsorted;
      int shkt = jvdw[shk];


      for (int j = 0; j < WARP_SIZE; ++j) {
         int srclane = (ilane + j) & (WARP_SIZE - 1);
         int atomk = __shfl_sync(ALL_LANES, shatomk, srclane);
         real xr = xi - __shfl_sync(ALL_LANES, shx, srclane);
         real yr = yi - __shfl_sync(ALL_LANES, shy, srclane);
         real zr = zi - __shfl_sync(ALL_LANES, shz, srclane);
         int kt = __shfl_sync(ALL_LANES, shkt, srclane);


         MAYBE_UNUSED real dedx = 0, dedy = 0, dedz = 0;


         real rik2 = image2(xr, yr, zr);
         if (atomi < atomk && rik2 <= off2) {
            real rik = REAL_SQRT(rik2);
            real rv = radmin[it * njvdw + kt];
            real eps = epsilon[it * njvdw + kt];


            MAYBE_UNUSED real e, de;
            pair_lj<do_g>(rik, rik2, rv, eps, 1, e, de);


            if (rik2 > cut2) {
               real taper, dtaper;
               switch_taper5<do_g>(rik, cut, off, taper, dtaper);
               if CONSTEXPR (do_g)
                  de = e * dtaper + de * taper;
               if CONSTEXPR (do_e)
                  e = e * taper;
            }


            if CONSTEXPR (do_a)
               if (e != 0)
                  ctl += 1;
            if CONSTEXPR (do_e)
               etl += to_cu<ebuf_prec>(e);
            if CONSTEXPR (do_g) {
               de *= REAL_RECIP(rik);
               dedx = de * xr;
               dedy = de * yr;
               dedz = de * zr;
               if CONSTEXPR (do_v) {
                  vtlxx += to_cu<vbuf_prec>(xr * dedx);
                  vtlyx += to_cu<vbuf_prec>(yr * dedx);
                  vtlzx += to_cu<vbuf_prec>(zr * dedx);
                  vtlyy += to_cu<vbuf_prec>(yr * dedy);
                  vtlzy += to_cu<vbuf_prec>(zr * dedy);
                  vtlzz += to_cu<vbuf_prec>(zr * dedz);
               }
            }
         } // end if (include)


         if CONSTEXPR (do_g) {
            int dstlane = (ilane + WARP_SIZE - j) & (WARP_SIZE - 1);
            gxi += to_cu<grad_prec>(dedx);
            gyi += to_cu<grad_prec>(dedy);
            gzi += to_cu<grad_prec>(dedz);
            gxk -= to_cu<grad_prec>(__shfl_sync(ALL_LANES, dedx, dstlane));
            gyk -= to_cu<grad_prec>(__shfl_sync(ALL_LANES, dedy, dstlane));
            gzk -= to_cu<grad_prec>(__shfl_sync(ALL_LANES, dedz, dstlane));
         }
         __syncwarp();
      }


      if CONSTEXPR (do_a)
         atomic_add(ctl, nev, offset);
      if CONSTEXPR (do_e)
         atomic_add(etl, ev, offset);
      if CONSTEXPR (do_g) {
         atomic_add(gxi, gx, i);
         atomic_add(gyi, gy, i);
         atomic_add(gzi, gz, i);
         atomic_add(gxk, gx, shk);
         atomic_add(gyk, gy, shk);
         atomic_add(gzk, gz, shk);
      }
      if CONSTEXPR (do_v)
         atomic_add(vtlxx, vtlyx, vtlzx, vtlyy, vtlzy, vtlzz, vir_ev, offset);
   } // end for (iw)
}


template <class Ver>
__global__
void elj_cu2(LJ_ARGS, const real* restrict x, const real* restrict y,
             const real* restrict z, int nvexclude,
             const int (*restrict vexclude)[2],
             const real* restrict vexclude_scale)
{
   constexpr bool do_e = Ver::e;
   constexpr bool do_a = Ver::a;
   constexpr bool do_g = Ver::g;
   constexpr bool do_v = Ver::v;


   const real cut2 = cut * cut;
   const real off2 = off * off;
   for (int ii = threadIdx.x + blockIdx.x * blockDim.x; ii < nvexclude;
        ii += blockDim.x * gridDim.x) {
      int offset = ii & (bufsize - 1);


      int i = vexclude[ii][0];
      int k = vexclude[ii][1];
      real vscale = vexclude_scale[ii];


      int it = jvdw[i];
      real xi = x[i];
      real yi = y[i];
      real zi = z[i];


      int kt = jvdw[k];
      real xr = xi - x[k];
      real yr = yi - y[k];
      real zr = zi - z[k];


      real rik2 = image2(xr, yr, zr);
      if (rik2 <= off2) {
         real rik = REAL_SQRT(rik2);
         real rv = radmin[it * njvdw + kt];
         real eps = epsilon[it * njvdw + kt];


         MAYBE_UNUSED real e, de;
         pair_lj<do_g>(rik, rik2, rv, eps, vscale, e, de);
         if (rik2 > cut2) {
            real taper, dtaper;
            switch_taper5<do_g>(rik, cut, off, taper, dtaper);
            if CONSTEXPR (do_g)
               de = e * dtaper + de * taper;
            if CONSTEXPR (do_e)
               e = e * taper;
         }


         if CONSTEXPR (do_a)
            if (vscale == -1 && e != 0)
               atomic_add(-1, nev, offset);
         if CONSTEXPR (do_e)
            atomic_add(e, ev, offset);
         if CONSTEXPR (do_g) {
            de *= REAL_RECIP(rik);
            real dedx = de * xr;
            real dedy = de * yr;
            real dedz = de * zr;
            atomic_add(dedx, gx, i);
            atomic_add(dedy, gy, i);
            atomic_add(dedz, gz, i);
            atomic_add(-dedx, gx, k);
            atomic_add(-dedy, gy, k);
            atomic_add(-dedz, gz, k);
            if CONSTEXPR (do_v) {
               real vxx = xr * dedx;
               real vyx = yr * dedx;
               real vzx = zr * dedx;
               real vyy = yr * dedy;
               real vzy = zr * dedy;
               real vzz = zr * dedz;
               atomic_add(vxx, vyx, vzx, vyy, vzy, vzz, vir_ev, offset);
            }
         }
      } // end if (include)
      __syncwarp();
   }
}


template <class Ver>
__global__
void elj_cu3(LJ_ARGS, const real* restrict x, const real* restrict y,
             const real* restrict z, real v4scale, int nvdw14,
             const int (*restrict vdw14ik)[2], const real* restrict radmin4,
             const real* restrict epsilon4)
{
   constexpr bool do_e = Ver::e;
   constexpr bool do_g = Ver::g;
   constexpr bool do_v = Ver::v;


   const real cut2 = cut * cut;
   const real off2 = off * off;
   for (int ii = threadIdx.x + blockIdx.x * blockDim.x; ii < nvdw14;
        ii += blockDim.x * gridDim.x) {
      int offset = ii & (bufsize - 1);


      int i = vdw14ik[ii][0];
      int k = vdw14ik[ii][1];


      int it = jvdw[i];
      real xi = x[i];
      real yi = y[i];
      real zi = z[i];


      int kt = jvdw[k];
      real xr = xi - x[k];
      real yr = yi - y[k];
      real zr = zi - z[k];


      real rik2 = image2(xr, yr, zr);
      if (rik2 <= off2) {
         real rik = REAL_SQRT(rik2);
         real rv = radmin[it * njvdw + kt];
         real eps = epsilon[it * njvdw + kt];
         real rv4 = radmin4[it * njvdw + kt];
         real eps4 = epsilon4[it * njvdw + kt];


         MAYBE_UNUSED real e, de, e4, de4;
         pair_lj<do_g>(rik, rik2, rv, eps, v4scale, e, de);
         pair_lj<do_g>(rik, rik2, rv4, eps4, v4scale, e4, de4);
         e = e4 - e;
         if CONSTEXPR (do_g)
            de = de4 - de;


         if (rik2 > cut2) {
            real taper, dtaper;
            switch_taper5<do_g>(rik, cut, off, taper, dtaper);
            if CONSTEXPR (do_g)
               de = e * dtaper + de * taper;
            if CONSTEXPR (do_e)
               e = e * taper;
         }


         // if CONSTEXPR (do_a) {}
         if CONSTEXPR (do_e)
            atomic_add(e, ev, offset);
         if CONSTEXPR (do_g) {
            de *= REAL_RECIP(rik);
            real dedx = de * xr;
            real dedy = de * yr;
            real dedz = de * zr;
            atomic_add(dedx, gx, i);
            atomic_add(dedy, gy, i);
            atomic_add(dedz, gz, i);
            atomic_add(-dedx, gx, k);
            atomic_add(-dedy, gy, k);
            atomic_add(-dedz, gz, k);
            if CONSTEXPR (do_v) {
               real vxx = xr * dedx;
               real vyx = yr * dedx;
               real vzx = zr * dedx;
               real vyy = yr * dedy;
               real vzy = zr * dedy;
               real vzz = zr * dedz;
               atomic_add(vxx, vyx, vzx, vyy, vzy, vzz, vir_ev, offset);
            }
         }
      } // end if (include)
      __syncwarp();
   }
}


template <class Ver>
void elj_cu4()
{
   const auto& st = *cspatial_unit;
   const real cut = switch_cut(switch_vdw);
   const real off = switch_off(switch_vdw);


   auto bufsize = buffer_size();


   if (st.niak > 0)
      launch_k1s(nonblk, WARP_SIZE * st.niak, elj_cu1<Ver>, bufsize, nev, ev,
                 vir_ev, gx, gy, gz, TINKER_IMAGE_ARGS, njvdw, jvdw, radmin,
                 epsilon, cut, off, n, st.sorted, st.niak, st.iak, st.lst);


   if (nvexclude > 0)
      launch_k1s(nonblk, nvexclude, elj_cu2<Ver>, bufsize, nev, ev, vir_ev, gx,
                 gy, gz, TINKER_IMAGE_ARGS, njvdw, jvdw, radmin, epsilon, cut,
                 off, x, y, z, nvexclude, vexclude, vexclude_scale);


   if (nvdw14 > 0)
      launch_k1s(nonblk, nvdw14, elj_cu3<Ver>, bufsize, nev, ev, vir_ev, gx, gy,
                 gz, TINKER_IMAGE_ARGS, njvdw, jvdw, radmin, epsilon, cut, off,
                 x, y, z, v4scale, nvdw14, vdw14ik, radmin4, epsilon4);
}


void elj_cu(int vers)
{
   if (vers == calc::v0)
      elj_cu4<calc::V0>();
   else if (vers == calc::v1)
      elj_cu4<calc::V1>();
   else if (vers == calc::v3)
      elj_cu4<calc::V3>();
   else if (vers == calc::v4)
      elj_cu4<calc::V4>();
   else if (vers == calc::v5)
      elj_cu4<calc::V5>();
   else if (vers == calc::v6)
      elj_cu4<calc::V6>();
}
TINKER_NAMESPACE_END
