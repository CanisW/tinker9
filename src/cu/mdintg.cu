#include "hip/hip_runtime.h"
#include "md/misc.h"
#include "md/pq.h"
#include "seq/launch.h"
#include "seq/reduce.h"
#include "tool/externfunc.h"
#include <tinker/detail/bound.hh>
#include <tinker/detail/inform.hh>
#include <tinker/detail/mdstuf.hh>
#include <tinker/detail/molcul.hh>
#include <tinker/detail/units.hh>

namespace tinker {
template <unsigned int B>
__global__
void mdrestSumP_cu(int n, vel_prec* restrict odata, const double* restrict mass,
   const vel_prec* restrict vx, const vel_prec* restrict vy, const vel_prec* restrict vz)
{
   static_assert(B == 64, "");
   const int ithread = threadIdx.x + blockIdx.x * blockDim.x;
   const int stride = blockDim.x * gridDim.x;
   const int t = threadIdx.x;

   vel_prec x = 0, y = 0, z = 0;
   for (int i = ithread; i < n; i += stride) {
      auto m = mass[i];
      x += m * vx[i];
      y += m * vy[i];
      z += m * vz[i];
   }

   __shared__ vel_prec tx[B], ty[B], tz[B];
   // clang-format off
   tx[t] = x; ty[t] = y; tz[t] = z;                                          __syncthreads();
   if (t < 32) { tx[t] += tx[t+32]; ty[t] += ty[t+32]; tz[t] += tz[t+32]; }  __syncthreads();
   if (t < 16) { tx[t] += tx[t+16]; ty[t] += ty[t+16]; tz[t] += tz[t+16]; }  __syncthreads();
   if (t <  8) { tx[t] += tx[t+ 8]; ty[t] += ty[t+ 8]; tz[t] += tz[t+ 8]; }  __syncthreads();
   if (t <  4) { tx[t] += tx[t+ 4]; ty[t] += ty[t+ 4]; tz[t] += tz[t+ 4]; }  __syncthreads();
   if (t <  2) { tx[t] += tx[t+ 2]; ty[t] += ty[t+ 2]; tz[t] += tz[t+ 2]; }  __syncthreads();
   // clang-format on
   if (t == 0) {
      const int b = blockIdx.x;
      odata[3 * b + 0] = tx[t] + tx[t + 1];
      odata[3 * b + 1] = ty[t] + ty[t + 1];
      odata[3 * b + 2] = tz[t] + tz[t + 1];
   }
}

template <int B>
__global__
void mdrestRemoveP_cu(int n, double invtotmass, const vel_prec* restrict idata,
   vel_prec* restrict vx, vel_prec* restrict vy, vel_prec* restrict vz, vel_prec* restrict xout)
{
   static_assert(B == 64, "");
   const int ithread = threadIdx.x + blockIdx.x * blockDim.x;
   const int stride = blockDim.x * gridDim.x;
   const int t = threadIdx.x;

   vel_prec x = 0, y = 0, z = 0;
   for (int i = t; i < gridDim.x; i += B) {
      x += idata[3 * i + 0];
      y += idata[3 * i + 1];
      z += idata[3 * i + 2];
   }

   __shared__ vel_prec tx[B], ty[B], tz[B];
   // clang-format off
   tx[t] = x; ty[t] = y; tz[t] = z;                                          __syncthreads();
   if (t < 32) { tx[t] += tx[t+32]; ty[t] += ty[t+32]; tz[t] += tz[t+32]; }  __syncthreads();
   if (t < 16) { tx[t] += tx[t+16]; ty[t] += ty[t+16]; tz[t] += tz[t+16]; }  __syncthreads();
   if (t <  8) { tx[t] += tx[t+ 8]; ty[t] += ty[t+ 8]; tz[t] += tz[t+ 8]; }  __syncthreads();
   if (t <  4) { tx[t] += tx[t+ 4]; ty[t] += ty[t+ 4]; tz[t] += tz[t+ 4]; }  __syncthreads();
   if (t <  2) { tx[t] += tx[t+ 2]; ty[t] += ty[t+ 2]; tz[t] += tz[t+ 2]; }  __syncthreads();
   // clang-format on
   x = (tx[0] + tx[1]) * invtotmass;
   y = (ty[0] + ty[1]) * invtotmass;
   z = (tz[0] + tz[1]) * invtotmass;
   xout[0] = x;
   xout[1] = y;
   xout[2] = z;
   for (int i = ithread; i < n; i += stride) {
      vx[i] -= x;
      vy[i] -= y;
      vz[i] -= z;
   }
}

void mdrestRemovePbcMomentum_cu(bool copyout, vel_prec& vtot1, vel_prec& vtot2, vel_prec& vtot3)
{
   vel_prec* xout;
   xout = (vel_prec*)dptr_buf;
   auto invtotmass = 1 / molcul::totmass;

   constexpr int HN = 3;
   constexpr int B = 64;
   vel_prec* ptr = &xout[4];
   int grid_siz1 = -4 + gpuGridSize(BLOCK_DIM);
   grid_siz1 /= HN;
   int grid_siz2 = (n + B - 1) / B;
   int ngrid = std::min(grid_siz1, grid_siz2);

   mdrestSumP_cu<B><<<ngrid, B, 0, g::s0>>>(n, ptr, mass, vx, vy, vz);
   mdrestRemoveP_cu<B><<<ngrid, B, 0, g::s0>>>(n, invtotmass, ptr, vx, vy, vz, xout);

   if (copyout) {
      vel_prec v[3];
      darray::copyout(g::q0, 3, v, xout);
      waitFor(g::q0);
      vtot1 = v[0];
      vtot2 = v[1];
      vtot3 = v[2];
   }
}

void mdrest_cu(int istep)
{
   if (not mdstuf::dorest)
      return;
   if ((istep % mdstuf::irest) != 0)
      return;

   // const energy_prec ekcal = units::ekcal;

   // zero out the total mass and overall linear velocity

   auto totmass = molcul::totmass;
   vel_prec vtot1 = 0, vtot2 = 0, vtot3 = 0;

   bool copyout = inform::debug or not bound::use_bounds;
   mdrestRemovePbcMomentum_cu(copyout, vtot1, vtot2, vtot3);

   // print the translational velocity of the overall system

   mdrestPrintP1(inform::debug, vtot1, vtot2, vtot3, totmass);

   if (not bound::use_bounds) {
      throwExceptionMissingFunction("mdrestRemoveAngularMomentum_cu", __FILE__, __LINE__);
   }
}
}
