#include "hip/hip_runtime.h"
#include "ff/amoebamod.h"
#include "ff/elec.h"
#include "ff/hippo/expol.h"
#include "ff/hippomod.h"
#include "ff/image.h"
#include "ff/spatial.h"
#include "ff/switch.h"
#include "seq/add.h"
#include "seq/launch.h"
#include "seq/pair_alterpol.h"
#include "seq/triangle.h"

namespace tinker {
// ck.py Version 2.0.3

__global__
static void alterpol_cu1(int n, TINKER_IMAGE_PARAMS, real cut, real off,
   const unsigned* restrict dinfo, int nexclude, const int (*restrict exclude)[2],
   const real (*restrict exclude_scale)[3], const real* restrict x, const real* restrict y,
   const real* restrict z, const Spatial::SortedAtom* restrict sorted, int nakpl,
   const int* restrict iakpl, int niak, const int* restrict iak, const int* restrict lst,
   real (*restrict polscale)[9], const real* restrict kpep, const real* restrict prepep,
   const real* restrict dmppep, const int* restrict lpep, ExpolScr scrtyp)
{
   const int ithread = threadIdx.x + blockIdx.x * blockDim.x;
   const int iwarp = ithread / WARP_SIZE;
   const int nwarp = blockDim.x * gridDim.x / WARP_SIZE;
   const int ilane = threadIdx.x & (WARP_SIZE - 1);

   __shared__ real xi[BLOCK_DIM];
   __shared__ real yi[BLOCK_DIM];
   __shared__ real zi[BLOCK_DIM];
   real xk;
   real yk;
   real zk;
   __shared__ real psci00[BLOCK_DIM];
   __shared__ real psci01[BLOCK_DIM];
   __shared__ real psci02[BLOCK_DIM];
   __shared__ real psci10[BLOCK_DIM];
   __shared__ real psci11[BLOCK_DIM];
   __shared__ real psci12[BLOCK_DIM];
   __shared__ real psci20[BLOCK_DIM];
   __shared__ real psci21[BLOCK_DIM];
   __shared__ real psci22[BLOCK_DIM];
   real psck00;
   real psck01;
   real psck02;
   real psck10;
   real psck11;
   real psck12;
   real psck20;
   real psck21;
   real psck22;
   __shared__ real springi[BLOCK_DIM];
   __shared__ real sizi[BLOCK_DIM];
   __shared__ real alphai[BLOCK_DIM];
   __shared__ int epli[BLOCK_DIM];
   real springk;
   real sizk;
   real alphak;
   int eplk;

   //* /
   for (int ii = ithread; ii < nexclude; ii += blockDim.x * gridDim.x) {
      const int klane = threadIdx.x;
      psci00[threadIdx.x] = 0;
      psci01[threadIdx.x] = 0;
      psci02[threadIdx.x] = 0;
      psci10[threadIdx.x] = 0;
      psci11[threadIdx.x] = 0;
      psci12[threadIdx.x] = 0;
      psci20[threadIdx.x] = 0;
      psci21[threadIdx.x] = 0;
      psci22[threadIdx.x] = 0;
      psck00 = 0;
      psck01 = 0;
      psck02 = 0;
      psck10 = 0;
      psck11 = 0;
      psck12 = 0;
      psck20 = 0;
      psck21 = 0;
      psck22 = 0;

      int i = exclude[ii][0];
      int k = exclude[ii][1];
      real scaleb = exclude_scale[ii][1];

      xi[klane] = x[i];
      yi[klane] = y[i];
      zi[klane] = z[i];
      xk = x[k];
      yk = y[k];
      zk = z[k];
      springi[klane] = kpep[i];
      sizi[klane] = prepep[i];
      alphai[klane] = dmppep[i];
      epli[klane] = lpep[i];
      springk = kpep[k];
      sizk = prepep[k];
      alphak = dmppep[k];
      eplk = lpep[k];

      constexpr bool incl = true;
      real xr = xk - xi[klane];
      real yr = yk - yi[klane];
      real zr = zk - zi[klane];
      real r2 = image2(xr, yr, zr);
      if ((eplk or epli[klane]) and r2 <= off * off and incl) {
         real r = REAL_SQRT(r2);
         real ks2i[3][3], ks2k[3][3];
         pair_alterpol(scrtyp, r, scaleb, cut, off, xr, yr, zr, springi[klane], sizi[klane],
            alphai[klane], springk, sizk, alphak, ks2i, ks2k);
         psci00[klane] += ks2i[0][0];
         psci01[klane] += ks2i[0][1];
         psci02[klane] += ks2i[0][2];
         psci10[klane] += ks2i[1][0];
         psci11[klane] += ks2i[1][1];
         psci12[klane] += ks2i[1][2];
         psci20[klane] += ks2i[2][0];
         psci21[klane] += ks2i[2][1];
         psci22[klane] += ks2i[2][2];
         psck00 += ks2k[0][0];
         psck01 += ks2k[0][1];
         psck02 += ks2k[0][2];
         psck10 += ks2k[1][0];
         psck11 += ks2k[1][1];
         psck12 += ks2k[1][2];
         psck20 += ks2k[2][0];
         psck21 += ks2k[2][1];
         psck22 += ks2k[2][2];
      }

      atomic_add(psci00[threadIdx.x], &polscale[i][0]);
      atomic_add(psci01[threadIdx.x], &polscale[i][1]);
      atomic_add(psci02[threadIdx.x], &polscale[i][2]);
      atomic_add(psci10[threadIdx.x], &polscale[i][3]);
      atomic_add(psci11[threadIdx.x], &polscale[i][4]);
      atomic_add(psci12[threadIdx.x], &polscale[i][5]);
      atomic_add(psci20[threadIdx.x], &polscale[i][6]);
      atomic_add(psci21[threadIdx.x], &polscale[i][7]);
      atomic_add(psci22[threadIdx.x], &polscale[i][8]);
      atomic_add(psck00, &polscale[k][0]);
      atomic_add(psck01, &polscale[k][1]);
      atomic_add(psck02, &polscale[k][2]);
      atomic_add(psck10, &polscale[k][3]);
      atomic_add(psck11, &polscale[k][4]);
      atomic_add(psck12, &polscale[k][5]);
      atomic_add(psck20, &polscale[k][6]);
      atomic_add(psck21, &polscale[k][7]);
      atomic_add(psck22, &polscale[k][8]);
   }
   // */

   for (int iw = iwarp; iw < nakpl; iw += nwarp) {
      psci00[threadIdx.x] = 0;
      psci01[threadIdx.x] = 0;
      psci02[threadIdx.x] = 0;
      psci10[threadIdx.x] = 0;
      psci11[threadIdx.x] = 0;
      psci12[threadIdx.x] = 0;
      psci20[threadIdx.x] = 0;
      psci21[threadIdx.x] = 0;
      psci22[threadIdx.x] = 0;
      psck00 = 0;
      psck01 = 0;
      psck02 = 0;
      psck10 = 0;
      psck11 = 0;
      psck12 = 0;
      psck20 = 0;
      psck21 = 0;
      psck22 = 0;

      int tri, tx, ty;
      tri = iakpl[iw];
      tri_to_xy(tri, tx, ty);

      int iid = ty * WARP_SIZE + ilane;
      int atomi = min(iid, n - 1);
      int i = sorted[atomi].unsorted;
      int kid = tx * WARP_SIZE + ilane;
      int atomk = min(kid, n - 1);
      int k = sorted[atomk].unsorted;
      xi[threadIdx.x] = sorted[atomi].x;
      yi[threadIdx.x] = sorted[atomi].y;
      zi[threadIdx.x] = sorted[atomi].z;
      xk = sorted[atomk].x;
      yk = sorted[atomk].y;
      zk = sorted[atomk].z;

      springi[threadIdx.x] = kpep[i];
      sizi[threadIdx.x] = prepep[i];
      alphai[threadIdx.x] = dmppep[i];
      epli[threadIdx.x] = lpep[i];
      springk = kpep[k];
      sizk = prepep[k];
      alphak = dmppep[k];
      eplk = lpep[k];

      unsigned int dinfo0 = dinfo[iw * WARP_SIZE + ilane];
      for (int j = 0; j < WARP_SIZE; ++j) {
         int srclane = (ilane + j) & (WARP_SIZE - 1);
         int klane = srclane + threadIdx.x - ilane;
         bool incl = iid < kid and kid < n;
         int srcmask = 1 << srclane;
         incl = incl and (dinfo0 & srcmask) == 0;
         real scaleb = 1;
         real xr = xk - xi[klane];
         real yr = yk - yi[klane];
         real zr = zk - zi[klane];
         real r2 = image2(xr, yr, zr);
         if ((eplk or epli[klane]) and r2 <= off * off and incl) {
            real r = REAL_SQRT(r2);
            real ks2i[3][3], ks2k[3][3];
            pair_alterpol(scrtyp, r, scaleb, cut, off, xr, yr, zr, springi[klane], sizi[klane],
               alphai[klane], springk, sizk, alphak, ks2i, ks2k);
            psci00[klane] += ks2i[0][0];
            psci01[klane] += ks2i[0][1];
            psci02[klane] += ks2i[0][2];
            psci10[klane] += ks2i[1][0];
            psci11[klane] += ks2i[1][1];
            psci12[klane] += ks2i[1][2];
            psci20[klane] += ks2i[2][0];
            psci21[klane] += ks2i[2][1];
            psci22[klane] += ks2i[2][2];
            psck00 += ks2k[0][0];
            psck01 += ks2k[0][1];
            psck02 += ks2k[0][2];
            psck10 += ks2k[1][0];
            psck11 += ks2k[1][1];
            psck12 += ks2k[1][2];
            psck20 += ks2k[2][0];
            psck21 += ks2k[2][1];
            psck22 += ks2k[2][2];
         }

         iid = __shfl_sync(ALL_LANES, iid, ilane + 1);
      }

      atomic_add(psci00[threadIdx.x], &polscale[i][0]);
      atomic_add(psci01[threadIdx.x], &polscale[i][1]);
      atomic_add(psci02[threadIdx.x], &polscale[i][2]);
      atomic_add(psci10[threadIdx.x], &polscale[i][3]);
      atomic_add(psci11[threadIdx.x], &polscale[i][4]);
      atomic_add(psci12[threadIdx.x], &polscale[i][5]);
      atomic_add(psci20[threadIdx.x], &polscale[i][6]);
      atomic_add(psci21[threadIdx.x], &polscale[i][7]);
      atomic_add(psci22[threadIdx.x], &polscale[i][8]);
      atomic_add(psck00, &polscale[k][0]);
      atomic_add(psck01, &polscale[k][1]);
      atomic_add(psck02, &polscale[k][2]);
      atomic_add(psck10, &polscale[k][3]);
      atomic_add(psck11, &polscale[k][4]);
      atomic_add(psck12, &polscale[k][5]);
      atomic_add(psck20, &polscale[k][6]);
      atomic_add(psck21, &polscale[k][7]);
      atomic_add(psck22, &polscale[k][8]);
   }

   for (int iw = iwarp; iw < niak; iw += nwarp) {
      psci00[threadIdx.x] = 0;
      psci01[threadIdx.x] = 0;
      psci02[threadIdx.x] = 0;
      psci10[threadIdx.x] = 0;
      psci11[threadIdx.x] = 0;
      psci12[threadIdx.x] = 0;
      psci20[threadIdx.x] = 0;
      psci21[threadIdx.x] = 0;
      psci22[threadIdx.x] = 0;
      psck00 = 0;
      psck01 = 0;
      psck02 = 0;
      psck10 = 0;
      psck11 = 0;
      psck12 = 0;
      psck20 = 0;
      psck21 = 0;
      psck22 = 0;

      int ty = iak[iw];
      int atomi = ty * WARP_SIZE + ilane;
      int i = sorted[atomi].unsorted;
      int atomk = lst[iw * WARP_SIZE + ilane];
      int k = sorted[atomk].unsorted;
      xi[threadIdx.x] = sorted[atomi].x;
      yi[threadIdx.x] = sorted[atomi].y;
      zi[threadIdx.x] = sorted[atomi].z;
      xk = sorted[atomk].x;
      yk = sorted[atomk].y;
      zk = sorted[atomk].z;

      springi[threadIdx.x] = kpep[i];
      sizi[threadIdx.x] = prepep[i];
      alphai[threadIdx.x] = dmppep[i];
      epli[threadIdx.x] = lpep[i];
      springk = kpep[k];
      sizk = prepep[k];
      alphak = dmppep[k];
      eplk = lpep[k];

      for (int j = 0; j < WARP_SIZE; ++j) {
         int srclane = (ilane + j) & (WARP_SIZE - 1);
         int klane = srclane + threadIdx.x - ilane;
         bool incl = atomk > 0;
         real scaleb = 1;
         real xr = xk - xi[klane];
         real yr = yk - yi[klane];
         real zr = zk - zi[klane];
         real r2 = image2(xr, yr, zr);
         if ((eplk or epli[klane]) and r2 <= off * off and incl) {
            real r = REAL_SQRT(r2);
            real ks2i[3][3], ks2k[3][3];
            pair_alterpol(scrtyp, r, scaleb, cut, off, xr, yr, zr, springi[klane], sizi[klane],
               alphai[klane], springk, sizk, alphak, ks2i, ks2k);
            psci00[klane] += ks2i[0][0];
            psci01[klane] += ks2i[0][1];
            psci02[klane] += ks2i[0][2];
            psci10[klane] += ks2i[1][0];
            psci11[klane] += ks2i[1][1];
            psci12[klane] += ks2i[1][2];
            psci20[klane] += ks2i[2][0];
            psci21[klane] += ks2i[2][1];
            psci22[klane] += ks2i[2][2];
            psck00 += ks2k[0][0];
            psck01 += ks2k[0][1];
            psck02 += ks2k[0][2];
            psck10 += ks2k[1][0];
            psck11 += ks2k[1][1];
            psck12 += ks2k[1][2];
            psck20 += ks2k[2][0];
            psck21 += ks2k[2][1];
            psck22 += ks2k[2][2];
         }
      }

      atomic_add(psci00[threadIdx.x], &polscale[i][0]);
      atomic_add(psci01[threadIdx.x], &polscale[i][1]);
      atomic_add(psci02[threadIdx.x], &polscale[i][2]);
      atomic_add(psci10[threadIdx.x], &polscale[i][3]);
      atomic_add(psci11[threadIdx.x], &polscale[i][4]);
      atomic_add(psci12[threadIdx.x], &polscale[i][5]);
      atomic_add(psci20[threadIdx.x], &polscale[i][6]);
      atomic_add(psci21[threadIdx.x], &polscale[i][7]);
      atomic_add(psci22[threadIdx.x], &polscale[i][8]);
      atomic_add(psck00, &polscale[k][0]);
      atomic_add(psck01, &polscale[k][1]);
      atomic_add(psck02, &polscale[k][2]);
      atomic_add(psck10, &polscale[k][3]);
      atomic_add(psck11, &polscale[k][4]);
      atomic_add(psck12, &polscale[k][5]);
      atomic_add(psck20, &polscale[k][6]);
      atomic_add(psck21, &polscale[k][7]);
      atomic_add(psck22, &polscale[k][8]);
   }
}

__global__
static void alterpolInit_cu1(int n, real (*restrict polscale)[3][3])
{
   for (int i = ITHREAD; i < n; i += STRIDE) {
      polscale[i][0][0] = 1.f;
      polscale[i][0][1] = 0.f;
      polscale[i][0][2] = 0.f;
      polscale[i][1][0] = 0.f;
      polscale[i][1][1] = 1.f;
      polscale[i][1][2] = 0.f;
      polscale[i][2][0] = 0.f;
      polscale[i][2][1] = 0.f;
      polscale[i][2][2] = 1.f;
   }
}

__global__
static void alterpolInvert_cu1(
   int n, real (*restrict polscale)[3][3], real (*restrict polinv)[3][3])
{
   for (int i = ITHREAD; i < n; i += STRIDE) {
      real det;
      real(&ps)[3][3] = polscale[i];
      det = ps[0][0] * (ps[1][1] * ps[2][2] - ps[1][2] * ps[2][1]) -
         ps[1][0] * (ps[0][1] * ps[2][2] - ps[2][1] * ps[0][2]) +
         ps[2][0] * (ps[0][1] * ps[1][2] - ps[1][1] * ps[0][2]);
      polinv[i][0][0] = (ps[1][1] * ps[2][2] - ps[1][2] * ps[2][1]) / det;
      polinv[i][1][0] = (ps[2][0] * ps[1][2] - ps[1][0] * ps[2][2]) / det;
      polinv[i][2][0] = (ps[1][0] * ps[2][1] - ps[2][0] * ps[1][1]) / det;
      polinv[i][0][1] = (ps[2][1] * ps[0][2] - ps[0][1] * ps[2][2]) / det;
      polinv[i][1][1] = (ps[0][0] * ps[2][2] - ps[2][0] * ps[0][2]) / det;
      polinv[i][2][1] = (ps[0][1] * ps[2][0] - ps[0][0] * ps[2][1]) / det;
      polinv[i][0][2] = (ps[0][1] * ps[1][2] - ps[0][2] * ps[1][1]) / det;
      polinv[i][1][2] = (ps[0][2] * ps[1][0] - ps[0][0] * ps[1][2]) / det;
      polinv[i][2][2] = (ps[0][0] * ps[1][1] - ps[0][1] * ps[1][0]) / det;
   }
}

void alterpol_cu(real (*polscale)[3][3], real (*polinv)[3][3])
{
   const auto& st = *mspatial_v2_unit;
   real cut = switchCut(Switch::REPULS);
   real off = switchOff(Switch::REPULS);

   launch_k1s(g::s0, n, alterpolInit_cu1, //
      n, polscale);

   int ngrid = gpuGridSize(BLOCK_DIM);
   alterpol_cu1<<<ngrid, BLOCK_DIM, 0, g::s0>>>(n, TINKER_IMAGE_ARGS, cut, off, st.si2.bit0,
      nmdwexclude, mdwexclude, mdwexclude_scale, st.x, st.y, st.z, st.sorted, st.nakpl, st.iakpl,
      st.niak, st.iak, st.lst, reinterpret_cast<real(*)[9]>(polscale), kpep, prepep, dmppep, lpep,
      scrtyp);

   launch_k1s(g::s0, n, alterpolInvert_cu1, //
      n, polscale, polinv);
}

// ck.py Version 2.0.3

template <class Ver>
__global__
void dexpol_cu1(int n, TINKER_IMAGE_PARAMS, VirialBuffer restrict vep, grad_prec* restrict gx,
   grad_prec* restrict gy, grad_prec* restrict gz, real cut, real off,
   const unsigned* restrict dinfo, int nexclude, const int (*restrict exclude)[2],
   const real (*restrict exclude_scale)[3], const real* restrict x, const real* restrict y,
   const real* restrict z, const Spatial::SortedAtom* restrict sorted, int nakpl,
   const int* restrict iakpl, int niak, const int* restrict iak, const int* restrict lst,
   const real* restrict polarity, const real (*restrict uind)[3], const real* restrict kpep,
   const real* restrict prepep, const real* restrict dmppep, const int* restrict lpep,
   ExpolScr scrtyp, real f)
{
   constexpr bool do_v = Ver::v;
   const int ithread = threadIdx.x + blockIdx.x * blockDim.x;
   const int iwarp = ithread / WARP_SIZE;
   const int nwarp = blockDim.x * gridDim.x / WARP_SIZE;
   const int ilane = threadIdx.x & (WARP_SIZE - 1);

   using vbuf_prec = VirialBufferTraits::type;
   vbuf_prec veptlxx, veptlyx, veptlzx, veptlyy, veptlzy, veptlzz;
   if CONSTEXPR (do_v) {
      veptlxx = 0;
      veptlyx = 0;
      veptlzx = 0;
      veptlyy = 0;
      veptlzy = 0;
      veptlzz = 0;
   }
   __shared__ real xi[BLOCK_DIM];
   __shared__ real yi[BLOCK_DIM];
   __shared__ real zi[BLOCK_DIM];
   real xk;
   real yk;
   real zk;
   __shared__ real frcxi[BLOCK_DIM];
   __shared__ real frcyi[BLOCK_DIM];
   __shared__ real frczi[BLOCK_DIM];
   real frcxk;
   real frcyk;
   real frczk;
   __shared__ real uix[BLOCK_DIM];
   __shared__ real uiy[BLOCK_DIM];
   __shared__ real uiz[BLOCK_DIM];
   __shared__ real springi[BLOCK_DIM];
   __shared__ real sizi[BLOCK_DIM];
   __shared__ real alphai[BLOCK_DIM];
   __shared__ int epli[BLOCK_DIM];
   __shared__ real poli[BLOCK_DIM];
   real ukx;
   real uky;
   real ukz;
   real springk;
   real sizk;
   real alphak;
   int eplk;
   real polk;

   //* /
   for (int ii = ithread; ii < nexclude; ii += blockDim.x * gridDim.x) {
      const int klane = threadIdx.x;
      frcxi[threadIdx.x] = 0;
      frcyi[threadIdx.x] = 0;
      frczi[threadIdx.x] = 0;
      frcxk = 0;
      frcyk = 0;
      frczk = 0;

      int i = exclude[ii][0];
      int k = exclude[ii][1];
      real scaleb = exclude_scale[ii][1];

      xi[klane] = x[i];
      yi[klane] = y[i];
      zi[klane] = z[i];
      xk = x[k];
      yk = y[k];
      zk = z[k];
      uix[klane] = uind[i][0];
      uiy[klane] = uind[i][1];
      uiz[klane] = uind[i][2];
      springi[klane] = kpep[i];
      sizi[klane] = prepep[i];
      alphai[klane] = dmppep[i];
      epli[klane] = lpep[i];
      poli[klane] = polarity[i];
      ukx = uind[k][0];
      uky = uind[k][1];
      ukz = uind[k][2];
      springk = kpep[k];
      sizk = prepep[k];
      alphak = dmppep[k];
      eplk = lpep[k];
      polk = polarity[k];

      constexpr bool incl = true;
      real xr = xk - xi[klane];
      real yr = yk - yi[klane];
      real zr = zk - zi[klane];
      real r2 = image2(xr, yr, zr);
      if ((eplk or epli[klane]) and r2 <= off * off and incl) {
         real r = REAL_SQRT(r2);
         real frc[3];
         pair_dexpol(scrtyp, r, scaleb, cut, off, xr, yr, zr, uix[klane], uiy[klane], uiz[klane],
            ukx, uky, ukz, springi[klane] / poli[klane], sizi[klane], alphai[klane], springk / polk,
            sizk, alphak, f, frc);
         frcxi[klane] += frc[0];
         frcyi[klane] += frc[1];
         frczi[klane] += frc[2];
         frcxk -= frc[0];
         frcyk -= frc[1];
         frczk -= frc[2];

         if CONSTEXPR (do_v) {
            real vxx = -xr * frc[0];
            real vxy = -0.5f * (yr * frc[0] + xr * frc[1]);
            real vxz = -0.5f * (zr * frc[0] + xr * frc[2]);
            real vyy = -yr * frc[1];
            real vyz = -0.5f * (zr * frc[1] + yr * frc[2]);
            real vzz = -zr * frc[2];
            veptlxx += floatTo<vbuf_prec>(vxx);
            veptlyx += floatTo<vbuf_prec>(vxy);
            veptlzx += floatTo<vbuf_prec>(vxz);
            veptlyy += floatTo<vbuf_prec>(vyy);
            veptlzy += floatTo<vbuf_prec>(vyz);
            veptlzz += floatTo<vbuf_prec>(vzz);
         }
      }

      atomic_add(frcxi[threadIdx.x], gx, i);
      atomic_add(frcyi[threadIdx.x], gy, i);
      atomic_add(frczi[threadIdx.x], gz, i);
      atomic_add(frcxk, gx, k);
      atomic_add(frcyk, gy, k);
      atomic_add(frczk, gz, k);
   }
   // */

   for (int iw = iwarp; iw < nakpl; iw += nwarp) {
      frcxi[threadIdx.x] = 0;
      frcyi[threadIdx.x] = 0;
      frczi[threadIdx.x] = 0;
      frcxk = 0;
      frcyk = 0;
      frczk = 0;

      int tri, tx, ty;
      tri = iakpl[iw];
      tri_to_xy(tri, tx, ty);

      int iid = ty * WARP_SIZE + ilane;
      int atomi = min(iid, n - 1);
      int i = sorted[atomi].unsorted;
      int kid = tx * WARP_SIZE + ilane;
      int atomk = min(kid, n - 1);
      int k = sorted[atomk].unsorted;
      xi[threadIdx.x] = sorted[atomi].x;
      yi[threadIdx.x] = sorted[atomi].y;
      zi[threadIdx.x] = sorted[atomi].z;
      xk = sorted[atomk].x;
      yk = sorted[atomk].y;
      zk = sorted[atomk].z;

      uix[threadIdx.x] = uind[i][0];
      uiy[threadIdx.x] = uind[i][1];
      uiz[threadIdx.x] = uind[i][2];
      springi[threadIdx.x] = kpep[i];
      sizi[threadIdx.x] = prepep[i];
      alphai[threadIdx.x] = dmppep[i];
      epli[threadIdx.x] = lpep[i];
      poli[threadIdx.x] = polarity[i];
      ukx = uind[k][0];
      uky = uind[k][1];
      ukz = uind[k][2];
      springk = kpep[k];
      sizk = prepep[k];
      alphak = dmppep[k];
      eplk = lpep[k];
      polk = polarity[k];

      unsigned int dinfo0 = dinfo[iw * WARP_SIZE + ilane];
      for (int j = 0; j < WARP_SIZE; ++j) {
         int srclane = (ilane + j) & (WARP_SIZE - 1);
         int klane = srclane + threadIdx.x - ilane;
         bool incl = iid < kid and kid < n;
         int srcmask = 1 << srclane;
         incl = incl and (dinfo0 & srcmask) == 0;
         real scaleb = 1;
         real xr = xk - xi[klane];
         real yr = yk - yi[klane];
         real zr = zk - zi[klane];
         real r2 = image2(xr, yr, zr);
         if ((eplk or epli[klane]) and r2 <= off * off and incl) {
            real r = REAL_SQRT(r2);
            real frc[3];
            pair_dexpol(scrtyp, r, scaleb, cut, off, xr, yr, zr, uix[klane], uiy[klane], uiz[klane],
               ukx, uky, ukz, springi[klane] / poli[klane], sizi[klane], alphai[klane],
               springk / polk, sizk, alphak, f, frc);
            frcxi[klane] += frc[0];
            frcyi[klane] += frc[1];
            frczi[klane] += frc[2];
            frcxk -= frc[0];
            frcyk -= frc[1];
            frczk -= frc[2];

            if CONSTEXPR (do_v) {
               real vxx = -xr * frc[0];
               real vxy = -0.5f * (yr * frc[0] + xr * frc[1]);
               real vxz = -0.5f * (zr * frc[0] + xr * frc[2]);
               real vyy = -yr * frc[1];
               real vyz = -0.5f * (zr * frc[1] + yr * frc[2]);
               real vzz = -zr * frc[2];
               veptlxx += floatTo<vbuf_prec>(vxx);
               veptlyx += floatTo<vbuf_prec>(vxy);
               veptlzx += floatTo<vbuf_prec>(vxz);
               veptlyy += floatTo<vbuf_prec>(vyy);
               veptlzy += floatTo<vbuf_prec>(vyz);
               veptlzz += floatTo<vbuf_prec>(vzz);
            }
         }

         iid = __shfl_sync(ALL_LANES, iid, ilane + 1);
      }

      atomic_add(frcxi[threadIdx.x], gx, i);
      atomic_add(frcyi[threadIdx.x], gy, i);
      atomic_add(frczi[threadIdx.x], gz, i);
      atomic_add(frcxk, gx, k);
      atomic_add(frcyk, gy, k);
      atomic_add(frczk, gz, k);
   }

   for (int iw = iwarp; iw < niak; iw += nwarp) {
      frcxi[threadIdx.x] = 0;
      frcyi[threadIdx.x] = 0;
      frczi[threadIdx.x] = 0;
      frcxk = 0;
      frcyk = 0;
      frczk = 0;

      int ty = iak[iw];
      int atomi = ty * WARP_SIZE + ilane;
      int i = sorted[atomi].unsorted;
      int atomk = lst[iw * WARP_SIZE + ilane];
      int k = sorted[atomk].unsorted;
      xi[threadIdx.x] = sorted[atomi].x;
      yi[threadIdx.x] = sorted[atomi].y;
      zi[threadIdx.x] = sorted[atomi].z;
      xk = sorted[atomk].x;
      yk = sorted[atomk].y;
      zk = sorted[atomk].z;

      uix[threadIdx.x] = uind[i][0];
      uiy[threadIdx.x] = uind[i][1];
      uiz[threadIdx.x] = uind[i][2];
      springi[threadIdx.x] = kpep[i];
      sizi[threadIdx.x] = prepep[i];
      alphai[threadIdx.x] = dmppep[i];
      epli[threadIdx.x] = lpep[i];
      poli[threadIdx.x] = polarity[i];
      ukx = uind[k][0];
      uky = uind[k][1];
      ukz = uind[k][2];
      springk = kpep[k];
      sizk = prepep[k];
      alphak = dmppep[k];
      eplk = lpep[k];
      polk = polarity[k];

      for (int j = 0; j < WARP_SIZE; ++j) {
         int srclane = (ilane + j) & (WARP_SIZE - 1);
         int klane = srclane + threadIdx.x - ilane;
         bool incl = atomk > 0;
         real scaleb = 1;
         real xr = xk - xi[klane];
         real yr = yk - yi[klane];
         real zr = zk - zi[klane];
         real r2 = image2(xr, yr, zr);
         if ((eplk or epli[klane]) and r2 <= off * off and incl) {
            real r = REAL_SQRT(r2);
            real frc[3];
            pair_dexpol(scrtyp, r, scaleb, cut, off, xr, yr, zr, uix[klane], uiy[klane], uiz[klane],
               ukx, uky, ukz, springi[klane] / poli[klane], sizi[klane], alphai[klane],
               springk / polk, sizk, alphak, f, frc);
            frcxi[klane] += frc[0];
            frcyi[klane] += frc[1];
            frczi[klane] += frc[2];
            frcxk -= frc[0];
            frcyk -= frc[1];
            frczk -= frc[2];

            if CONSTEXPR (do_v) {
               real vxx = -xr * frc[0];
               real vxy = -0.5f * (yr * frc[0] + xr * frc[1]);
               real vxz = -0.5f * (zr * frc[0] + xr * frc[2]);
               real vyy = -yr * frc[1];
               real vyz = -0.5f * (zr * frc[1] + yr * frc[2]);
               real vzz = -zr * frc[2];
               veptlxx += floatTo<vbuf_prec>(vxx);
               veptlyx += floatTo<vbuf_prec>(vxy);
               veptlzx += floatTo<vbuf_prec>(vxz);
               veptlyy += floatTo<vbuf_prec>(vyy);
               veptlzy += floatTo<vbuf_prec>(vyz);
               veptlzz += floatTo<vbuf_prec>(vzz);
            }
         }
      }

      atomic_add(frcxi[threadIdx.x], gx, i);
      atomic_add(frcyi[threadIdx.x], gy, i);
      atomic_add(frczi[threadIdx.x], gz, i);
      atomic_add(frcxk, gx, k);
      atomic_add(frcyk, gy, k);
      atomic_add(frczk, gz, k);
   }

   if CONSTEXPR (do_v) {
      atomic_add(veptlxx, veptlyx, veptlzx, veptlyy, veptlzy, veptlzz, vep, ithread);
   }
}

void dexpol_cu(int vers, const real (*uind)[3], grad_prec* depx, grad_prec* depy, grad_prec* depz,
   VirialBuffer restrict vir_ep)
{
   const auto& st = *mspatial_v2_unit;
   real cut = switchCut(Switch::REPULS);
   real off = switchOff(Switch::REPULS);

   real f = 0.5f * electric / dielec;

   int ngrid = gpuGridSize(BLOCK_DIM);

#define DEXPOL_CU1_ARGS                                                                            \
   n, TINKER_IMAGE_ARGS, vir_ep, depx, depy, depz, cut, off, st.si2.bit0, nmdwexclude, mdwexclude, \
      mdwexclude_scale, st.x, st.y, st.z, st.sorted, st.nakpl, st.iakpl, st.niak, st.iak, st.lst,  \
      polarity, uind, kpep, prepep, dmppep, lpep, scrtyp, f

   if (vers & calc::virial) {
      dexpol_cu1<calc::V6><<<ngrid, BLOCK_DIM, 0, g::s0>>>(DEXPOL_CU1_ARGS);
   } else if (vers & calc::grad) {
      dexpol_cu1<calc::V5><<<ngrid, BLOCK_DIM, 0, g::s0>>>(DEXPOL_CU1_ARGS);
   } else {
      assert(false &&
         "This function should not have been called if neither gradient nor virial is calculated.");
   }

#undef DEXPOL_CU1_ARGS
}
}

#include "ff/amoeba/induce.h"
#include "ff/amoebamod.h"
#include "ff/atom.h"
#include "ff/hippo/induce.h"
#include "ff/hippomod.h"
#include "ff/switch.h"
#include "seq/launch.h"
#include "tool/darray.h"
#include "tool/error.h"
#include "tool/ioprint.h"
#include <tinker/detail/inform.hh>
#include <tinker/detail/polpcg.hh>
#include <tinker/detail/polpot.hh>
#include <tinker/detail/units.hh>

namespace tinker {
__global__
void eppcgUdirDonly(
   int n, const real* restrict polarity, real (*restrict udir)[3], const real (*restrict field)[3])
{
   for (int i = ITHREAD; i < n; i += STRIDE) {
      real poli = polarity[i];
      #pragma unroll
      for (int j = 0; j < 3; ++j) {
         udir[i][j] = poli * field[i][j];
      }
   }
}

__global__
void eppcgUdirGuess(int n, const real* restrict polarity, real (*restrict uind)[3],
   const real (*restrict field)[3], const real (*restrict polinv)[3][3])
{
   for (int i = ITHREAD; i < n; i += STRIDE) {
      real poli = polarity[i];
      #pragma unroll
      for (int j = 0; j < 3; ++j) {
         uind[i][j] = poli *
            (polinv[i][0][j] * field[i][0] + polinv[i][1][j] * field[i][1] +
               polinv[i][2][j] * field[i][2]);
      }
   }
}

__global__
void eppcgRsd2(int n, const real* restrict polarity_inv, //
   real (*restrict rsd)[3],                              //
   const real (*restrict udir)[3], const real (*restrict uind)[3], const real (*restrict field)[3],
   const real (*restrict polscale)[3][3])
{
   for (int i = ITHREAD; i < n; i += STRIDE) {
      real poli_inv = polarity_inv[i];
      #pragma unroll
      for (int j = 0; j < 3; ++j) {
         rsd[i][j] = (udir[i][j] - uind[i][0] * polscale[i][0][j] - uind[i][1] * polscale[i][1][j] -
                        uind[i][2] * polscale[i][2][j]) *
               poli_inv +
            field[i][j];
      }
   }
}

__global__
void eppcgRsd1(int n, const real* restrict polarity, real (*restrict rsd)[3])
{
   for (int i = ITHREAD; i < n; i += STRIDE) {
      if (polarity[i] == 0) {
         rsd[i][0] = 0;
         rsd[i][1] = 0;
         rsd[i][2] = 0;
      }
   }
}

__global__
void eppcgP4(int n, const real* restrict polarity_inv, real (*restrict vec)[3],
   const real (*restrict conj)[3], const real (*restrict field)[3],
   const real (*restrict polscale)[3][3])
{
   for (int i = ITHREAD; i < n; i += STRIDE) {
      real poli_inv = polarity_inv[i];
      #pragma unroll
      for (int j = 0; j < 3; ++j)
         vec[i][j] = poli_inv *
               (conj[i][0] * polscale[i][0][j] + conj[i][1] * polscale[i][1][j] +
                  conj[i][2] * polscale[i][2][j]) -
            field[i][j];
   }
}

__global__
void eppcgP5(int n, const real* restrict polarity, //
   const real* restrict ka,                        //
   const real* restrict ksum, real (*restrict uind)[3], const real (*restrict conj)[3],
   real (*restrict rsd)[3], const real (*restrict vec)[3])
{
   real kaval = *ka;
   real a = *ksum / kaval;
   if (kaval == 0) a = 0;
   for (int i = ITHREAD; i < n; i += STRIDE) {
      #pragma unroll
      for (int j = 0; j < 3; ++j) {
         uind[i][j] += a * conj[i][j];
         rsd[i][j] -= a * vec[i][j];
      }
      if (polarity[i] == 0) {
         rsd[i][0] = 0;
         rsd[i][1] = 0;
         rsd[i][2] = 0;
      }
   }
}

__global__
void eppcgP6(int n, const real* restrict ksum, const real* restrict ksum1, real (*restrict conj)[3],
   real (*restrict zrsd)[3])
{
   real ksumval = *ksum;
   real b = *ksum1 / ksumval;
   if (ksumval == 0) b = 0;
   for (int i = ITHREAD; i < n; i += STRIDE) {
      #pragma unroll
      for (int j = 0; j < 3; ++j)
         conj[i][j] = zrsd[i][j] + b * conj[i][j];
   }
}

__global__
void eppcgPeek1(int n, float pcgpeek, const real* restrict polarity, real (*restrict uind)[3],
   const real (*restrict rsd)[3])
{
   for (int i = ITHREAD; i < n; i += STRIDE) {
      real term = pcgpeek * polarity[i];
      #pragma unroll
      for (int j = 0; j < 3; ++j)
         uind[i][j] += term * rsd[i][j];
   }
}

void induceMutualPcg4_cu(real (*uind)[3])
{
   auto* field = work01_;
   auto* rsd = work02_;
   auto* zrsd = work03_;
   auto* conj = work04_;
   auto* vec = work05_;

   const bool sparse_prec = polpcg::pcgprec and (switchOff(Switch::USOLVE) > 0);
   bool dirguess = polpcg::pcgguess;
   bool predict = polpred != UPred::NONE;
   if (predict and nualt < maxualt) {
      predict = false;
      dirguess = true;
   }

   // get the electrostatic field due to permanent multipoles
   dfieldChgpen(field);
   // direct induced dipoles
   launch_k1s(g::s0, n, eppcgUdirDonly, n, polarity, udir, field);

   alterpol(polscale, polinv);

   // initial induced dipole
   if (predict) {
      ulspredSum(uind, nullptr);
   } else if (dirguess) {
      launch_k1s(g::s0, n, eppcgUdirGuess, n, polarity, uind, field, polinv);
   } else {
      darray::zero(g::q0, n, uind);
   }

   if (predict) {
      ufieldChgpen(uind, field);
      launch_k1s(g::s0, n, eppcgRsd2, n, polarity_inv, rsd, udir, uind, field, polscale);
   } else if (dirguess) {
      // uind is used here instead of udir since without exchange polarization udir = uind
      // but with exchange polarization udir != uind (for dirguess).
      ufieldChgpen(uind, rsd);
   } else {
      darray::copy(g::q0, n, rsd, field);
   }
   launch_k1s(g::s0, n, eppcgRsd1, n, polarity, rsd);

   // initial M r(0) and p(0)
   if (sparse_prec) {
      sparsePrecondBuild2();
      sparsePrecondApply2(rsd, zrsd);
   } else {
      diagPrecond2(rsd, zrsd);
   }
   darray::copy(g::q0, n, conj, zrsd);

   // initial r(0) M r(0)
   real* sum = &((real*)dptr_buf)[0];
   darray::dot(g::q0, n, sum, rsd, zrsd);

   // conjugate gradient iteration of the mutual induced dipoles
   const bool debug = inform::debug;
   const int politer = polpot::politer;
   const real poleps = polpot::poleps;
   const real debye = units::debye;
   const real pcgpeek = polpcg::pcgpeek;
   const int maxiter = 100; // see also subroutine induce0a in induce.f
   const int miniter = std::min(3, n);

   bool done = false;
   int iter = 0;
   real eps = 100;
   // real epsold;

   while (not done) {
      ++iter;

      // T p and p
      // vec = (inv_alpha + Tu) conj, field = -Tu conj
      // vec = inv_alpha * conj - field
      ufieldChgpen(conj, field);
      launch_k1s(g::s0, n, eppcgP4, n, polarity_inv, vec, conj, field, polscale);

      // a <- p T p
      real* a = &((real*)dptr_buf)[1];
      // a <- r M r / p T p; a = sum / a; ap = sump / ap
      darray::dot(g::q0, n, a, conj, vec);

      // u <- u + a p
      // r <- r - a T p
      launch_k1s(g::s0, n, eppcgP5, n, polarity, a, sum, uind, conj, rsd, vec);

      // calculate/update M r
      if (sparse_prec)
         sparsePrecondApply2(rsd, zrsd);
      else
         diagPrecond2(rsd, zrsd);

      // b = sum1 / sum; bp = sump1 / sump
      real* sum1 = &((real*)dptr_buf)[2];
      darray::dot(g::q0, n, sum1, rsd, zrsd);

      // calculate/update p
      launch_k1s(g::s0, n, eppcgP6, n, sum, sum1, conj, zrsd);

      // copy sum1/p to sum/p
      darray::copy(g::q0, 2, sum, sum1);

      real* epsd = &((real*)dptr_buf)[3];
      darray::dot(g::q0, n, epsd, rsd, rsd);
      check_rt(
         hipMemcpyAsync((real*)pinned_buf, epsd, sizeof(real), hipMemcpyDeviceToHost, g::s0));
      check_rt(hipStreamSynchronize(g::s0));
      // epsold = eps;
      eps = ((real*)pinned_buf)[0];
      eps = debye * REAL_SQRT(eps / n);

      if (debug) {
         if (iter == 1) {
            print(stdout,
               "\n Determination of SCF Induced Dipole Moments\n\n"
               "    Iter    RMS Residual (Debye)\n\n");
         }
         print(stdout, " %8d       %-16.10f\n", iter, eps);
      }

      if (eps < poleps) done = true;
      // if (eps > epsold) done = true;
      if (iter < miniter) done = false;
      if (iter >= politer) done = true;

      // apply a "peek" iteration to the mutual induced dipoles
      if (done) launch_k1s(g::s0, n, eppcgPeek1, n, pcgpeek, polarity, uind, rsd);
   }

   // print the results from the conjugate gradient iteration
   if (debug) {
      print(stdout,
         " Induced Dipoles :    Iterations %4d      RMS"
         " Residual %14.10f\n",
         iter, eps);
   }

   // terminate the calculation if dipoles failed to converge
   if (iter >= maxiter) {
      printError();
      TINKER_THROW("INDUCE  --  Warning, Induced Dipoles are not Converged");
   }
}
}
