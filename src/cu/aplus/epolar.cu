#include "hip/hip_runtime.h"
#include "ff/amoebamod.h"
#include "ff/aplusmod.h"
#include "ff/hippomod.h"
#include "ff/image.h"
#include "ff/pme.h"
#include "ff/spatial.h"
#include "ff/switch.h"
#include "seq/epolartorque.h"
#include "seq/launch.h"
#include "seq/pairpolaraplus.h"
#include "seq/triangle.h"

namespace tinker {
template <class Ver, class ETYP, bool CFLX>
__global__
void epolarAplus_cu1(int n, TINKER_IMAGE_PARAMS, CountBuffer restrict nep, EnergyBuffer restrict ep,
   VirialBuffer restrict vep, grad_prec* restrict gx, grad_prec* restrict gy,
   grad_prec* restrict gz, real off, const unsigned* restrict mdpuinfo, int nexclude,
   const int (*restrict exclude)[2], const real (*restrict exclude_scale)[4],
   const real* restrict x, const real* restrict y, const real* restrict z,
   const Spatial::SortedAtom* restrict sorted, int nakpl, const int* restrict iakpl, int niak,
   const int* restrict iak, const int* restrict lst, real (*restrict ufld)[3],
   real (*restrict dufld)[6], const real (*restrict uind)[3], real* restrict pot,
   const real (*restrict rpole)[10], const real* restrict pdamp, const real* restrict thole,
   const real* restrict dirdamp, real aewald, real f)
{
   constexpr bool do_e = Ver::e;
   constexpr bool do_a = Ver::a;
   constexpr bool do_g = Ver::g;
   constexpr bool do_v = Ver::v;
   const int ithread = threadIdx.x + blockIdx.x * blockDim.x;
   const int iwarp = ithread / WARP_SIZE;
   const int nwarp = blockDim.x * gridDim.x / WARP_SIZE;
   const int ilane = threadIdx.x & (WARP_SIZE - 1);

   int neptl;
   if CONSTEXPR (do_a) {
      neptl = 0;
   }
   using ebuf_prec = EnergyBufferTraits::type;
   ebuf_prec eptl;
   if CONSTEXPR (do_e) {
      eptl = 0;
   }
   using vbuf_prec = VirialBufferTraits::type;
   vbuf_prec veptlxx, veptlyx, veptlzx, veptlyy, veptlzy, veptlzz;
   if CONSTEXPR (do_v) {
      veptlxx = 0;
      veptlyx = 0;
      veptlzx = 0;
      veptlyy = 0;
      veptlzy = 0;
      veptlzz = 0;
   }
   __shared__ real xi[BLOCK_DIM];
   __shared__ real yi[BLOCK_DIM];
   __shared__ real zi[BLOCK_DIM];
   real xk;
   real yk;
   real zk;
   __shared__ real frcxi[BLOCK_DIM];
   __shared__ real frcyi[BLOCK_DIM];
   __shared__ real frczi[BLOCK_DIM];
   __shared__ real ufld0i[BLOCK_DIM];
   __shared__ real ufld1i[BLOCK_DIM];
   __shared__ real ufld2i[BLOCK_DIM];
   __shared__ real dufld0i[BLOCK_DIM];
   __shared__ real dufld1i[BLOCK_DIM];
   __shared__ real dufld2i[BLOCK_DIM];
   __shared__ real dufld3i[BLOCK_DIM];
   __shared__ real dufld4i[BLOCK_DIM];
   __shared__ real dufld5i[BLOCK_DIM];
   __shared__ real poti[BLOCK_DIM];
   real frcxk;
   real frcyk;
   real frczk;
   real ufld0k;
   real ufld1k;
   real ufld2k;
   real dufld0k;
   real dufld1k;
   real dufld2k;
   real dufld3k;
   real dufld4k;
   real dufld5k;
   real potk;
   __shared__ real ci[BLOCK_DIM];
   __shared__ real dix[BLOCK_DIM];
   __shared__ real diy[BLOCK_DIM];
   __shared__ real diz[BLOCK_DIM];
   __shared__ real qixx[BLOCK_DIM];
   __shared__ real qixy[BLOCK_DIM];
   __shared__ real qixz[BLOCK_DIM];
   __shared__ real qiyy[BLOCK_DIM];
   __shared__ real qiyz[BLOCK_DIM];
   __shared__ real qizz[BLOCK_DIM];
   __shared__ real uix[BLOCK_DIM];
   __shared__ real uiy[BLOCK_DIM];
   __shared__ real uiz[BLOCK_DIM];
   __shared__ real pdi[BLOCK_DIM];
   __shared__ real pti[BLOCK_DIM];
   __shared__ real ddi[BLOCK_DIM];
   real ck;
   real dkx;
   real dky;
   real dkz;
   real qkxx;
   real qkxy;
   real qkxz;
   real qkyy;
   real qkyz;
   real qkzz;
   real ukx;
   real uky;
   real ukz;
   real pdk;
   real ptk;
   real ddk;

   //* /
   for (int ii = ithread; ii < nexclude; ii += blockDim.x * gridDim.x) {
      const int klane = threadIdx.x;
      if CONSTEXPR (do_g) {
         frcxi[threadIdx.x] = 0;
         frcyi[threadIdx.x] = 0;
         frczi[threadIdx.x] = 0;
         ufld0i[threadIdx.x] = 0;
         ufld1i[threadIdx.x] = 0;
         ufld2i[threadIdx.x] = 0;
         dufld0i[threadIdx.x] = 0;
         dufld1i[threadIdx.x] = 0;
         dufld2i[threadIdx.x] = 0;
         dufld3i[threadIdx.x] = 0;
         dufld4i[threadIdx.x] = 0;
         dufld5i[threadIdx.x] = 0;
         if CONSTEXPR (CFLX)
            poti[threadIdx.x] = 0;
         frcxk = 0;
         frcyk = 0;
         frczk = 0;
         ufld0k = 0;
         ufld1k = 0;
         ufld2k = 0;
         dufld0k = 0;
         dufld1k = 0;
         dufld2k = 0;
         dufld3k = 0;
         dufld4k = 0;
         dufld5k = 0;
         if CONSTEXPR (CFLX)
            potk = 0;
      }

      int i = exclude[ii][0];
      int k = exclude[ii][1];
      real scaleb = exclude_scale[ii][2]; // p
      real scaled = exclude_scale[ii][3]; // u

      xi[klane] = x[i];
      yi[klane] = y[i];
      zi[klane] = z[i];
      xk = x[k];
      yk = y[k];
      zk = z[k];
      ci[klane] = rpole[i][MPL_PME_0];
      dix[klane] = rpole[i][MPL_PME_X];
      diy[klane] = rpole[i][MPL_PME_Y];
      diz[klane] = rpole[i][MPL_PME_Z];
      qixx[klane] = rpole[i][MPL_PME_XX];
      qixy[klane] = rpole[i][MPL_PME_XY];
      qixz[klane] = rpole[i][MPL_PME_XZ];
      qiyy[klane] = rpole[i][MPL_PME_YY];
      qiyz[klane] = rpole[i][MPL_PME_YZ];
      qizz[klane] = rpole[i][MPL_PME_ZZ];
      uix[klane] = uind[i][0];
      uiy[klane] = uind[i][1];
      uiz[klane] = uind[i][2];
      pdi[klane] = pdamp[i];
      pti[klane] = thole[i];
      ddi[klane] = dirdamp[i];
      ck = rpole[k][MPL_PME_0];
      dkx = rpole[k][MPL_PME_X];
      dky = rpole[k][MPL_PME_Y];
      dkz = rpole[k][MPL_PME_Z];
      qkxx = rpole[k][MPL_PME_XX];
      qkxy = rpole[k][MPL_PME_XY];
      qkxz = rpole[k][MPL_PME_XZ];
      qkyy = rpole[k][MPL_PME_YY];
      qkyz = rpole[k][MPL_PME_YZ];
      qkzz = rpole[k][MPL_PME_ZZ];
      ukx = uind[k][0];
      uky = uind[k][1];
      ukz = uind[k][2];
      pdk = pdamp[k];
      ptk = thole[k];
      ddk = dirdamp[k];

      constexpr bool incl = true;
      real xr = xk - xi[klane];
      real yr = yk - yi[klane];
      real zr = zk - zi[klane];
      real r2 = image2(xr, yr, zr);
      if (r2 <= off * off and incl) {
         real e, vxx, vyx, vzx, vyy, vzy, vzz;
         real e1, vxx1, vyx1, vzx1, vyy1, vzy1, vzz1;
         real pota, potb;
         real pota1, potb1;
         pair_polar_aplus_v2<Ver, ETYP, CFLX>(                               //
            r2, xr, yr, zr, 1, 1,                                            //
            ci[klane], dix[klane], diy[klane], diz[klane], qixx[klane],      //
            qixy[klane], qixz[klane], qiyy[klane], qiyz[klane], qizz[klane], //
            uix[klane], uiy[klane], uiz[klane],                              //
            pdi[klane], pti[klane], ddi[klane],                              //
            ck, dkx, dky, dkz, qkxx, qkxy, qkxz, qkyy, qkyz, qkzz, ukx,      //
            uky, ukz, pdk, ptk, ddk,                                         //
            f, aewald,                                                       //
            frcxi[klane], frcyi[klane], frczi[klane], frcxk, frcyk, frczk, ufld0i[klane],
            ufld1i[klane], ufld2i[klane], ufld0k, ufld1k,
            ufld2k, //
            dufld0i[klane], dufld1i[klane], dufld2i[klane], dufld3i[klane], dufld4i[klane],
            dufld5i[klane], dufld0k, dufld1k, dufld2k, dufld3k, dufld4k, dufld5k, //
            e1, vxx1, vyx1, vzx1, vyy1, vzy1, vzz1, pota1, potb1);
         pair_polar_aplus_v2<Ver, NON_EWALD, CFLX>(                          //
            r2, xr, yr, zr, scaleb - 1, scaled - 1,                          //
            ci[klane], dix[klane], diy[klane], diz[klane], qixx[klane],      //
            qixy[klane], qixz[klane], qiyy[klane], qiyz[klane], qizz[klane], //
            uix[klane], uiy[klane], uiz[klane],                              //
            pdi[klane], pti[klane], ddi[klane],                              //
            ck, dkx, dky, dkz, qkxx, qkxy, qkxz, qkyy, qkyz, qkzz, ukx,      //
            uky, ukz, pdk, ptk, ddk,                                         //
            f, aewald,                                                       //
            frcxi[klane], frcyi[klane], frczi[klane], frcxk, frcyk, frczk, ufld0i[klane],
            ufld1i[klane], ufld2i[klane], ufld0k, ufld1k,
            ufld2k, //
            dufld0i[klane], dufld1i[klane], dufld2i[klane], dufld3i[klane], dufld4i[klane],
            dufld5i[klane], dufld0k, dufld1k, dufld2k, dufld3k, dufld4k, dufld5k, //
            e, vxx, vyx, vzx, vyy, vzy, vzz, pota, potb);
         if CONSTEXPR (do_e) {
            e = e + e1;
            eptl += floatTo<ebuf_prec>(e);
            if CONSTEXPR (do_a) {
               if (e != 0 and scaleb != 0)
                  neptl += 1;
            }
         }
         if CONSTEXPR (do_v) {
            veptlxx += floatTo<vbuf_prec>(vxx + vxx1);
            veptlyx += floatTo<vbuf_prec>(vyx + vyx1);
            veptlzx += floatTo<vbuf_prec>(vzx + vzx1);
            veptlyy += floatTo<vbuf_prec>(vyy + vyy1);
            veptlzy += floatTo<vbuf_prec>(vzy + vzy1);
            veptlzz += floatTo<vbuf_prec>(vzz + vzz1);
         }
         if CONSTEXPR (CFLX) {
            poti[klane] += (pota + pota1);
            potk += (potb + potb1);
         }
      } // end if (include)

      if CONSTEXPR (do_g) {
         atomic_add(frcxi[threadIdx.x], gx, i);
         atomic_add(frcyi[threadIdx.x], gy, i);
         atomic_add(frczi[threadIdx.x], gz, i);
         atomic_add(ufld0i[threadIdx.x], &ufld[i][0]);
         atomic_add(ufld1i[threadIdx.x], &ufld[i][1]);
         atomic_add(ufld2i[threadIdx.x], &ufld[i][2]);
         atomic_add(dufld0i[threadIdx.x], &dufld[i][0]);
         atomic_add(dufld1i[threadIdx.x], &dufld[i][1]);
         atomic_add(dufld2i[threadIdx.x], &dufld[i][2]);
         atomic_add(dufld3i[threadIdx.x], &dufld[i][3]);
         atomic_add(dufld4i[threadIdx.x], &dufld[i][4]);
         atomic_add(dufld5i[threadIdx.x], &dufld[i][5]);
         if CONSTEXPR (CFLX)
            atomic_add(poti[threadIdx.x], pot, i);
         atomic_add(frcxk, gx, k);
         atomic_add(frcyk, gy, k);
         atomic_add(frczk, gz, k);
         atomic_add(ufld0k, &ufld[k][0]);
         atomic_add(ufld1k, &ufld[k][1]);
         atomic_add(ufld2k, &ufld[k][2]);
         atomic_add(dufld0k, &dufld[k][0]);
         atomic_add(dufld1k, &dufld[k][1]);
         atomic_add(dufld2k, &dufld[k][2]);
         atomic_add(dufld3k, &dufld[k][3]);
         atomic_add(dufld4k, &dufld[k][4]);
         atomic_add(dufld5k, &dufld[k][5]);
         if CONSTEXPR (CFLX)
            atomic_add(potk, pot, k);
      }
   }
   // */

   for (int iw = iwarp; iw < nakpl; iw += nwarp) {
      if CONSTEXPR (do_g) {
         frcxi[threadIdx.x] = 0;
         frcyi[threadIdx.x] = 0;
         frczi[threadIdx.x] = 0;
         ufld0i[threadIdx.x] = 0;
         ufld1i[threadIdx.x] = 0;
         ufld2i[threadIdx.x] = 0;
         dufld0i[threadIdx.x] = 0;
         dufld1i[threadIdx.x] = 0;
         dufld2i[threadIdx.x] = 0;
         dufld3i[threadIdx.x] = 0;
         dufld4i[threadIdx.x] = 0;
         dufld5i[threadIdx.x] = 0;
         if CONSTEXPR (CFLX)
            poti[threadIdx.x] = 0;
         frcxk = 0;
         frcyk = 0;
         frczk = 0;
         ufld0k = 0;
         ufld1k = 0;
         ufld2k = 0;
         dufld0k = 0;
         dufld1k = 0;
         dufld2k = 0;
         dufld3k = 0;
         dufld4k = 0;
         dufld5k = 0;
         if CONSTEXPR (CFLX)
            potk = 0;
      }

      int tri, tx, ty;
      tri = iakpl[iw];
      tri_to_xy(tri, tx, ty);

      int iid = ty * WARP_SIZE + ilane;
      int atomi = min(iid, n - 1);
      int i = sorted[atomi].unsorted;
      int kid = tx * WARP_SIZE + ilane;
      int atomk = min(kid, n - 1);
      int k = sorted[atomk].unsorted;
      xi[threadIdx.x] = sorted[atomi].x;
      yi[threadIdx.x] = sorted[atomi].y;
      zi[threadIdx.x] = sorted[atomi].z;
      xk = sorted[atomk].x;
      yk = sorted[atomk].y;
      zk = sorted[atomk].z;

      ci[threadIdx.x] = rpole[i][MPL_PME_0];
      dix[threadIdx.x] = rpole[i][MPL_PME_X];
      diy[threadIdx.x] = rpole[i][MPL_PME_Y];
      diz[threadIdx.x] = rpole[i][MPL_PME_Z];
      qixx[threadIdx.x] = rpole[i][MPL_PME_XX];
      qixy[threadIdx.x] = rpole[i][MPL_PME_XY];
      qixz[threadIdx.x] = rpole[i][MPL_PME_XZ];
      qiyy[threadIdx.x] = rpole[i][MPL_PME_YY];
      qiyz[threadIdx.x] = rpole[i][MPL_PME_YZ];
      qizz[threadIdx.x] = rpole[i][MPL_PME_ZZ];
      uix[threadIdx.x] = uind[i][0];
      uiy[threadIdx.x] = uind[i][1];
      uiz[threadIdx.x] = uind[i][2];
      pdi[threadIdx.x] = pdamp[i];
      pti[threadIdx.x] = thole[i];
      ddi[threadIdx.x] = dirdamp[i];
      ck = rpole[k][MPL_PME_0];
      dkx = rpole[k][MPL_PME_X];
      dky = rpole[k][MPL_PME_Y];
      dkz = rpole[k][MPL_PME_Z];
      qkxx = rpole[k][MPL_PME_XX];
      qkxy = rpole[k][MPL_PME_XY];
      qkxz = rpole[k][MPL_PME_XZ];
      qkyy = rpole[k][MPL_PME_YY];
      qkyz = rpole[k][MPL_PME_YZ];
      qkzz = rpole[k][MPL_PME_ZZ];
      ukx = uind[k][0];
      uky = uind[k][1];
      ukz = uind[k][2];
      pdk = pdamp[k];
      ptk = thole[k];
      ddk = dirdamp[k];

      unsigned int mdpuinfo0 = mdpuinfo[iw * WARP_SIZE + ilane];
      for (int j = 0; j < WARP_SIZE; ++j) {
         int srclane = (ilane + j) & (WARP_SIZE - 1);
         int klane = srclane + threadIdx.x - ilane;
         bool incl = iid < kid and kid < n;
         int srcmask = 1 << srclane;
         incl = incl and (mdpuinfo0 & srcmask) == 0;
         real xr = xk - xi[klane];
         real yr = yk - yi[klane];
         real zr = zk - zi[klane];
         real r2 = image2(xr, yr, zr);
         if (r2 <= off * off and incl) {
            real e, vxx, vyx, vzx, vyy, vzy, vzz;
            real pota, potb;
            pair_polar_aplus_v2<Ver, ETYP, CFLX>(                                           //
               r2, xr, yr, zr, 1, 1, ci[klane], dix[klane], diy[klane], diz[klane], qixx[klane], //
               qixy[klane], qixz[klane], qiyy[klane], qiyz[klane], qizz[klane],                  //
               uix[klane], uiy[klane], uiz[klane],                                               //
               pdi[klane], pti[klane], ddi[klane],                                               //
               ck, dkx, dky, dkz, qkxx, qkxy, qkxz, qkyy, qkyz, qkzz, ukx,                       //
               uky, ukz, pdk, ptk, ddk,                                                          //
               f, aewald,                                                                        //
               frcxi[klane], frcyi[klane], frczi[klane], frcxk, frcyk, frczk, ufld0i[klane],
               ufld1i[klane], ufld2i[klane], ufld0k, ufld1k,
               ufld2k, //
               dufld0i[klane], dufld1i[klane], dufld2i[klane], dufld3i[klane], dufld4i[klane],
               dufld5i[klane], dufld0k, dufld1k, dufld2k, dufld3k, dufld4k, dufld5k, //
               e, vxx, vyx, vzx, vyy, vzy, vzz, pota, potb);
            if CONSTEXPR (do_e) {
               eptl += floatTo<ebuf_prec>(e);
               if CONSTEXPR (do_a) {
                  if (e != 0)
                     neptl += 1;
               }
            }
            if CONSTEXPR (do_v) {
               veptlxx += floatTo<vbuf_prec>(vxx);
               veptlyx += floatTo<vbuf_prec>(vyx);
               veptlzx += floatTo<vbuf_prec>(vzx);
               veptlyy += floatTo<vbuf_prec>(vyy);
               veptlzy += floatTo<vbuf_prec>(vzy);
               veptlzz += floatTo<vbuf_prec>(vzz);
            }
            if CONSTEXPR (CFLX) {
               poti[klane] += pota;
               potk += potb;
            }
         } // end if (include)

         iid = __shfl_sync(ALL_LANES, iid, ilane + 1);
      }

      if CONSTEXPR (do_g) {
         atomic_add(frcxi[threadIdx.x], gx, i);
         atomic_add(frcyi[threadIdx.x], gy, i);
         atomic_add(frczi[threadIdx.x], gz, i);
         atomic_add(ufld0i[threadIdx.x], &ufld[i][0]);
         atomic_add(ufld1i[threadIdx.x], &ufld[i][1]);
         atomic_add(ufld2i[threadIdx.x], &ufld[i][2]);
         atomic_add(dufld0i[threadIdx.x], &dufld[i][0]);
         atomic_add(dufld1i[threadIdx.x], &dufld[i][1]);
         atomic_add(dufld2i[threadIdx.x], &dufld[i][2]);
         atomic_add(dufld3i[threadIdx.x], &dufld[i][3]);
         atomic_add(dufld4i[threadIdx.x], &dufld[i][4]);
         atomic_add(dufld5i[threadIdx.x], &dufld[i][5]);
         if CONSTEXPR (CFLX)
            atomic_add(poti[threadIdx.x], pot, i);
         atomic_add(frcxk, gx, k);
         atomic_add(frcyk, gy, k);
         atomic_add(frczk, gz, k);
         atomic_add(ufld0k, &ufld[k][0]);
         atomic_add(ufld1k, &ufld[k][1]);
         atomic_add(ufld2k, &ufld[k][2]);
         atomic_add(dufld0k, &dufld[k][0]);
         atomic_add(dufld1k, &dufld[k][1]);
         atomic_add(dufld2k, &dufld[k][2]);
         atomic_add(dufld3k, &dufld[k][3]);
         atomic_add(dufld4k, &dufld[k][4]);
         atomic_add(dufld5k, &dufld[k][5]);
         if CONSTEXPR (CFLX)
            atomic_add(potk, pot, k);
      }
   }

   for (int iw = iwarp; iw < niak; iw += nwarp) {
      if CONSTEXPR (do_g) {
         frcxi[threadIdx.x] = 0;
         frcyi[threadIdx.x] = 0;
         frczi[threadIdx.x] = 0;
         ufld0i[threadIdx.x] = 0;
         ufld1i[threadIdx.x] = 0;
         ufld2i[threadIdx.x] = 0;
         dufld0i[threadIdx.x] = 0;
         dufld1i[threadIdx.x] = 0;
         dufld2i[threadIdx.x] = 0;
         dufld3i[threadIdx.x] = 0;
         dufld4i[threadIdx.x] = 0;
         dufld5i[threadIdx.x] = 0;
         frcxk = 0;
         frcyk = 0;
         frczk = 0;
         ufld0k = 0;
         if CONSTEXPR (CFLX)
            poti[threadIdx.x] = 0;
         frcxk = 0;
         frcyk = 0;
         frczk = 0;
         ufld0k = 0;
         ufld1k = 0;
         ufld2k = 0;
         dufld0k = 0;
         dufld1k = 0;
         dufld2k = 0;
         dufld3k = 0;
         dufld4k = 0;
         dufld5k = 0;
         if CONSTEXPR (CFLX)
            potk = 0;
      }

      int ty = iak[iw];
      int atomi = ty * WARP_SIZE + ilane;
      int i = sorted[atomi].unsorted;
      int atomk = lst[iw * WARP_SIZE + ilane];
      int k = sorted[atomk].unsorted;
      xi[threadIdx.x] = sorted[atomi].x;
      yi[threadIdx.x] = sorted[atomi].y;
      zi[threadIdx.x] = sorted[atomi].z;
      xk = sorted[atomk].x;
      yk = sorted[atomk].y;
      zk = sorted[atomk].z;

      ci[threadIdx.x] = rpole[i][MPL_PME_0];
      dix[threadIdx.x] = rpole[i][MPL_PME_X];
      diy[threadIdx.x] = rpole[i][MPL_PME_Y];
      diz[threadIdx.x] = rpole[i][MPL_PME_Z];
      qixx[threadIdx.x] = rpole[i][MPL_PME_XX];
      qixy[threadIdx.x] = rpole[i][MPL_PME_XY];
      qixz[threadIdx.x] = rpole[i][MPL_PME_XZ];
      qiyy[threadIdx.x] = rpole[i][MPL_PME_YY];
      qiyz[threadIdx.x] = rpole[i][MPL_PME_YZ];
      qizz[threadIdx.x] = rpole[i][MPL_PME_ZZ];
      uix[threadIdx.x] = uind[i][0];
      uiy[threadIdx.x] = uind[i][1];
      uiz[threadIdx.x] = uind[i][2];
      pdi[threadIdx.x] = pdamp[i];
      pti[threadIdx.x] = thole[i];
      ddi[threadIdx.x] = dirdamp[i];
      ck = rpole[k][MPL_PME_0];
      dkx = rpole[k][MPL_PME_X];
      dky = rpole[k][MPL_PME_Y];
      dkz = rpole[k][MPL_PME_Z];
      qkxx = rpole[k][MPL_PME_XX];
      qkxy = rpole[k][MPL_PME_XY];
      qkxz = rpole[k][MPL_PME_XZ];
      qkyy = rpole[k][MPL_PME_YY];
      qkyz = rpole[k][MPL_PME_YZ];
      qkzz = rpole[k][MPL_PME_ZZ];
      ukx = uind[k][0];
      uky = uind[k][1];
      ukz = uind[k][2];
      pdk = pdamp[k];
      ptk = thole[k];
      ddk = dirdamp[k];

      for (int j = 0; j < WARP_SIZE; ++j) {
         int srclane = (ilane + j) & (WARP_SIZE - 1);
         int klane = srclane + threadIdx.x - ilane;
         bool incl = atomk > 0;
         real xr = xk - xi[klane];
         real yr = yk - yi[klane];
         real zr = zk - zi[klane];
         real r2 = image2(xr, yr, zr);
         if (r2 <= off * off and incl) {
            real e, vxx, vyx, vzx, vyy, vzy, vzz;
            real pota, potb;
            pair_polar_aplus_v2<Ver, ETYP, CFLX>(                                           //
               r2, xr, yr, zr, 1, 1, ci[klane], dix[klane], diy[klane], diz[klane], qixx[klane], //
               qixy[klane], qixz[klane], qiyy[klane], qiyz[klane], qizz[klane],                  //
               uix[klane], uiy[klane], uiz[klane],                                               //
               pdi[klane], pti[klane], ddi[klane],                                               //
               ck, dkx, dky, dkz, qkxx, qkxy, qkxz, qkyy, qkyz, qkzz, ukx,                       //
               uky, ukz, pdk, ptk, ddk,                                                          //
               f, aewald,                                                                        //
               frcxi[klane], frcyi[klane], frczi[klane], frcxk, frcyk, frczk, ufld0i[klane],
               ufld1i[klane], ufld2i[klane], ufld0k, ufld1k,
               ufld2k, //
               dufld0i[klane], dufld1i[klane], dufld2i[klane], dufld3i[klane], dufld4i[klane],
               dufld5i[klane], dufld0k, dufld1k, dufld2k, dufld3k, dufld4k, dufld5k, //
               e, vxx, vyx, vzx, vyy, vzy, vzz, pota, potb);
            if CONSTEXPR (do_e) {
               eptl += floatTo<ebuf_prec>(e);
               if CONSTEXPR (do_a) {
                  if (e != 0)
                     neptl += 1;
               }
            }
            if CONSTEXPR (do_v) {
               veptlxx += floatTo<vbuf_prec>(vxx);
               veptlyx += floatTo<vbuf_prec>(vyx);
               veptlzx += floatTo<vbuf_prec>(vzx);
               veptlyy += floatTo<vbuf_prec>(vyy);
               veptlzy += floatTo<vbuf_prec>(vzy);
               veptlzz += floatTo<vbuf_prec>(vzz);
            }
            if CONSTEXPR (CFLX) {
               poti[klane] += pota;
               potk += potb;
            }
         } // end if (include)
      }

      if CONSTEXPR (do_g) {
         atomic_add(frcxi[threadIdx.x], gx, i);
         atomic_add(frcyi[threadIdx.x], gy, i);
         atomic_add(frczi[threadIdx.x], gz, i);
         atomic_add(ufld0i[threadIdx.x], &ufld[i][0]);
         atomic_add(ufld1i[threadIdx.x], &ufld[i][1]);
         atomic_add(ufld2i[threadIdx.x], &ufld[i][2]);
         atomic_add(dufld0i[threadIdx.x], &dufld[i][0]);
         atomic_add(dufld1i[threadIdx.x], &dufld[i][1]);
         atomic_add(dufld2i[threadIdx.x], &dufld[i][2]);
         atomic_add(dufld3i[threadIdx.x], &dufld[i][3]);
         atomic_add(dufld4i[threadIdx.x], &dufld[i][4]);
         atomic_add(dufld5i[threadIdx.x], &dufld[i][5]);
         if CONSTEXPR (CFLX)
            atomic_add(poti[threadIdx.x], pot, i);
         atomic_add(frcxk, gx, k);
         atomic_add(frcyk, gy, k);
         atomic_add(frczk, gz, k);
         atomic_add(ufld0k, &ufld[k][0]);
         atomic_add(ufld1k, &ufld[k][1]);
         atomic_add(ufld2k, &ufld[k][2]);
         atomic_add(dufld0k, &dufld[k][0]);
         atomic_add(dufld1k, &dufld[k][1]);
         atomic_add(dufld2k, &dufld[k][2]);
         atomic_add(dufld3k, &dufld[k][3]);
         atomic_add(dufld4k, &dufld[k][4]);
         atomic_add(dufld5k, &dufld[k][5]);
         if CONSTEXPR (CFLX)
            atomic_add(potk, pot, k);
      }
   }

   if CONSTEXPR (do_a) {
      atomic_add(neptl, nep, ithread);
   }
   if CONSTEXPR (do_e) {
      atomic_add(eptl, ep, ithread);
   }
   if CONSTEXPR (do_v) {
      atomic_add(veptlxx, veptlyx, veptlzx, veptlyy, veptlzy, veptlzz, vep, ithread);
   }
}

template <class Ver, class ETYP, int CFLX>
static void epolarAplus_cu(const real (*uind)[3])
{
   constexpr bool do_g = Ver::g;

   const auto& st = *mspatial_v2_unit;
   real off;
   if CONSTEXPR (eq<ETYP, EWALD>())
      off = switchOff(Switch::EWALD);
   else
      off = switchOff(Switch::MPOLE);

   const real f = 0.5f * electric / dielec;
   real aewald = 0;
   if CONSTEXPR (eq<ETYP, EWALD>()) {
      PMEUnit pu = ppme_unit;
      aewald = pu->aewald;
   }

   if CONSTEXPR (do_g) {
      darray::zero(g::q0, n, ufld, dufld);
   }
   int ngrid = gpuGridSize(BLOCK_DIM);
   epolarAplus_cu1<Ver, ETYP, CFLX><<<ngrid, BLOCK_DIM, 0, g::s0>>>(st.n, TINKER_IMAGE_ARGS, nep,
      ep, vir_ep, depx, depy, depz, off, st.si2.bit0, nmdpuexclude, mdpuexclude, mdpuexclude_scale,
      st.x, st.y, st.z, st.sorted, st.nakpl, st.iakpl, st.niak, st.iak, st.lst, ufld, dufld, uind,
      pot, rpole, pdamp, thole, dirdamp, aewald, f);

   // torque
   if CONSTEXPR (do_g) {
      launch_k1s(g::s0, n, epolarTorque_cu, //
         trqx, trqy, trqz, n, rpole, ufld, dufld);
   }
}

void epolarAplusNonEwald_cu(int vers, int use_cf, const real (*uind)[3])
{
   if (use_cf) {
      if (vers == calc::v0) {
         epolarAplus_cu<calc::V0, NON_EWALD, 1>(uind);
      } else if (vers == calc::v1) {
         epolarAplus_cu<calc::V1, NON_EWALD, 1>(uind);
      } else if (vers == calc::v3) {
         epolarAplus_cu<calc::V3, NON_EWALD, 1>(uind);
      } else if (vers == calc::v4) {
         epolarAplus_cu<calc::V4, NON_EWALD, 1>(uind);
      } else if (vers == calc::v5) {
         epolarAplus_cu<calc::V5, NON_EWALD, 1>(uind);
      } else if (vers == calc::v6) {
         epolarAplus_cu<calc::V6, NON_EWALD, 1>(uind);
      }
   } else {
      if (vers == calc::v0) {
         epolarAplus_cu<calc::V0, NON_EWALD, 0>(uind);
      } else if (vers == calc::v1) {
         epolarAplus_cu<calc::V1, NON_EWALD, 0>(uind);
      } else if (vers == calc::v3) {
         epolarAplus_cu<calc::V3, NON_EWALD, 0>(uind);
      } else if (vers == calc::v4) {
         epolarAplus_cu<calc::V4, NON_EWALD, 0>(uind);
      } else if (vers == calc::v5) {
         epolarAplus_cu<calc::V5, NON_EWALD, 0>(uind);
      } else if (vers == calc::v6) {
         epolarAplus_cu<calc::V6, NON_EWALD, 0>(uind);
      }
   }
}

void epolarAplusEwaldReal_cu(int vers, int use_cf, const real (*uind)[3])
{
   if (use_cf) {
      if (vers == calc::v0) {
         epolarAplus_cu<calc::V0, EWALD, 1>(uind);
         // assert(false && "CFLX must compute gradient.");
      } else if (vers == calc::v1) {
         epolarAplus_cu<calc::V1, EWALD, 1>(uind);
      } else if (vers == calc::v3) {
         epolarAplus_cu<calc::V3, EWALD, 1>(uind);
         // assert(false && "CFLX must compute gradient.");
      } else if (vers == calc::v4) {
         epolarAplus_cu<calc::V4, EWALD, 1>(uind);
      } else if (vers == calc::v5) {
         epolarAplus_cu<calc::V5, EWALD, 1>(uind);
      } else if (vers == calc::v6) {
         epolarAplus_cu<calc::V6, EWALD, 1>(uind);
      }
   } else {
      if (vers == calc::v0) {
         epolarAplus_cu<calc::V0, EWALD, 0>(uind);
      } else if (vers == calc::v1) {
         epolarAplus_cu<calc::V1, EWALD, 0>(uind);
      } else if (vers == calc::v3) {
         epolarAplus_cu<calc::V3, EWALD, 0>(uind);
      } else if (vers == calc::v4) {
         epolarAplus_cu<calc::V4, EWALD, 0>(uind);
      } else if (vers == calc::v5) {
         epolarAplus_cu<calc::V5, EWALD, 0>(uind);
      } else if (vers == calc::v6) {
         epolarAplus_cu<calc::V6, EWALD, 0>(uind);
      }
   }
}
}
