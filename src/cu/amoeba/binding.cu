#include "ff/amoebacumod.h"
#include "ff/amoebamod.h"
#include "ff/potent.h"
#include "tool/cudalib.h"
#include "tool/darray.h"
#include "tool/error.h"

namespace tinker {
static hipMemcpyKind h2d = hipMemcpyHostToDevice;

void mpoleDataBinding_cu(RcOp op)
{
   if (op & RcOp::ALLOC) {
      void *k1, *k2, *k3;
      check_rt(hipGetSymbolAddress(&k1, HIP_SYMBOL((const void*))&d::zaxis));
      check_rt(hipGetSymbolAddress(&k2, HIP_SYMBOL((const void*))&d::pole));
      check_rt(hipGetSymbolAddress(&k3, HIP_SYMBOL((const void*))&d::rpole));
      check_rt(hipMemcpyAsync(k1, &zaxis, sizeof(void*), h2d, g::s0));
      check_rt(hipMemcpyAsync(k2, &pole, sizeof(void*), h2d, g::s0));
      check_rt(hipMemcpyAsync(k3, &rpole, sizeof(void*), h2d, g::s0));

      check_rt(hipStreamSynchronize(g::s0));
   }
}

void epolarDataBinding_cu(RcOp op)
{
   if (op & RcOp::ALLOC) {
      void *p1, *p2, *p3;
      check_rt(hipGetSymbolAddress(&p1, HIP_SYMBOL((const void*))&d::njpolar));
      check_rt(hipGetSymbolAddress(&p2, HIP_SYMBOL((const void*))&d::jpolar));
      check_rt(hipGetSymbolAddress(&p3, HIP_SYMBOL((const void*))&d::thlval));
      check_rt(hipMemcpyAsync(p1, &njpolar, sizeof(int), h2d, g::s0));
      check_rt(hipMemcpyAsync(p2, &jpolar, sizeof(void*), h2d, g::s0));
      check_rt(hipMemcpyAsync(p3, &thlval, sizeof(void*), h2d, g::s0));

      void *p4, *p5, *p6;
      check_rt(hipGetSymbolAddress(&p4, HIP_SYMBOL((const void*))&d::polarity));
      check_rt(hipGetSymbolAddress(&p5, HIP_SYMBOL((const void*))&d::thole));
      check_rt(hipGetSymbolAddress(&p6, HIP_SYMBOL((const void*))&d::pdamp));
      check_rt(hipMemcpyAsync(p4, &polarity, sizeof(void*), h2d, g::s0));
      check_rt(hipMemcpyAsync(p5, &thole, sizeof(void*), h2d, g::s0));
      check_rt(hipMemcpyAsync(p6, &pdamp, sizeof(void*), h2d, g::s0));

      void* p7;
      check_rt(hipGetSymbolAddress(&p7, HIP_SYMBOL((const void*))&d::polarity_inv));
      check_rt(hipMemcpyAsync(p7, &polarity_inv, sizeof(void*), h2d, g::s0));

      check_rt(hipStreamSynchronize(g::s0));
   }
}
}
