#include "hip/hip_runtime.h"
#include "box.h"
#include "cu_launch.h"
#include "mathfunc.cuh"
#include "md.h"
#include "nblist.h"
#include "seq_image.h"
#include "seq_spatial_box.h"
#include "spatial.h"
#include "thrust_cache.h"
#include <thrust/extrema.h>
#include <thrust/remove.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/transform_reduce.h>
#include <thrust/transform_scan.h>


TINKER_NAMESPACE_BEGIN
struct POPC
{
   __device__
   int operator()(int flag)
   {
      return __popc(flag);
   }
};


struct Int32
{
   long4 lx, ly, lz, lw;


   __device__
   static bool is_long4_zero(const long4& l)
   {
      return l.x == 0 && l.y == 0 && l.z == 0 && l.w == 0;
   }


   __device__
   static bool is_zero(const Int32& i32)
   {
      return is_long4_zero(i32.lx) && is_long4_zero(i32.ly) &&
         is_long4_zero(i32.lz) && is_long4_zero(i32.lw);
   }
};


struct IntInt32Pair
{
   struct Int32IsZero
   {
      __device__
      bool operator()(const thrust::tuple<int, Int32>& t)
      {
         return Int32::is_zero(thrust::get<1>(t));
      }
   };
};


__device__
bool nearby_box0(int boxj, int px, int py, int pz, const Box* restrict box,
                 real cutbuf)
{
   int dimx = 1 << px;
   int dimy = 1 << py;
   int dimz = 1 << pz;
   int ix, iy, iz;
   box_to_ixyz(ix, iy, iz, px, py, pz, boxj);

   // (a, b): (-0.5, a+1/dim)
   // (c, d): (a+ix/dim, c+1/dim)
   // da = a+(ix+1)/dim - a = (ix+1)/dim
   // cb = a+ix/dim - a-1/dim = (ix-1)/dim
   // min(imagen(da), imagen(cb))
   real rx = 0;
   real ry = 0;
   real rz = 0;
   if (2 <= ix && ix <= dimx - 2) {
      real da = ((real)ix + 1) / dimx;
      real cb = ((real)ix - 1) / dimx;
      da -= REAL_FLOOR(da + 0.5f);
      cb -= REAL_FLOOR(cb + 0.5f);
      rx = REAL_MIN(REAL_ABS(da), REAL_ABS(cb));
   }
   if (2 <= iy && iy <= dimy - 2) {
      real da = ((real)iy + 1) / dimy;
      real cb = ((real)iy - 1) / dimy;
      da -= REAL_FLOOR(da + 0.5f);
      cb -= REAL_FLOOR(cb + 0.5f);
      ry = REAL_MIN(REAL_ABS(da), REAL_ABS(cb));
   }
   if (2 <= iz && iz <= dimz - 2) {
      real da = ((real)iz + 1) / dimz;
      real cb = ((real)iz - 1) / dimz;
      da -= REAL_FLOOR(da + 0.5f);
      cb -= REAL_FLOOR(cb + 0.5f);
      rz = REAL_MIN(REAL_ABS(da), REAL_ABS(cb));
   }
   frac_image(rx, ry, rz, box);
   real r2 = rx * rx + ry * ry + rz * rz;
   if (r2 <= cutbuf * cutbuf)
      return true;
   else
      return false;
}


__device__
inline int offset_box(int nx, int ny, int nz, int ix1, int iy1, int iz1,
                      int offset)
{
   int dimx = (1 << nx);
   int dimy = (1 << ny);
   int dimz = (1 << nz);
   int ix, iy, iz;
   box_to_ixyz(ix, iy, iz, nx, ny, nz, offset);
   ix = (ix + ix1) & (dimx - 1);
   iy = (iy + iy1) & (dimy - 1);
   iz = (iz + iz1) & (dimz - 1);
   int id = ixyz_to_box(nx, ny, nz, ix, iy, iz);
   return id;
}


__global__
void spatial_bc(Spatial* restrict sp, const real* restrict x,
                const real* restrict y, const real* restrict z,
                const Box* restrict box, real cutbuf)
{
   int n = sp->n;
   int px = sp->px;
   int py = sp->py;
   int pz = sp->pz;
   auto* restrict sorted = sp->sorted;
   auto* restrict boxnum = sp->boxnum;
   auto* restrict nax = sp->ax_scan + 1; // D.2
   for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < n;
        i += blockDim.x * gridDim.x) {
      real xr = x[i];
      real yr = y[i];
      real zr = z[i];
      real fx, fy, fz;
      frac(fx, fy, fz, xr, yr, zr, box);
      sorted[i].x = xr;       // B.2
      sorted[i].y = yr;       // B.2
      sorted[i].z = zr;       // B.2
      sorted[i].unsorted = i; // B.2
      int id = frac_to_box(px, py, pz, fx, fy, fz);
      boxnum[i] = id;         // B.3
      atomicAdd(&nax[id], 1); // B.4
   }


   int nx = sp->nx;
   auto* restrict nearby = sp->nearby;
   for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < nx;
        i += blockDim.x * gridDim.x) {
      if (nearby_box0(i, px, py, pz, box, cutbuf))
         nearby[i] = i; // C.1 (close enough)
      else
         nearby[i] = -1; // C.1 (otherwise)
   }
}


__global__
void spatial_e(Spatial* restrict sp)
{
   const int ithread = threadIdx.x + blockIdx.x * blockDim.x;
   const int iwarp = ithread / WARP_SIZE;
   const int nwarp = blockDim.x * gridDim.x / WARP_SIZE;
   const int ilane = threadIdx.x & (WARP_SIZE - 1);
   const int prevlane = (ilane + WARP_SIZE - 1) & (WARP_SIZE - 1); // E.2


   const int n = sp->n;
   const int nak = sp->nak;
   const auto* restrict boxnum = sp->boxnum;
   auto* restrict xakf = sp->xakf;
   for (int iw = iwarp; iw < nak; iw += nwarp) {
      int atomi = iw * WARP_SIZE + ilane;
      int id1 = ((atomi < n) ? boxnum[atomi] : boxnum[n - 1]); // E.3
      int id0 = __shfl_sync(ALL_LANES, id1, prevlane);         // E.5
      int diff = (id0 == id1 ? 0 : 1);                         // E.1
      int flag = __ballot_sync(ALL_LANES, diff);               // E.6
      if (ilane == 0)
         xakf[iw] = (flag == 0 ? 1 : flag); // E.4
   }
}


__global__
void spatial_ghi(Spatial* restrict sp, const Box* restrict box, real cutbuf)
{
   const int ithread = threadIdx.x + blockIdx.x * blockDim.x;
   const int iwarp = ithread / WARP_SIZE;
   const int nwarp = blockDim.x * gridDim.x / WARP_SIZE;
   const int ilane = threadIdx.x & (WARP_SIZE - 1);

   const int nak = sp->nak;
   const int px = sp->px;
   const int py = sp->py;
   const int pz = sp->pz;
   const int nxk = sp->nxk;
   const int near = sp->near;

   const auto* restrict boxnum = sp->boxnum;
   const auto* restrict xakf = sp->xakf;
   const auto* restrict xakf_scan = sp->xakf_scan;
   const auto* restrict nearby = sp->nearby;
   const auto* restrict begin = sp->ax_scan; // D.4
   const auto* restrict end = begin + 1;     // D.4

   auto* restrict iak = sp->iak;
   auto* restrict lst = sp->lst;
   auto* restrict naak = sp->naak;
   auto* restrict xkf = sp->xkf;

   for (int iw = iwarp; iw < nak; iw += nwarp) {
      int offset = xakf_scan[iw]; // F.5
      int flag = xakf[iw];        // E.7
      int nbox = __popc(flag);    // E.7

      auto* restrict iakbuf = iak + near * offset;             // G.4
      auto* restrict lstbuf = lst + near * offset * WARP_SIZE; // G.5
      auto* restrict ixkf = xkf + iw * nxk;                    // H.2
      const int atom_block_min = iw * WARP_SIZE;               // H.4
      for (int j = ilane; j < nbox * near; j += WARP_SIZE) {
         iakbuf[j] = iw;    // G.4
         int i0 = j / near; // the i-th least significant bit is i0 + 1
         int pos = ffsn(flag, i0 + 1) - 1;        // E.8
         int ibox = boxnum[iw * WARP_SIZE + pos]; // E.8
         int ix1, iy1, iz1;
         box_to_ixyz(ix1, iy1, iz1, px, py, pz, ibox);
         int j0 = nearby[j - i0 * near];
         int jbox = offset_box(px, py, pz, ix1, iy1, iz1, j0);
         // the (jbox%32)-th bit of the (jbox/32) flag will be set to 1
         int ii = jbox / WARP_SIZE;
         int jj = jbox & (WARP_SIZE - 1);
         int oldflag = atomicOr(&ixkf[ii], 1 << jj); // H.3
         // the atomicOr() will return the old value;
         // code in the following if body will only run
         // when the bit(ii,jj) gets set for the first time
         if ((oldflag & (1 << jj)) == 0) {
            // copy atoms in jbox to lstbuf
            int begin_i = begin[jbox];
            begin_i = max(atom_block_min + 1, begin_i);        // H.4
            int len = end[jbox] - begin_i;                     // H.5
            int start_pos = atomicAdd(&naak[iw], max(0, len)); // H.6
            // atomicAdd() will return the old value;
            // skip the loop if len is less than 1
            for (int kk = 0; kk < len; ++kk) {
               lstbuf[start_pos + kk] = begin_i + kk; // H.4
            }
         }
      }
   }


   real off2 = cutbuf * cutbuf;
   const int n = sp->n;
   const auto* restrict sorted = sp->sorted;
   for (int iw = iwarp; iw < nak; iw += nwarp) {
      int offset = xakf_scan[iw];
      const auto* restrict iakbuf = iak + near * offset;
      auto* restrict lstbuf = lst + near * offset * WARP_SIZE;
      int naak_coarse = naak[iw]; // I.1


      int start_pos = 0;
      int atomi;
      atomi = min(iakbuf[0] * WARP_SIZE + ilane, n - 1); // I.4
      real xi = sorted[atomi].x;
      real yi = sorted[atomi].y;
      real zi = sorted[atomi].z;
      int shatomk;
      real shx, shy, shz;
      int idx_max = (naak_coarse + WARP_SIZE - 1) / WARP_SIZE;
      idx_max *= WARP_SIZE; // I.2a
      for (int idx = ilane; idx < idx_max; idx += WARP_SIZE) {
         shatomk = lstbuf[idx]; // I.2b
         shx = sorted[shatomk].x;
         shy = sorted[shatomk].y;
         shz = sorted[shatomk].z;
         lstbuf[idx] = 0; // I.3


         int jflag = 0;
         for (int j = 0; j < WARP_SIZE; ++j) {
            int srclane = j;
            int atomk = __shfl_sync(ALL_LANES, shatomk, srclane);
            real xr = xi - __shfl_sync(ALL_LANES, shx, srclane);
            real yr = yi - __shfl_sync(ALL_LANES, shy, srclane);
            real zr = zi - __shfl_sync(ALL_LANES, shz, srclane);
            imagen(xr, yr, zr, box);
            real rik2 = xr * xr + yr * yr + zr * zr;
            int ilane_incl_j = (atomi < atomk && rik2 <= off2) ? 1 : 0; // I.5
            int incl_j = __ballot_sync(ALL_LANES, ilane_incl_j);
            if (incl_j)
               jflag |= (1 << j); // I.5
         }


         int njbit = __popc(jflag);
         int jth = ffsn(jflag, ilane + 1) - 1;
         int atomnb = __shfl_sync(ALL_LANES, shatomk, jth); // I.6a
         if (ilane < njbit)
            lstbuf[start_pos + ilane] = atomnb; // I.6b
         start_pos += njbit;
      }
   }
}
TINKER_NAMESPACE_END


TINKER_NAMESPACE_BEGIN
void spatial_data_init_cu(SpatialUnit u, NBListUnit nu)
{
   const real cutbuf = nu->cutoff + nu->buffer;
   const int& nak = u->nak;
   const int padded = nak * Spatial::BLOCK;
   int& px = u->px;
   int& py = u->py;
   int& pz = u->pz;
   int& nx = u->nx;
   int& nxk = u->nxk;
   int& near = u->near;
   int& xak_sum = u->xak_sum;
   int& xak_sum_cap = u->xak_sum_cap;
   int& niak = u->niak;


   auto*& sorted = u->sorted;
   auto*& boxnum = u->boxnum;
   auto*& naak = u->naak;
   auto*& xakf = u->xakf;
   auto*& xakf_scan = u->xakf_scan;
   auto*& nearby = u->nearby;
   auto*& ax_scan = u->ax_scan;
   auto*& xkf = u->xkf;


   // auto policy = thrust::device;
   auto policy = thrust::cuda::par(thrust_cache);


   // B.1 D.1
   device_array::zero(nx + 1, ax_scan);
   // B.2 B.3 B.4 C.1
   const auto* lx = nu->x;
   const auto* ly = nu->y;
   const auto* lz = nu->z;
   launch_kernel1(n, spatial_bc, u.deviceptr(), lx, ly, lz, box, cutbuf);
   // find max(nax) and compare to Spatial::BLOCK
   // ax_scan[0] == 0 can never be the maximum
   int level = 1 + floor_log2(nak - 1);
   int mnax;
   const int* mnaxptr = thrust::max_element(policy, ax_scan, ax_scan + 1 + nx);
   device_array::copyout(1, &mnax, mnaxptr);
   while (mnax > Spatial::BLOCK) {
      device_array::deallocate(nearby, ax_scan, xkf);

      int scale = (mnax - 1) / Spatial::BLOCK;
      // mnax / mnax-1 / scale / 2^p / p
      // 33   / 32     / 1     / 2   / 1
      // 64   / 63     / 1     / 2   / 1
      // 65   / 64     / 2     / 4   / 2
      // 128  / 127    / 3     / 4   / 2
      // 129  / 128    / 4     / 8   / 3
      int p = 1 + floor_log2(scale);
      level += p;
      px = (level + 2) / 3;
      py = (level + 1) / 3;
      pz = (level + 0) / 3;
      nx = ct::pow2(px + py + pz);
      nxk = (nx + Spatial::BLOCK - 1) / Spatial::BLOCK;

      device_array::allocate(nx, &nearby);
      device_array::allocate(nx + 1, &ax_scan);
      device_array::allocate(nak * nxk, &xkf);

      u.update_deviceptr(*u);

      device_array::zero(nx + 1, ax_scan);
      launch_kernel1(n, spatial_bc, u.deviceptr(), lx, ly, lz, box, cutbuf);
      mnaxptr = thrust::max_element(policy, ax_scan, ax_scan + 1 + nx);
      device_array::copyout(1, &mnax, mnaxptr);
   }
   // B.5
   thrust::stable_sort_by_key(policy, boxnum, boxnum + n, sorted);
   // C.2
   int* nearby_end = thrust::remove(policy, nearby, nearby + nx, -1);
   // C.3
   near = nearby_end - nearby;
   // D.2
   int* nax = ax_scan + 1;
   // D.3
   thrust::inclusive_scan(policy, nax, nax + nx, nax);


   // E
   launch_kernel1(padded, spatial_e, u.deviceptr());
   // F.1
   xak_sum = thrust::transform_reduce(policy, xakf, xakf + nak, POPC(), 0,
                                      thrust::plus<int>());
   // F.2
   thrust::transform_exclusive_scan(policy, xakf, xakf + nak, xakf_scan, POPC(),
                                    0, thrust::plus<int>());
   if (xak_sum > xak_sum_cap) {
      device_array::deallocate(u->iak, u->lst);
      xak_sum_cap = xak_sum;
      size_t iak_size = near * xak_sum;            // F.3
      size_t lst_size = iak_size * Spatial::BLOCK; // F.4
      // allocate iak and lst together
      device_array::allocate(iak_size + lst_size, &u->lst);
      u->iak = u->lst + lst_size;
   }
   // must update the device pointer to apply the changes in xak_sum
   u.update_deviceptr(*u);


   device_array::zero(near * xak_sum * Spatial::BLOCK, u->lst); // G.6
   device_array::zero(nak, naak);                               // H.1
   device_array::zero(nak * nxk, xkf);                          // H.1
   launch_kernel1(padded, spatial_ghi, u.deviceptr(), box, cutbuf);


   Int32* lst32 = (Int32*)u->lst;
   auto tup_begin =
      thrust::make_zip_iterator(thrust::make_tuple(u->iak, lst32));
   auto tup_end = thrust::make_zip_iterator(
      thrust::make_tuple(u->iak + near * xak_sum, lst32 + near * xak_sum));
   auto end2 = thrust::remove_if(policy, tup_begin, tup_end,
                                 IntInt32Pair::Int32IsZero());  // G.7
   u->niak = thrust::get<1>(end2.get_iterator_tuple()) - lst32; // G.7
   assert((thrust::get<0>(end2.get_iterator_tuple()) - u->iak) == u->niak);
   u.update_deviceptr(*u);
}
TINKER_NAMESPACE_END
