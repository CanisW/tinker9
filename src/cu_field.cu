#include "hip/hip_runtime.h"
#include "add.cuh"
#include "e_polar.h"
#include "launch.cuh"
#include "md.h"
#include "pme.h"
#include "seq_image.h"
#include "seq_pair_field.h"
#include "spatial.h"
#include "switch.h"


TINKER_NAMESPACE_BEGIN
#define DFIELD_ARGS                                                            \
   real(*restrict field)[3], real(*restrict fieldp)[3],                        \
      const real *restrict thole, const real *restrict pdamp,                  \
      const real(*restrict rpole)[10], const Box *restrict box, real off


template <elec_t ETYP>
__global__
void dfield_cu1(DFIELD_ARGS, const Spatial* restrict sp, real aewald)
{
   const int ithread = threadIdx.x + blockIdx.x * blockDim.x;
   const int iwarp = ithread / WARP_SIZE;
   const int nwarp = blockDim.x * gridDim.x / WARP_SIZE;
   const int ilane = threadIdx.x & (WARP_SIZE - 1);


   real gxi, gyi, gzi, txi, tyi, tzi;
   __shared__ real gxk[BLOCK_DIM], gyk[BLOCK_DIM], gzk[BLOCK_DIM],
      txk[BLOCK_DIM], tyk[BLOCK_DIM], tzk[BLOCK_DIM];


   const real off2 = off * off;
   const int n = sp->n;
   const int niak = sp->niak;
   const auto* restrict sorted = sp->sorted;
   const auto* restrict iak = sp->iak;
   const auto* restrict lst = sp->lst;
   for (int iw = iwarp; iw < niak; iw += nwarp) {
      gxi = 0;
      gyi = 0;
      gzi = 0;
      txi = 0;
      tyi = 0;
      tzi = 0;
      gxk[threadIdx.x] = 0;
      gyk[threadIdx.x] = 0;
      gzk[threadIdx.x] = 0;
      txk[threadIdx.x] = 0;
      tyk[threadIdx.x] = 0;
      tzk[threadIdx.x] = 0;


      int atomi;
      atomi = min(iak[iw] * WARP_SIZE + ilane, n - 1);
      real xi = sorted[atomi].x;
      real yi = sorted[atomi].y;
      real zi = sorted[atomi].z;
      int i = sorted[atomi].unsorted;
      real ci = rpole[i][mpl_pme_0];
      real dix = rpole[i][mpl_pme_x];
      real diy = rpole[i][mpl_pme_y];
      real diz = rpole[i][mpl_pme_z];
      real qixx = rpole[i][mpl_pme_xx];
      real qixy = rpole[i][mpl_pme_xy];
      real qixz = rpole[i][mpl_pme_xz];
      real qiyy = rpole[i][mpl_pme_yy];
      real qiyz = rpole[i][mpl_pme_yz];
      real qizz = rpole[i][mpl_pme_zz];
      real pdi = pdamp[i];
      real pti = thole[i];


      int shatomk;
      shatomk = lst[iw * WARP_SIZE + ilane];
      real shx = sorted[shatomk].x;
      real shy = sorted[shatomk].y;
      real shz = sorted[shatomk].z;
      int shk = sorted[shatomk].unsorted;
      real shck = rpole[shk][mpl_pme_0];
      real shdkx = rpole[shk][mpl_pme_x];
      real shdky = rpole[shk][mpl_pme_y];
      real shdkz = rpole[shk][mpl_pme_z];
      real shqkxx = rpole[shk][mpl_pme_xx];
      real shqkxy = rpole[shk][mpl_pme_xy];
      real shqkxz = rpole[shk][mpl_pme_xz];
      real shqkyy = rpole[shk][mpl_pme_yy];
      real shqkyz = rpole[shk][mpl_pme_yz];
      real shqkzz = rpole[shk][mpl_pme_zz];
      real shpdk = pdamp[shk];
      real shptk = thole[shk];


      for (int j = 0; j < WARP_SIZE; ++j) {
         int srclane = (ilane + j) & (WARP_SIZE - 1);
         int atomk = __shfl_sync(ALL_LANES, shatomk, srclane);
         real xr = __shfl_sync(ALL_LANES, shx, srclane) - xi;
         real yr = __shfl_sync(ALL_LANES, shy, srclane) - yi;
         real zr = __shfl_sync(ALL_LANES, shz, srclane) - zi;
         int k = __shfl_sync(ALL_LANES, shk, srclane);
         real ck = __shfl_sync(ALL_LANES, shck, srclane);
         real dkx = __shfl_sync(ALL_LANES, shdkx, srclane);
         real dky = __shfl_sync(ALL_LANES, shdky, srclane);
         real dkz = __shfl_sync(ALL_LANES, shdkz, srclane);
         real qkxx = __shfl_sync(ALL_LANES, shqkxx, srclane);
         real qkxy = __shfl_sync(ALL_LANES, shqkxy, srclane);
         real qkxz = __shfl_sync(ALL_LANES, shqkxz, srclane);
         real qkyy = __shfl_sync(ALL_LANES, shqkyy, srclane);
         real qkyz = __shfl_sync(ALL_LANES, shqkyz, srclane);
         real qkzz = __shfl_sync(ALL_LANES, shqkzz, srclane);
         real pdk = __shfl_sync(ALL_LANES, shpdk, srclane);
         real ptk = __shfl_sync(ALL_LANES, shptk, srclane);


         PairField pairf;
         zero(pairf);


         image(xr, yr, zr, box);
         real r2 = xr * xr + yr * yr + zr * zr;
         if (atomi < atomk && r2 <= off2) {
            if CONSTEXPR (ETYP == elec_t::ewald) {
               pair_dfield<elec_t::ewald>(
                  r2, xr, yr, zr, 1, 1, ci, dix, diy, diz, qixx, qixy, qixz,
                  qiyy, qiyz, qizz, pdi, pti, ck, dkx, dky, dkz, qkxx, qkxy,
                  qkxz, qkyy, qkyz, qkzz, pdk, ptk, aewald, pairf);
            }
            if CONSTEXPR (ETYP == elec_t::coulomb) {
               pair_dfield<elec_t::coulomb>(
                  r2, xr, yr, zr, 1, 1, ci, dix, diy, diz, qixx, qixy, qixz,
                  qiyy, qiyz, qizz, pdi, pti, ck, dkx, dky, dkz, qkxx, qkxy,
                  qkxz, qkyy, qkyz, qkzz, pdk, ptk, 0, pairf);
            }
         } // end if (include)


         gxi += pairf.fid[0];
         gyi += pairf.fid[1];
         gzi += pairf.fid[2];
         gxk[srclane + (threadIdx.x - ilane)] += pairf.fkd[0];
         gyk[srclane + (threadIdx.x - ilane)] += pairf.fkd[1];
         gzk[srclane + (threadIdx.x - ilane)] += pairf.fkd[2];
         txi += pairf.fip[0];
         tyi += pairf.fip[1];
         tzi += pairf.fip[2];
         txk[srclane + (threadIdx.x - ilane)] += pairf.fkp[0];
         tyk[srclane + (threadIdx.x - ilane)] += pairf.fkp[1];
         tzk[srclane + (threadIdx.x - ilane)] += pairf.fkp[2];
      } // end for (j)


      atomic_add(gxi, &field[i][0]);
      atomic_add(gyi, &field[i][1]);
      atomic_add(gzi, &field[i][2]);
      atomic_add(txi, &fieldp[i][0]);
      atomic_add(tyi, &fieldp[i][1]);
      atomic_add(tzi, &fieldp[i][2]);
      atomic_add(gxk[threadIdx.x], &field[shk][0]);
      atomic_add(gyk[threadIdx.x], &field[shk][1]);
      atomic_add(gzk[threadIdx.x], &field[shk][2]);
      atomic_add(txk[threadIdx.x], &fieldp[shk][0]);
      atomic_add(tyk[threadIdx.x], &fieldp[shk][1]);
      atomic_add(tzk[threadIdx.x], &fieldp[shk][2]);
   } // end for (iw)
}


__global__
void dfield_cu2(DFIELD_ARGS, const real* restrict x, const real* restrict y,
                const real* restrict z, int ndpexclude_,
                const int (*restrict dpexclude_)[2],
                const real (*restrict dpexclude_scale_)[2])
{
   const real off2 = off * off;
   for (int ii = threadIdx.x + blockIdx.x * blockDim.x; ii < ndpexclude_;
        ii += blockDim.x * gridDim.x) {
      int i = dpexclude_[ii][0];
      int k = dpexclude_[ii][1];
      real dscale = dpexclude_scale_[ii][0];
      real pscale = dpexclude_scale_[ii][1];


      real xi = x[i];
      real yi = y[i];
      real zi = z[i];
      real ci = rpole[i][mpl_pme_0];
      real dix = rpole[i][mpl_pme_x];
      real diy = rpole[i][mpl_pme_y];
      real diz = rpole[i][mpl_pme_z];
      real qixx = rpole[i][mpl_pme_xx];
      real qixy = rpole[i][mpl_pme_xy];
      real qixz = rpole[i][mpl_pme_xz];
      real qiyy = rpole[i][mpl_pme_yy];
      real qiyz = rpole[i][mpl_pme_yz];
      real qizz = rpole[i][mpl_pme_zz];
      real pdi = pdamp[i];
      real pti = thole[i];


      real xr = x[k] - xi;
      real yr = y[k] - yi;
      real zr = z[k] - zi;


      image(xr, yr, zr, box);
      real r2 = xr * xr + yr * yr + zr * zr;
      if (r2 <= off2) {
         PairField pairf;
         pair_dfield<elec_t::coulomb>(
            r2, xr, yr, zr, dscale, pscale, ci, dix, diy, diz, qixx, qixy, qixz,
            qiyy, qiyz, qizz, pdi, pti, rpole[k][mpl_pme_0],
            rpole[k][mpl_pme_x], rpole[k][mpl_pme_y], rpole[k][mpl_pme_z],
            rpole[k][mpl_pme_xx], rpole[k][mpl_pme_xy], rpole[k][mpl_pme_xz],
            rpole[k][mpl_pme_yy], rpole[k][mpl_pme_yz], rpole[k][mpl_pme_zz],
            pdamp[k], thole[k], 0, pairf);


         atomic_add(pairf.fid[0], &field[i][0]);
         atomic_add(pairf.fid[1], &field[i][1]);
         atomic_add(pairf.fid[2], &field[i][2]);
         atomic_add(pairf.fip[0], &fieldp[i][0]);
         atomic_add(pairf.fip[1], &fieldp[i][1]);
         atomic_add(pairf.fip[2], &fieldp[i][2]);


         atomic_add(pairf.fkd[0], &field[k][0]);
         atomic_add(pairf.fkd[1], &field[k][1]);
         atomic_add(pairf.fkd[2], &field[k][2]);
         atomic_add(pairf.fkp[0], &fieldp[k][0]);
         atomic_add(pairf.fkp[1], &fieldp[k][1]);
         atomic_add(pairf.fkp[2], &fieldp[k][2]);
      } // end if (include)
   }    // end for (ii)
}


void dfield_ewald_real_cu(real (*field)[3], real (*fieldp)[3])
{
   const auto& st = *mspatial_unit;
   const real off = st.cutoff;
   const auto* sp = mspatial_unit.deviceptr();


   const PMEUnit pu = ppme_unit;
   const real aewald = pu->aewald;


   if (st.niak > 0) {
      launch_kernel1(WARP_SIZE * st.niak, dfield_cu1<elec_t::ewald>, field,
                     fieldp, thole, pdamp, rpole, box, off, sp, aewald);
   }
   if (ndpexclude_ > 0) {
      launch_kernel1(ndpexclude_, dfield_cu2, field, fieldp, thole, pdamp,
                     rpole, box, off, x, y, z, ndpexclude_, dpexclude_,
                     dpexclude_scale_);
   }
}


void dfield_coulomb_cu(real (*field)[3], real (*fieldp)[3])
{
   const auto& st = *mspatial_unit;
   const real off = st.cutoff;
   const auto* sp = mspatial_unit.deviceptr();


   device_array::zero(n, field, fieldp);
   if (st.niak > 0) {
      launch_kernel1(WARP_SIZE * st.niak, dfield_cu1<elec_t::coulomb>, field,
                     fieldp, thole, pdamp, rpole, box, off, sp, 0);
   }
   if (ndpexclude_ > 0) {
      launch_kernel1(ndpexclude_, dfield_cu2, field, fieldp, thole, pdamp,
                     rpole, box, off, x, y, z, ndpexclude_, dpexclude_,
                     dpexclude_scale_);
   }
}


#define UFIELD_ARGS                                                            \
   const real(*restrict uind)[3], const real(*restrict uinp)[3],               \
      real(*restrict field)[3], real(*restrict fieldp)[3],                     \
      const real *restrict thole, const real *restrict pdamp,                  \
      TINKER_IMAGE_PARAMS, real off2


template <elec_t ETYP>
__launch_bounds__(BLOCK_DIM) __global__
void ufield_cu1(UFIELD_ARGS, int n, const Spatial::SortedAtom* restrict sorted,
                int niak, const int* restrict iak, const int* restrict lst,
                real aewald)
{
   const int iwarp = (threadIdx.x + blockIdx.x * blockDim.x) / WARP_SIZE;
   const int nwarp = blockDim.x * gridDim.x / WARP_SIZE;
   const int ilane = threadIdx.x & (WARP_SIZE - 1);


   struct Data
   {
      real3 fkd, fkp, ukd, ukp, rk;
      real pdk, ptk;
   };
   __shared__ Data data[BLOCK_DIM];


   for (int iw = iwarp; iw < niak; iw += nwarp) {
      real3 fid = make_real3(0, 0, 0);
      real3 fip = make_real3(0, 0, 0);
      int atomi = min(iak[iw] * WARP_SIZE + ilane, n - 1);
      real3 ri = make_real3(sorted[atomi].x, sorted[atomi].y, sorted[atomi].z);
      int i = sorted[atomi].unsorted;
      real3 uid = make_real3(uind[i][0], uind[i][1], uind[i][2]);
      real3 uip = make_real3(uinp[i][0], uinp[i][1], uinp[i][2]);
      real pdi = pdamp[i];
      real pti = thole[i];


      data[threadIdx.x].fkd = make_real3(0, 0, 0);
      data[threadIdx.x].fkp = make_real3(0, 0, 0);
      int shatomk = lst[iw * WARP_SIZE + ilane];
      data[threadIdx.x].rk =
         make_real3(sorted[shatomk].x, sorted[shatomk].y, sorted[shatomk].z);
      int shk = sorted[shatomk].unsorted;
      data[threadIdx.x].ukd =
         make_real3(uind[shk][0], uind[shk][1], uind[shk][2]);
      data[threadIdx.x].ukp =
         make_real3(uinp[shk][0], uinp[shk][1], uinp[shk][2]);
      data[threadIdx.x].pdk = pdamp[shk];
      data[threadIdx.x].ptk = thole[shk];


      for (int j = 0; j < WARP_SIZE; ++j) {
         int srclane = (ilane + j) & (WARP_SIZE - 1);
         int klane = srclane + threadIdx.x - ilane;
         int atomk = __shfl_sync(ALL_LANES, shatomk, srclane);
         real3 dr = data[klane].rk - ri;


         real r2 = image2(dr.x, dr.y, dr.z);
         if (atomi < atomk && r2 <= off2) {
            if CONSTEXPR (ETYP == elec_t::ewald) {
               pair_ufield<elec_t::ewald>(
                  r2, dr.x, dr.y, dr.z, 1, uid.x, uid.y, uid.z, uip.x, uip.y,
                  uip.z, pdi, pti, data[klane].ukd.x, data[klane].ukd.y,
                  data[klane].ukd.z, data[klane].ukp.x, data[klane].ukp.y,
                  data[klane].ukp.z, data[klane].pdk, data[klane].ptk, aewald,
                  fid, fip, data[klane].fkd, data[klane].fkp);
            }
            if CONSTEXPR (ETYP == elec_t::coulomb) {
               pair_ufield<elec_t::coulomb>(
                  r2, dr.x, dr.y, dr.z, 1, uid.x, uid.y, uid.z, uip.x, uip.y,
                  uip.z, pdi, pti, data[klane].ukd.x, data[klane].ukd.y,
                  data[klane].ukd.z, data[klane].ukp.x, data[klane].ukp.y,
                  data[klane].ukp.z, data[klane].pdk, data[klane].ptk, 0, fid,
                  fip, data[klane].fkd, data[klane].fkp);
            }
         } // end if (include)
      }    // end for (j)


      atomic_add(fid.x, &field[i][0]);
      atomic_add(fid.y, &field[i][1]);
      atomic_add(fid.z, &field[i][2]);
      atomic_add(fip.x, &fieldp[i][0]);
      atomic_add(fip.y, &fieldp[i][1]);
      atomic_add(fip.z, &fieldp[i][2]);
      atomic_add(data[threadIdx.x].fkd.x, &field[shk][0]);
      atomic_add(data[threadIdx.x].fkd.y, &field[shk][1]);
      atomic_add(data[threadIdx.x].fkd.z, &field[shk][2]);
      atomic_add(data[threadIdx.x].fkp.x, &fieldp[shk][0]);
      atomic_add(data[threadIdx.x].fkp.y, &fieldp[shk][1]);
      atomic_add(data[threadIdx.x].fkp.z, &fieldp[shk][2]);
   } // end for (iw)
}


__global__
void ufield_cu2(UFIELD_ARGS, const real* restrict x, const real* restrict y,
                const real* restrict z, int nuexclude_,
                const int (*restrict uexclude_)[2],
                const real* restrict uexclude_scale_)
{
   for (int ii = threadIdx.x + blockIdx.x * blockDim.x; ii < nuexclude_;
        ii += blockDim.x * gridDim.x) {
      int i = uexclude_[ii][0];
      int k = uexclude_[ii][1];
      real uscale = uexclude_scale_[ii];


      real xi = x[i];
      real yi = y[i];
      real zi = z[i];
      real3 uid = make_real3(uind[i][0], uind[i][1], uind[i][2]);
      real3 uip = make_real3(uinp[i][0], uinp[i][1], uinp[i][2]);
      real pdi = pdamp[i];
      real pti = thole[i];


      real xr = x[k] - xi;
      real yr = y[k] - yi;
      real zr = z[k] - zi;


      real r2 = image2(xr, yr, zr);
      if (r2 <= off2) {
         real3 fid = make_real3(0, 0, 0);
         real3 fip = make_real3(0, 0, 0);
         real3 fkd = make_real3(0, 0, 0);
         real3 fkp = make_real3(0, 0, 0);
         pair_ufield<elec_t::coulomb>(
            r2, xr, yr, zr, uscale, uid.x, uid.y, uid.z, uip.x, uip.y, uip.z,
            pdi, pti, uind[k][0], uind[k][1], uind[k][2], uinp[k][0],
            uinp[k][1], uinp[k][2], pdamp[k], thole[k], 0, fid, fip, fkd, fkp);


         atomic_add(fid.x, &field[i][0]);
         atomic_add(fid.y, &field[i][1]);
         atomic_add(fid.z, &field[i][2]);
         atomic_add(fip.x, &fieldp[i][0]);
         atomic_add(fip.y, &fieldp[i][1]);
         atomic_add(fip.z, &fieldp[i][2]);


         atomic_add(fkd.x, &field[k][0]);
         atomic_add(fkd.y, &field[k][1]);
         atomic_add(fkd.z, &field[k][2]);
         atomic_add(fkp.x, &fieldp[k][0]);
         atomic_add(fkp.y, &fieldp[k][1]);
         atomic_add(fkp.z, &fieldp[k][2]);
      } // end if (include)
   }
}


void ufield_ewald_real_cu(const real (*uind)[3], const real (*uinp)[3],
                          real (*field)[3], real (*fieldp)[3])
{
   const auto& st = *mspatial_unit;
   const real off2 = st.cutoff * st.cutoff;


   const PMEUnit pu = ppme_unit;
   const real aewald = pu->aewald;

   if (st.niak > 0) {
      launch_kernel1(WARP_SIZE * st.niak, ufield_cu1<elec_t::ewald>, uind, uinp,
                     field, fieldp, thole, pdamp, TINKER_IMAGE_ARGS, off2, n,
                     st.sorted, st.niak, st.iak, st.lst, aewald);
   }
   if (nuexclude_) {
      launch_kernel1(nuexclude_, ufield_cu2, uind, uinp, field, fieldp, thole,
                     pdamp, TINKER_IMAGE_ARGS, off2, x, y, z, nuexclude_,
                     uexclude_, uexclude_scale_);
   }
}


void ufield_coulomb_cu(float const (*uind)[3], float const (*uinp)[3],
                       float (*field)[3], float (*fieldp)[3])
{
   const auto& st = *mspatial_unit;
   const real off2 = st.cutoff * st.cutoff;


   device_array::zero(n, field, fieldp);
   if (st.niak > 0) {
      launch_kernel1(WARP_SIZE * st.niak, ufield_cu1<elec_t::coulomb>, uind,
                     uinp, field, fieldp, thole, pdamp, TINKER_IMAGE_ARGS, off2,
                     n, st.sorted, st.niak, st.iak, st.lst, 0);
   }
   if (nuexclude_) {
      launch_kernel1(nuexclude_, ufield_cu2, uind, uinp, field, fieldp, thole,
                     pdamp, TINKER_IMAGE_ARGS, off2, x, y, z, nuexclude_,
                     uexclude_, uexclude_scale_);
   }
}
TINKER_NAMESPACE_END
