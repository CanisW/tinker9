#include "hip/hip_runtime.h"
#include "add.cuh"
#include "e_polar.h"
#include "launch.cuh"
#include "md.h"
#include "pme.h"
#include "seq_image.h"
#include "seq_pair_field.h"
#include "spatial.h"
#include "switch.h"


TINKER_NAMESPACE_BEGIN
#define DFIELD_ARGS                                                            \
   real(*restrict field)[3], real(*restrict fieldp)[3],                        \
      const real *restrict thole, const real *restrict pdamp,                  \
      const real(*restrict rpole)[10], real aewald, const Box *restrict box,   \
      real off
#define SHFL_SYMB(s) __shfl_sync(ALL_LANES, sh##s, srclane)


__global__
void dfield_ewald_real_cu1(DFIELD_ARGS, const Spatial* restrict sp)
{
   const int ithread = threadIdx.x + blockIdx.x * blockDim.x;
   const int iwarp = ithread / WARP_SIZE;
   const int nwarp = blockDim.x * gridDim.x / WARP_SIZE;
   const int ilane = threadIdx.x & (WARP_SIZE - 1);


   real gxi, gyi, gzi, txi, tyi, tzi;
   __shared__ real gxk[BLOCK_DIM], gyk[BLOCK_DIM], gzk[BLOCK_DIM],
      txk[BLOCK_DIM], tyk[BLOCK_DIM], tzk[BLOCK_DIM];


   const real off2 = off * off;
   const int n = sp->n;
   const int niak = sp->niak;
   const auto* restrict sorted = sp->sorted;
   const auto* restrict iak = sp->iak;
   const auto* restrict lst = sp->lst;
   for (int iw = iwarp; iw < niak; iw += nwarp) {
      gxi = 0;
      gyi = 0;
      gzi = 0;
      txi = 0;
      tyi = 0;
      tzi = 0;
      gxk[threadIdx.x] = 0;
      gyk[threadIdx.x] = 0;
      gzk[threadIdx.x] = 0;
      txk[threadIdx.x] = 0;
      tyk[threadIdx.x] = 0;
      tzk[threadIdx.x] = 0;


      int atomi;
      atomi = min(iak[iw] * WARP_SIZE + ilane, n - 1);
      real xi = sorted[atomi].x;
      real yi = sorted[atomi].y;
      real zi = sorted[atomi].z;
      int i = sorted[atomi].unsorted;
      real ci = rpole[i][mpl_pme_0];
      real dix = rpole[i][mpl_pme_x];
      real diy = rpole[i][mpl_pme_y];
      real diz = rpole[i][mpl_pme_z];
      real qixx = rpole[i][mpl_pme_xx];
      real qixy = rpole[i][mpl_pme_xy];
      real qixz = rpole[i][mpl_pme_xz];
      real qiyy = rpole[i][mpl_pme_yy];
      real qiyz = rpole[i][mpl_pme_yz];
      real qizz = rpole[i][mpl_pme_zz];
      real pdi = pdamp[i];
      real pti = thole[i];


      int shatomk;
      shatomk = lst[iw * WARP_SIZE + ilane];
      real shx = sorted[shatomk].x;
      real shy = sorted[shatomk].y;
      real shz = sorted[shatomk].z;
      int shk = sorted[shatomk].unsorted;
      real shck = rpole[shk][mpl_pme_0];
      real shdkx = rpole[shk][mpl_pme_x];
      real shdky = rpole[shk][mpl_pme_y];
      real shdkz = rpole[shk][mpl_pme_z];
      real shqkxx = rpole[shk][mpl_pme_xx];
      real shqkxy = rpole[shk][mpl_pme_xy];
      real shqkxz = rpole[shk][mpl_pme_xz];
      real shqkyy = rpole[shk][mpl_pme_yy];
      real shqkyz = rpole[shk][mpl_pme_yz];
      real shqkzz = rpole[shk][mpl_pme_zz];
      real shpdk = pdamp[shk];
      real shptk = thole[shk];


      for (int j = 0; j < WARP_SIZE; ++j) {
         int srclane = (ilane + j) & (WARP_SIZE - 1);
         int atomk = SHFL_SYMB(atomk);
         real xr = __shfl_sync(ALL_LANES, shx, srclane) - xi;
         real yr = __shfl_sync(ALL_LANES, shy, srclane) - yi;
         real zr = __shfl_sync(ALL_LANES, shz, srclane) - zi;
         int k = SHFL_SYMB(k);
         real ck = SHFL_SYMB(ck);
         real dkx = SHFL_SYMB(dkx);
         real dky = SHFL_SYMB(dky);
         real dkz = SHFL_SYMB(dkz);
         real qkxx = SHFL_SYMB(qkxx);
         real qkxy = SHFL_SYMB(qkxy);
         real qkxz = SHFL_SYMB(qkxz);
         real qkyy = SHFL_SYMB(qkyy);
         real qkyz = SHFL_SYMB(qkyz);
         real qkzz = SHFL_SYMB(qkzz);
         real pdk = SHFL_SYMB(pdk);
         real ptk = SHFL_SYMB(ptk);


         PairField pairf;
         pairf.fid[0] = 0;
         pairf.fid[1] = 0;
         pairf.fid[2] = 0;
         pairf.fkd[0] = 0;
         pairf.fkd[1] = 0;
         pairf.fkd[2] = 0;
         pairf.fip[0] = 0;
         pairf.fip[1] = 0;
         pairf.fip[2] = 0;
         pairf.fkp[0] = 0;
         pairf.fkp[1] = 0;
         pairf.fkp[2] = 0;


         image(xr, yr, zr, box);
         real r2 = xr * xr + yr * yr + zr * zr;
         if (atomi < atomk && r2 <= off2) {
            pair_dfield<elec_t::ewald>(
               r2, xr, yr, zr, 1, 1, ci, dix, diy, diz, qixx, qixy, qixz, qiyy,
               qiyz, qizz, pdi, pti, ck, dkx, dky, dkz, qkxx, qkxy, qkxz, qkyy,
               qkyz, qkzz, pdk, ptk, aewald, pairf);
         } // end if (r2 <= off2)


         gxi += pairf.fid[0];
         gyi += pairf.fid[1];
         gzi += pairf.fid[2];
         gxk[srclane + (threadIdx.x - ilane)] += pairf.fkd[0];
         gyk[srclane + (threadIdx.x - ilane)] += pairf.fkd[1];
         gzk[srclane + (threadIdx.x - ilane)] += pairf.fkd[2];
         txi += pairf.fip[0];
         tyi += pairf.fip[1];
         tzi += pairf.fip[2];
         txk[srclane + (threadIdx.x - ilane)] += pairf.fkp[0];
         tyk[srclane + (threadIdx.x - ilane)] += pairf.fkp[1];
         tzk[srclane + (threadIdx.x - ilane)] += pairf.fkp[2];
      } // end for (j)


      atomic_add(gxi, &field[i][0]);
      atomic_add(gyi, &field[i][1]);
      atomic_add(gzi, &field[i][2]);
      atomic_add(txi, &fieldp[i][0]);
      atomic_add(tyi, &fieldp[i][1]);
      atomic_add(tzi, &fieldp[i][2]);
      atomic_add(gxk[threadIdx.x], &field[shk][0]);
      atomic_add(gyk[threadIdx.x], &field[shk][1]);
      atomic_add(gzk[threadIdx.x], &field[shk][2]);
      atomic_add(txk[threadIdx.x], &fieldp[shk][0]);
      atomic_add(tyk[threadIdx.x], &fieldp[shk][1]);
      atomic_add(tzk[threadIdx.x], &fieldp[shk][2]);
   } // end for (iw)
}


__global__
void dfield_real_cu2(DFIELD_ARGS, const real* restrict x,
                     const real* restrict y, const real* restrict z,
                     int ndpexclude_, const int (*restrict dpexclude_)[2],
                     const real (*restrict dpexclude_scale_)[2])
{
   const real off2 = off * off;
   for (int ii = threadIdx.x + blockIdx.x * blockDim.x; ii < ndpexclude_;
        ii += blockDim.x * gridDim.x) {
      int i = dpexclude_[ii][0];
      int k = dpexclude_[ii][1];
      real dscale = dpexclude_scale_[ii][0];
      real pscale = dpexclude_scale_[ii][1];


      real xi = x[i];
      real yi = y[i];
      real zi = z[i];
      real ci = rpole[i][mpl_pme_0];
      real dix = rpole[i][mpl_pme_x];
      real diy = rpole[i][mpl_pme_y];
      real diz = rpole[i][mpl_pme_z];
      real qixx = rpole[i][mpl_pme_xx];
      real qixy = rpole[i][mpl_pme_xy];
      real qixz = rpole[i][mpl_pme_xz];
      real qiyy = rpole[i][mpl_pme_yy];
      real qiyz = rpole[i][mpl_pme_yz];
      real qizz = rpole[i][mpl_pme_zz];
      real pdi = pdamp[i];
      real pti = thole[i];


      real xr = x[k] - xi;
      real yr = y[k] - yi;
      real zr = z[k] - zi;


      image(xr, yr, zr, box);
      real r2 = xr * xr + yr * yr + zr * zr;
      if (r2 <= off2) {
         PairField pairf;
         pair_dfield<elec_t::coulomb>(
            r2, xr, yr, zr, dscale, pscale, ci, dix, diy, diz, qixx, qixy, qixz,
            qiyy, qiyz, qizz, pdi, pti, rpole[k][mpl_pme_0],
            rpole[k][mpl_pme_x], rpole[k][mpl_pme_y], rpole[k][mpl_pme_z],
            rpole[k][mpl_pme_xx], rpole[k][mpl_pme_xy], rpole[k][mpl_pme_xz],
            rpole[k][mpl_pme_yy], rpole[k][mpl_pme_yz], rpole[k][mpl_pme_zz],
            pdamp[k], thole[k], 0, pairf);


         atomic_add(pairf.fid[0], &field[i][0]);
         atomic_add(pairf.fid[1], &field[i][1]);
         atomic_add(pairf.fid[2], &field[i][2]);
         atomic_add(pairf.fip[0], &fieldp[i][0]);
         atomic_add(pairf.fip[1], &fieldp[i][1]);
         atomic_add(pairf.fip[2], &fieldp[i][2]);


         atomic_add(pairf.fkd[0], &field[k][0]);
         atomic_add(pairf.fkd[1], &field[k][1]);
         atomic_add(pairf.fkd[2], &field[k][2]);
         atomic_add(pairf.fkp[0], &fieldp[k][0]);
         atomic_add(pairf.fkp[1], &fieldp[k][1]);
         atomic_add(pairf.fkp[2], &fieldp[k][2]);
      } // end if (r2 <= off2)
   }    // end for (ii)
}


void dfield_ewald_real_cu(real (*field)[3], real (*fieldp)[3])
{
   const real off = switch_off(switch_ewald);
   const auto& st = *mspatial_unit;
   const auto* sp = mspatial_unit.deviceptr();


   const PMEUnit pu = ppme_unit;
   const real aewald = pu->aewald;


   launch_kernel1(WARP_SIZE * st.niak, dfield_ewald_real_cu1, field, fieldp,
                  thole, pdamp, rpole, aewald, box, off, sp);
   if (ndpexclude_ > 0)
      launch_kernel1(ndpexclude_, dfield_real_cu2, field, fieldp, thole, pdamp,
                     rpole, aewald, box, off, x, y, z, ndpexclude_, dpexclude_,
                     dpexclude_scale_);
}
TINKER_NAMESPACE_END
