#include "hip/hip_runtime.h"
#include "cudalib.h"
#include "epolar.h"
#include "induce.h"
#include "io_print.h"
#include "launch.h"
#include "tinker_rt.h"
#include <tinker/detail/inform.hh>
#include <tinker/detail/polpcg.hh>
#include <tinker/detail/polpot.hh>
#include <tinker/detail/units.hh>


TINKER_NAMESPACE_BEGIN
#define ITHREAD threadIdx.x + blockIdx.x* blockDim.x
#define STRIDE blockDim.x* gridDim.x


__global__
void pcg_udir(int n, const real* restrict polarity, real (*restrict udir)[3],
              real (*restrict udirp)[3], const real (*restrict field)[3],
              const real (*restrict fieldp)[3])
{
   for (int i = ITHREAD; i < n; i += STRIDE) {
      real poli = polarity[i];
      #pragma unroll
      for (int j = 0; j < 3; ++j) {
         udir[i][j] = poli * field[i][j];
         udirp[i][j] = poli * fieldp[i][j];
      }
   }
}


__global__
void pcg_p1(int n, const real* restrict polarity_inv, real (*restrict vec)[3],
            real (*restrict vecp)[3], const real (*restrict conj)[3],
            const real (*restrict conjp)[3], const real (*restrict field)[3],
            const real (*restrict fieldp)[3])
{
   for (int i = ITHREAD; i < n; i += STRIDE) {
      real poli_inv = polarity_inv[i];
      #pragma unroll
      for (int j = 0; j < 3; ++j) {
         vec[i][j] = poli_inv * conj[i][j] - field[i][j];
         vecp[i][j] = poli_inv * conjp[i][j] - fieldp[i][j];
      }
   }
}


__global__
void pcg_p2(int n, const real* restrict ka, const real* restrict kap,
            const real* restrict ksum, const real* restrict ksump,
            real (*restrict uind)[3], real (*restrict uinp)[3],
            const real (*restrict conj)[3], const real (*restrict conjp)[3],
            real (*restrict rsd)[3], real (*restrict rsdp)[3],
            const real (*restrict vec)[3], const real (*restrict vecp)[3])
{
   real a = *ksum / *ka;
   real ap = *ksump / *kap;
   for (int i = ITHREAD; i < n; i += STRIDE) {
      #pragma unroll
      for (int j = 0; j < 3; ++j) {
         uind[i][j] += a * conj[i][j];
         uinp[i][j] += ap * conjp[i][j];
         rsd[i][j] -= a * vec[i][j];
         rsdp[i][j] -= ap * vecp[i][j];
      }
   }
}


__global__
void pcg_p3(int n, const real* restrict ksum, const real* restrict ksump,
            const real* restrict ksum1, const real* restrict ksump1,
            real (*restrict conj)[3], real (*restrict conjp)[3],
            real (*restrict zrsd)[3], real (*restrict zrsdp)[3])
{
   real b = *ksum1 / *ksum;
   real bp = *ksump1 / *ksump;
   for (int i = ITHREAD; i < n; i += STRIDE) {
      #pragma unroll
      for (int j = 0; j < 3; ++j) {
         conj[i][j] = zrsd[i][j] + b * conj[i][j];
         conjp[i][j] = zrsdp[i][j] + bp * conjp[i][j];
      }
   }
}


__global__
void pcg_peek(int n, float pcgpeek, const real* restrict polarity,
              real (*restrict uind)[3], real (*restrict uinp)[3],
              const real (*restrict rsd)[3], const real (*restrict rsdp)[3])
{
   for (int i = ITHREAD; i < n; i += STRIDE) {
      real term = pcgpeek * polarity[i];
      #pragma unroll
      for (int j = 0; j < 3; ++j) {
         uind[i][j] += term * rsd[i][j];
         uinp[i][j] += term * rsdp[i][j];
      }
   }
}


void induce_mutual_pcg1_cu(real (*uind)[3], real (*uinp)[3])
{
   auto* field = work01_;
   auto* fieldp = work02_;
   auto* rsd = work03_;
   auto* rsdp = work04_;
   auto* zrsd = work05_;
   auto* zrsdp = work06_;
   auto* conj = work07_;
   auto* conjp = work08_;
   auto* vec = work09_;
   auto* vecp = work10_;


   const bool dirguess = polpcg::pcgguess;
   const bool sparse_prec = polpcg::pcgprec;


   // zero out the induced dipoles at each site
   darray::zero(PROCEED_NEW_Q, n, uind, uinp);


   // get the electrostatic field due to permanent multipoles
   dfield(field, fieldp);


   // direct induced dipoles
   launch_k1s(nonblk, n, pcg_udir, n, polarity, udir, udirp, field, fieldp);
   if (dirguess) {
      darray::copy(PROCEED_NEW_Q, n, uind, udir);
      darray::copy(PROCEED_NEW_Q, n, uinp, udirp);
   }


   // initial residual r(0)
   // if do not use pcgguess, r(0) = E - T Zero = E
   // if use pcgguess, r(0) = E - (inv_alpha + Tu) alpha E
   //                       = E - E -Tu udir
   //                       = -Tu udir
   if (dirguess) {
      ufield(udir, udirp, rsd, rsdp);
   } else {
      darray::copy(PROCEED_NEW_Q, n, rsd, field);
      darray::copy(PROCEED_NEW_Q, n, rsdp, fieldp);
   }


   // initial M r(0) and p(0)
   if (sparse_prec) {
      sparse_precond_build();
      sparse_precond_apply(rsd, rsdp, zrsd, zrsdp);
   } else {
      diag_precond(rsd, rsdp, zrsd, zrsdp);
   }
   darray::copy(PROCEED_NEW_Q, n, conj, zrsd);
   darray::copy(PROCEED_NEW_Q, n, conjp, zrsdp);


   // initial r(0) M r(0)
   real* sum = &((real*)dptr_buf)[0];
   real* sump = &((real*)dptr_buf)[1];
   darray::dot(PROCEED_NEW_Q, n, sum, rsd, zrsd);
   darray::dot(PROCEED_NEW_Q, n, sump, rsdp, zrsdp);


   // conjugate gradient iteration of the mutual induced dipoles
   const bool debug = inform::debug;
   const int politer = polpot::politer;
   const real poleps = polpot::poleps;
   const real debye = units::debye;
   const real pcgpeek = polpcg::pcgpeek;
   const int maxiter = 100; // see also subroutine induce0a in induce.f


   bool done = false;
   int iter = 0;
   real eps = 100;
   real epsold;


   while (!done) {
      ++iter;


      // T p and p
      // vec = (inv_alpha + Tu) conj, field = -Tu conj
      // vec = inv_alpha * conj - field
      ufield(conj, conjp, field, fieldp);
      launch_k1s(nonblk, n, pcg_p1, n, polarity_inv, vec, vecp, conj, conjp,
                 field, fieldp);


      // a <- p T p
      real* a = &((real*)dptr_buf)[2];
      real* ap = &((real*)dptr_buf)[3];
      // a <- r M r / p T p; a = sum / a; ap = sump / ap
      darray::dot(PROCEED_NEW_Q, n, a, conj, vec);
      darray::dot(PROCEED_NEW_Q, n, ap, conjp, vecp);


      // u <- u + a p
      // r <- r - a T p
      launch_k1s(nonblk, n, pcg_p2, n, a, ap, sum, sump, uind, uinp, conj,
                 conjp, rsd, rsdp, vec, vecp);


      // calculate/update M r
      if (sparse_prec)
         sparse_precond_apply(rsd, rsdp, zrsd, zrsdp);
      else
         diag_precond(rsd, rsdp, zrsd, zrsdp);


      // b = sum1 / sum; bp = sump1 / sump
      real* sum1 = &((real*)dptr_buf)[4];
      real* sump1 = &((real*)dptr_buf)[5];
      darray::dot(PROCEED_NEW_Q, n, sum1, rsd, zrsd);
      darray::dot(PROCEED_NEW_Q, n, sump1, rsdp, zrsdp);


      // calculate/update p
      launch_k1s(nonblk, n, pcg_p3, n, sum, sump, sum1, sump1, conj, conjp,
                 zrsd, zrsdp);


      // copy sum1/p to sum/p
      darray::copy(PROCEED_NEW_Q, 2, sum, sum1);


      real* epsd = &((real*)dptr_buf)[6];
      real* epsp = &((real*)dptr_buf)[7];
      darray::dot(PROCEED_NEW_Q, n, epsd, rsd, rsd);
      darray::dot(PROCEED_NEW_Q, n, epsp, rsdp, rsdp);
      check_rt(hipMemcpyAsync((real*)pinned_buf, epsd, 2 * sizeof(real),
                               hipMemcpyDeviceToHost, nonblk));
      check_rt(hipStreamSynchronize(nonblk));
      epsold = eps;
      eps = REAL_MAX(((real*)pinned_buf)[0], ((real*)pinned_buf)[1]);
      eps = debye * REAL_SQRT(eps / n);


      if (debug) {
         if (iter == 1) {
            print(stdout,
                  "\n Determination of SCF Induced Dipole Moments\n\n"
                  "{0:4s}Iter{0:4s}RMS Residual (Debye)\n\n",
                  "");
         }
         print(stdout, "{0:>8d}{2:8s}{1:<16.10f}\n", iter, eps, "");
      }


      if (eps < poleps)
         done = true;
      if (eps > epsold)
         done = true;
      if (iter >= politer)
         done = true;


      // apply a "peek" iteration to the mutual induced dipoles
      if (done) {
         launch_k1s(nonblk, n, pcg_peek, n, pcgpeek, polarity, uind, uinp, rsd,
                    rsdp);
      }
   }


   // print the results from the conjugate gradient iteration
   if (debug) {
      print(stdout,
            " Induced Dipoles :{2:4s}Iterations{0:>5d}{2:6s}RMS "
            "Residual{1:>15.10f}\n",
            iter, eps, "");
   }


   // terminate the calculation if dipoles failed to converge
   if (iter >= maxiter || eps > epsold) {
      TINKER_RT(prterr)();
      TINKER_THROW("INDUCE  --  Warning, Induced Dipoles are not Converged");
   }
}
TINKER_NAMESPACE_END
