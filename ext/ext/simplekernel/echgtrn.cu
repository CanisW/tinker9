#include "hip/hip_runtime.h"

template <class Ver>
__global__
void echgtrn_cu1(
   int n, TINKER_IMAGE_PARAMS, count_buffer restrict nc,
   energy_buffer restrict ec, virial_buffer restrict vc, grad_prec* restrict gx,
   grad_prec* restrict gy, grad_prec* restrict gz, real cut, real off,
   const unsigned* restrict minfo, int nexclude,
   const int (*restrict exclude)[2], const real (*restrict exclude_scale)[3],
   const real* restrict x, const real* restrict y, const real* restrict z,
   const Spatial::SortedAtom* restrict sorted, int nakpl,
   const int* restrict iakpl, int niak, const int* restrict iak,
   const int* restrict lst, real* restrict chgct, real* restrict dmpct, real f)
{
   constexpr bool do_a = Ver::a;
   constexpr bool do_e = Ver::e;
   constexpr bool do_v = Ver::v;
   constexpr bool do_g = Ver::g;


   const int ithread = threadIdx.x + blockIdx.x * blockDim.x;
   const int iwarp = ithread / WARP_SIZE;
   const int nwarp = blockDim.x * gridDim.x / WARP_SIZE;
   const int ilane = threadIdx.x & (WARP_SIZE - 1);


   int nctl;
   if CONSTEXPR (do_a) {
      nctl = 0;
   }
   using ebuf_prec = energy_buffer_traits::type;
   ebuf_prec ectl;
   if CONSTEXPR (do_e) {
      ectl = 0;
   }
   using vbuf_prec = virial_buffer_traits::type;
   vbuf_prec vctlxx, vctlyx, vctlzx, vctlyy, vctlzy, vctlzz;
   if CONSTEXPR (do_v) {
      vctlxx = 0;
      vctlyx = 0;
      vctlzx = 0;
      vctlyy = 0;
      vctlzy = 0;
      vctlzz = 0;
   }


   real shxi;
   real shyi;
   real shzi;
   real xk;
   real yk;
   real zk;
   real shgxi;
   real shgyi;
   real shgzi;
   real gxk;
   real gyk;
   real gzk;
   real shchgi;
   real shalphai;
   real chgk;
   real alphak;


   //* /
   // exclude
   for (int ii = ithread; ii < nexclude; ii += blockDim.x * gridDim.x) {
      if CONSTEXPR (do_g) {
         shgxi = 0;
         shgyi = 0;
         shgzi = 0;
         gxk = 0;
         gyk = 0;
         gzk = 0;
      }


      int shi = exclude[ii][0];
      int k = exclude[ii][1];
      real scalea = exclude_scale[ii][0];


      real xi = x[shi];
      real yi = y[shi];
      real zi = z[shi];
      xk = x[k];
      yk = y[k];
      zk = z[k];
      real chgi = chgct[shi];
      real alphai = dmpct[shi];
      chgk = chgct[k];
      alphak = dmpct[k];


      constexpr bool incl = true;
      real xr = xk - xi;
      real yr = yk - yi;
      real zr = zk - zi;

      MAYBE_UNUSED real dedx = 0, dedy = 0, dedz = 0;
      real r2 = image2(xr, yr, zr);
      if (r2 <= off * off and incl) {
         real r = REAL_SQRT(r2);

         MAYBE_UNUSED e_prec e, de;
         pair_chgtrn<do_g>(r, scalea, f, alphai, chgi, alphak, chgk, e, de);


         if (r2 > cut * cut) {
            real taper, dtaper;
            switch_taper5<do_g>(r, cut, off, taper, dtaper);
            if CONSTEXPR (do_g)
               de = e * dtaper + de * taper;
            if CONSTEXPR (do_e)
               e *= taper;
         }


         if CONSTEXPR (do_a)
            if (e != 0)
               nctl += 1;
         if CONSTEXPR (do_e)
            ectl += cvt_to<ebuf_prec>(e);
         if CONSTEXPR (do_g) {
            de *= REAL_RECIP(r);
            dedx = de * xr;
            dedy = de * yr;
            dedz = de * zr;

            shgxi += dedx;
            shgyi += dedy;
            shgzi += dedz;
            gxk -= dedx;
            gyk -= dedy;
            gzk -= dedz;

            if CONSTEXPR (do_v) {
               vctlxx += cvt_to<vbuf_prec>(xr * dedx);
               vctlyx += cvt_to<vbuf_prec>(yr * dedx);
               vctlzx += cvt_to<vbuf_prec>(zr * dedx);
               vctlyy += cvt_to<vbuf_prec>(yr * dedy);
               vctlzy += cvt_to<vbuf_prec>(zr * dedy);
               vctlzz += cvt_to<vbuf_prec>(zr * dedz);
            }
         }
      } // end if (include)


      if CONSTEXPR (do_g) {
         atomic_add(shgxi, gx, shi);
         atomic_add(shgyi, gy, shi);
         atomic_add(shgzi, gz, shi);
         atomic_add(gxk, gx, k);
         atomic_add(gyk, gy, k);
         atomic_add(gzk, gz, k);
      }
   }
   // */


   //* /
   // block pairs that have scale factors
   for (int iw = iwarp; iw < nakpl; iw += nwarp) {
      if CONSTEXPR (do_g) {
         shgxi = 0;
         shgyi = 0;
         shgzi = 0;
         gxk = 0;
         gyk = 0;
         gzk = 0;
      }


      int tri, tx, ty;
      tri = iakpl[iw];
      tri_to_xy(tri, tx, ty);


      int shiid = ty * WARP_SIZE + ilane;
      int shatomi = min(shiid, n - 1);
      int shi = sorted[shatomi].unsorted;
      int kid = tx * WARP_SIZE + ilane;
      int atomk = min(kid, n - 1);
      int k = sorted[atomk].unsorted;
      shxi = sorted[shatomi].x;
      shyi = sorted[shatomi].y;
      shzi = sorted[shatomi].z;
      xk = sorted[atomk].x;
      yk = sorted[atomk].y;
      zk = sorted[atomk].z;


      shchgi = chgct[shi];
      shalphai = dmpct[shi];
      chgk = chgct[k];
      alphak = dmpct[k];


      unsigned int minfo0 = minfo[iw * WARP_SIZE + ilane];


      for (int j = 0; j < WARP_SIZE; ++j) {
         int srclane = (ilane + j) & (WARP_SIZE - 1);
         int srcmask = 1 << srclane;

         int iid = shiid;
         real xi = shxi;
         real yi = shyi;
         real zi = shzi;
         real chgi = shchgi;
         real alphai = shalphai;


         bool incl = iid < kid and kid < n;
         incl = incl and (minfo0 & srcmask) == 0;
         real scalea = 1;
         real xr = xk - xi;
         real yr = yk - yi;
         real zr = zk - zi;

         MAYBE_UNUSED real dedx = 0, dedy = 0, dedz = 0;
         real r2 = image2(xr, yr, zr);
         if (r2 <= off * off and incl) {
            real r = REAL_SQRT(r2);

            MAYBE_UNUSED e_prec e, de;
            pair_chgtrn<do_g>(r, scalea, f, alphai, chgi, alphak, chgk, e, de);


            if (r2 > cut * cut) {
               real taper, dtaper;
               switch_taper5<do_g>(r, cut, off, taper, dtaper);
               if CONSTEXPR (do_g)
                  de = e * dtaper + de * taper;
               if CONSTEXPR (do_e)
                  e *= taper;
            }


            if CONSTEXPR (do_a)
               if (e != 0)
                  nctl += 1;
            if CONSTEXPR (do_e)
               ectl += cvt_to<ebuf_prec>(e);
            if CONSTEXPR (do_g) {
               de *= REAL_RECIP(r);
               dedx = de * xr;
               dedy = de * yr;
               dedz = de * zr;

               shgxi += dedx;
               shgyi += dedy;
               shgzi += dedz;
               gxk -= dedx;
               gyk -= dedy;
               gzk -= dedz;

               if CONSTEXPR (do_v) {
                  vctlxx += cvt_to<vbuf_prec>(xr * dedx);
                  vctlyx += cvt_to<vbuf_prec>(yr * dedx);
                  vctlzx += cvt_to<vbuf_prec>(zr * dedx);
                  vctlyy += cvt_to<vbuf_prec>(yr * dedy);
                  vctlzy += cvt_to<vbuf_prec>(zr * dedy);
                  vctlzz += cvt_to<vbuf_prec>(zr * dedz);
               }
            }
         } // end if (include)


         shiid = __shfl_sync(ALL_LANES, shiid, ilane + 1);
         shxi = __shfl_sync(ALL_LANES, shxi, ilane + 1);
         shyi = __shfl_sync(ALL_LANES, shyi, ilane + 1);
         shzi = __shfl_sync(ALL_LANES, shzi, ilane + 1);
         shchgi = __shfl_sync(ALL_LANES, shchgi, ilane + 1);
         shalphai = __shfl_sync(ALL_LANES, shalphai, ilane + 1);
         if CONSTEXPR (do_g) {
            shgxi = __shfl_sync(ALL_LANES, shgxi, ilane + 1);
            shgyi = __shfl_sync(ALL_LANES, shgyi, ilane + 1);
            shgzi = __shfl_sync(ALL_LANES, shgzi, ilane + 1);
         }
      }


      if CONSTEXPR (do_g) {
         atomic_add(shgxi, gx, shi);
         atomic_add(shgyi, gy, shi);
         atomic_add(shgzi, gz, shi);
         atomic_add(gxk, gx, k);
         atomic_add(gyk, gy, k);
         atomic_add(gzk, gz, k);
      }
   }
   // */


   //* /
   // block-atoms
   for (int iw = iwarp; iw < niak; iw += nwarp) {
      if CONSTEXPR (do_g) {
         shgxi = 0;
         shgyi = 0;
         shgzi = 0;
         gxk = 0;
         gyk = 0;
         gzk = 0;
      }


      int ty = iak[iw];
      int shatomi = ty * WARP_SIZE + ilane;
      int shi = sorted[shatomi].unsorted;
      int atomk = lst[iw * WARP_SIZE + ilane];
      int k = sorted[atomk].unsorted;
      shxi = sorted[shatomi].x;
      shyi = sorted[shatomi].y;
      shzi = sorted[shatomi].z;
      xk = sorted[atomk].x;
      yk = sorted[atomk].y;
      zk = sorted[atomk].z;


      shchgi = chgct[shi];
      shalphai = dmpct[shi];
      chgk = chgct[k];
      alphak = dmpct[k];


      for (int j = 0; j < WARP_SIZE; ++j) {

         real xi = shxi;
         real yi = shyi;
         real zi = shzi;
         real chgi = shchgi;
         real alphai = shalphai;


         bool incl = atomk > 0;
         real scalea = 1;
         real xr = xk - xi;
         real yr = yk - yi;
         real zr = zk - zi;

         MAYBE_UNUSED real dedx = 0, dedy = 0, dedz = 0;
         real r2 = image2(xr, yr, zr);
         if (r2 <= off * off and incl) {
            real r = REAL_SQRT(r2);

            MAYBE_UNUSED e_prec e, de;
            pair_chgtrn<do_g>(r, scalea, f, alphai, chgi, alphak, chgk, e, de);


            if (r2 > cut * cut) {
               real taper, dtaper;
               switch_taper5<do_g>(r, cut, off, taper, dtaper);
               if CONSTEXPR (do_g)
                  de = e * dtaper + de * taper;
               if CONSTEXPR (do_e)
                  e *= taper;
            }


            if CONSTEXPR (do_a)
               if (e != 0)
                  nctl += 1;
            if CONSTEXPR (do_e)
               ectl += cvt_to<ebuf_prec>(e);
            if CONSTEXPR (do_g) {
               de *= REAL_RECIP(r);
               dedx = de * xr;
               dedy = de * yr;
               dedz = de * zr;

               shgxi += dedx;
               shgyi += dedy;
               shgzi += dedz;
               gxk -= dedx;
               gyk -= dedy;
               gzk -= dedz;

               if CONSTEXPR (do_v) {
                  vctlxx += cvt_to<vbuf_prec>(xr * dedx);
                  vctlyx += cvt_to<vbuf_prec>(yr * dedx);
                  vctlzx += cvt_to<vbuf_prec>(zr * dedx);
                  vctlyy += cvt_to<vbuf_prec>(yr * dedy);
                  vctlzy += cvt_to<vbuf_prec>(zr * dedy);
                  vctlzz += cvt_to<vbuf_prec>(zr * dedz);
               }
            }
         } // end if (include)


         shxi = __shfl_sync(ALL_LANES, shxi, ilane + 1);
         shyi = __shfl_sync(ALL_LANES, shyi, ilane + 1);
         shzi = __shfl_sync(ALL_LANES, shzi, ilane + 1);
         shchgi = __shfl_sync(ALL_LANES, shchgi, ilane + 1);
         shalphai = __shfl_sync(ALL_LANES, shalphai, ilane + 1);
         if CONSTEXPR (do_g) {
            shgxi = __shfl_sync(ALL_LANES, shgxi, ilane + 1);
            shgyi = __shfl_sync(ALL_LANES, shgyi, ilane + 1);
            shgzi = __shfl_sync(ALL_LANES, shgzi, ilane + 1);
         }
      }


      if CONSTEXPR (do_g) {
         atomic_add(shgxi, gx, shi);
         atomic_add(shgyi, gy, shi);
         atomic_add(shgzi, gz, shi);
         atomic_add(gxk, gx, k);
         atomic_add(gyk, gy, k);
         atomic_add(gzk, gz, k);
      }
   }
   // */


   if CONSTEXPR (do_a) {
      atomic_add(nctl, nc, ithread);
   }
   if CONSTEXPR (do_e) {
      atomic_add(ectl, ec, ithread);
   }
   if CONSTEXPR (do_v) {
      atomic_add(vctlxx, vctlyx, vctlzx, vctlyy, vctlzy, vctlzz, vc, ithread);
   }
} // generated by ComplexKernelBuilder (ck.py) 1.5.1
